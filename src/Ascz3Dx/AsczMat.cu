#include <AsczMat.cuh>
#include <hip/hip_runtime.h>

int AsczMat::appendMaterial(Material mtl) {
    h_mtls.push_back(mtl);
    return mtlsNum++;
}

void AsczMat::freeDevice() {
    if (d_mtls) {
        hipFree(d_mtls);
        d_mtls = nullptr;
    }
}

void AsczMat::toDevice() {
    freeDevice();
    if (mtlsNum) {
        hipMalloc(&d_mtls, mtlsNum * sizeof(Material));
        hipMemcpy(d_mtls, h_mtls.data(), mtlsNum * sizeof(Material), hipMemcpyHostToDevice);
    }
}