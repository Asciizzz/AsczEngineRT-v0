#include <AsczMtl.cuh>
#include <hip/hip_runtime.h>

int AsczMtl::appendMaterial(Material mtl) {
    h_mtls.push_back(mtl);
    return mtlsNum++;
}

void AsczMtl::freeDevice() {
    if (d_mtls) {
        hipFree(d_mtls);
        d_mtls = nullptr;
    }
}

void AsczMtl::toDevice() {
    freeDevice();
    if (mtlsNum) {
        hipMalloc(&d_mtls, mtlsNum * sizeof(Material));
        hipMemcpy(d_mtls, h_mtls.data(), mtlsNum * sizeof(Material), hipMemcpyHostToDevice);
    }
}