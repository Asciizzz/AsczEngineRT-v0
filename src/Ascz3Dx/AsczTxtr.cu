#define STB_IMAGE_IMPLEMENTATION

#include <AsczTxtr.cuh>
#include <stb_image.h>
#include <hip/hip_runtime.h>
#include <iostream>

int AsczTxtr::appendTexture(const char *path) {
    int w, h, n;
    unsigned char *data = stbi_load(path, &w, &h, &n, 4);

    if (data == nullptr) return -1;

    h_paths.push_back(path);
    h_txtrPtr.push_back({w, h, txtrSize});
    txtrCount++;

    for (int y = 0; y < h; y++) {
        for (int x = 0; x < w; x++) {
            int i = ((h - y - 1) * w + x) * 4;
            float r = data[i + 0] / 255.0f;
            float g = data[i + 1] / 255.0f;
            float b = data[i + 2] / 255.0f;
            float a = data[i + 3] / 255.0f;

            h_txtrFlat.push_back({r, g, b, a});
        }
    }

    txtrSize += w * h;

    return txtrCount - 1;
}

void AsczTxtr::freeDevice() {
    if (txtrSize == 0) return;

    hipFree(d_txtrFlat);
    hipFree(d_txtrPtr);
}

void AsczTxtr::toDevice() {
    freeDevice();

    hipMalloc(&d_txtrFlat, txtrSize * sizeof(Flt4));
    hipMalloc(&d_txtrPtr, txtrCount * sizeof(TxtrPtr));

    hipMemcpy(d_txtrFlat, h_txtrFlat.data(), txtrSize * sizeof(Flt4), hipMemcpyHostToDevice);
    hipMemcpy(d_txtrPtr, h_txtrPtr.data(), txtrCount * sizeof(TxtrPtr), hipMemcpyHostToDevice);
}