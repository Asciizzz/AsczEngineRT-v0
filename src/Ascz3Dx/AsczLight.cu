#include <AsczLight.cuh>

void AsczLight::appendLight(LightSrc light) {
    h_lSrc.push_back(light);
}

void AsczLight::toDevice() {
    num = h_lSrc.size();
    hipMalloc(&d_lSrc, num * sizeof(LightSrc));
    hipMemcpy(d_lSrc, h_lSrc.data(), num * sizeof(LightSrc), hipMemcpyHostToDevice);
}

void AsczLight::freeDevice() {
    if (num == 0) return;
    hipFree(d_lSrc);
}