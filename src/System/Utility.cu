#include "hip/hip_runtime.h"
#include <Utility.cuh>

#define VectF std::vector<float>

#define VectI std::vector<int>
#define VectLLI std::vector<LLInt>
#define VectULLI std::vector<ULLInt>

#define VectBool std::vector<bool>

#define VectStr std::vector<std::string>

void Utils::appendObj(
    MeshManager &meshMgr, BvhManager &bvhMgr,
    MatManager &matMgr, TxtrManager &txtrMgr,
    const char *objPath, short placement, float scale, short fIdxBased 
) {
    std::ifstream file(objPath);
    if (!file.is_open()) return;

    Vecs3f mv;
    Vecs2f mt;
    Vecs3f mn;

    Vecs3i mfv;
    Vecs3i mft;
    Vecs3i mfn;
    VectI mfm;
    VectI mfo;

    int matIdx = 0;
    std::unordered_map<std::string, int> matMap;

    int bvhIdx = -1;

    std::string path(objPath);

    // We will use these value to shift the mesh to desired position
    float minX = INFINITY, minY = INFINITY, minZ = INFINITY;
    float maxX = -INFINITY, maxY = -INFINITY, maxZ = -INFINITY;

    VectStr lines;
    std::string line;
    while (std::getline(file, line)) {
        if (line.size() == 0 || line[0] == '#') continue;

        std::stringstream ss(line);
        std::string type; ss >> type;

        // The datatype has been sorted by their frequency
        /* Ranking:
            * v: 1 - most frequent
            * f: 2 - very frequent
            * vt: 3 - quite frequent
            * vn: 4 - quite frequent
            * o: 5 - not so frequent
            * usemtl: 6 - not so frequent
            * mtllib: 7 - only once
        */

        if (type == "v") {
            Vec3f v; ss >> v.x >> v.y >> v.z;
            v.scale(Vec3f(), scale);
            mv.push_back(v);

            minX = std::min(minX, v.x);
            minY = std::min(minY, v.y);
            minZ = std::min(minZ, v.z);
            maxX = std::max(maxX, v.x);
            maxY = std::max(maxY, v.y);
            maxZ = std::max(maxZ, v.z);
        }

        else if (type == "f") {
            Vec3i fv, ft, fn;

            VectI vs, ts, ns;
            while (ss.good()) {
                std::string vtn; ss >> vtn;
                std::stringstream ss2(vtn);

                int v, t, n;

                // Read vertex index
                ss2 >> v;

                // Check for texture index (skip if missing)
                if (ss2.peek() == '/') {
                    ss2.ignore(1); // Ignore the first '/'
                    if (ss2.peek() != '/') {
                        ss2 >> t; // Read texture index if present
                    } else {
                        t = fIdxBased - 1; // No texture index provided
                    }
                } else {
                    t = fIdxBased - 1; // No slashes, so no texture coordinate
                }

                // Check for normal index
                if (ss2.peek() == '/') {
                    ss2.ignore(1); // Ignore the second '/'
                    ss2 >> n; // Read normal index
                } else {
                    n = fIdxBased - 1 ; // No normal index provided
                }

                // Note, setting it to fIdxBased - 1
                // Helps the operation below to work
                // Not exist = -1

                vs.push_back(v - fIdxBased);
                ts.push_back(t - fIdxBased);
                ns.push_back(n - fIdxBased);
            }

            // Triangulate the face
            for (int i = 1; i < vs.size() - 1; i++) {
                fv = Vec3i(vs[0], vs[i], vs[i + 1]);
                ft = Vec3i(ts[0], ts[i], ts[i + 1]);
                fn = Vec3i(ns[0], ns[i], ns[i + 1]);

                mfv.push_back(fv);
                mft.push_back(ft);
                mfn.push_back(fn);
                mfm.push_back(matIdx);
            }
        }

        else if (type == "vt") {
            Vec2f t; ss >> t.x >> t.y;
            mt.push_back(t);
        }

        else if (type == "vn") {
            Vec3f n; ss >> n.x >> n.y >> n.z;
            n.norm(); // Just in case
            mn.push_back(n);
        }

        else if (type == "o") {
            mfo.push_back(mfv.size());

            bvhIdx = bvhMgr.appendNode(BvhNode());
            bvhMgr.h_nodes[bvhIdx].fl = mfv.size();

            if (bvhIdx > 0) {
                bvhMgr.h_nodes[bvhIdx - 1].fr = mfv.size();
            }
        }

        else if (type == "usemtl") {
            std::string matName;
            ss >> matName;

            matIdx = matMap[matName];
        }

        else if (type == "mtllib") {
            std::string mtlPath;
            ss >> mtlPath;

            std::string mtlDir = path.substr(0, path.find_last_of("/\\") + 1);
            std::ifstream mtlFile(mtlDir + mtlPath);
            if (!mtlFile.is_open()) continue;

            std::string mtlLine;
            while (std::getline(mtlFile, mtlLine)) {
                if (mtlLine.size() == 0 || mtlLine[0] == '#') continue;

                std::stringstream mtlSS(mtlLine);
                std::string mtlType;
                mtlSS >> mtlType;

                if (mtlType == "newmtl") {
                    matIdx = matMgr.appendMaterial(Material());
                    std::string matName; mtlSS >> matName;
                    matMap[matName] = matIdx;
                }

                if (mtlType == "Kd") {
                    Vec3f Kd; mtlSS >> Kd.x >> Kd.y >> Kd.z;
                    matMgr.h_mats[matIdx].Kd = Kd;
                }

                if (mtlType == "map_Kd") {
                    std::string txtrPath; mtlSS >> txtrPath;

                    matMgr.h_mats[matIdx].mapKd = txtrMgr.appendTexture(
                        (mtlDir + txtrPath).c_str()
                    );
                }

                // Additional attributes can be added here
                // Even those that do no exist in a typical .mtl file
                // for debugging of course

                // Those that do not exist in a typical .mtl file
                if (mtlType == "refl") {
                    float refl; mtlSS >> refl;
                    matMgr.h_mats[matIdx].reflect = refl;
                } else if (mtlType == "transm") {
                    float transm; mtlSS >> transm;
                    matMgr.h_mats[matIdx].transmit = transm;
                } else if (mtlType == "Fresnel") {
                    float fresnel; mtlSS >> fresnel;
                    matMgr.h_mats[matIdx].Fresnel = fresnel;
                } else if (mtlType == "Phong") { // 0: no, 1: yes
                    int phong; mtlSS >> phong;
                    matMgr.h_mats[matIdx].Phong = phong;
                }
            }
        }
    }
    mfo.push_back(mfv.size());
    mfo.erase(mfo.begin());

    bvhMgr.h_nodes.back().fr = mfv.size();

    #pragma omp parallel for
    for (size_t i = 0; i < mv.size(); i++) {
        // Shift to center of xz plane
        if (placement > 0) {
            mv[i].x -= (minX + maxX) / 2;
            mv[i].z -= (minZ + maxZ) / 2;
        }

        // Shift to center
        if (placement == 1) mv[i].y -= minY;
        // Shift to floor (y = 0)
        else if (placement == 2) mv[i].y -= minY;
    }

    #pragma omp parallel for
    for (int n = 0; n < bvhMgr.num; n++) {
        BvhNode &node = bvhMgr.h_nodes[n];

        for (int i = node.fl; i < node.fr; i++) {
            Vec3i fv = mfv[i];

            node.recalc(mv[fv.x]);
            node.recalc(mv[fv.y]);
            node.recalc(mv[fv.z]);
        }
    }

    MeshStruct mesh;
    mesh.v = mv;
    mesh.t = mt;
    mesh.n = mn;
    mesh.fv = mfv;
    mesh.ft = mft;
    mesh.fn = mfn;
    mesh.fm = mfm;
    mesh.fo = mfo;

    meshMgr.appendMesh(mesh);
}