#include <MatManager.cuh>
#include <hip/hip_runtime.h>

int MatManager::appendMaterial(Material mat) {
    h_mats.push_back(mat);
    return matsNum++;
}

void MatManager::freeDevice() {
    if (d_mats) {
        hipFree(d_mats);
        d_mats = nullptr;
    }
}

void MatManager::hostToDevice() {
    freeDevice();
    if (matsNum) {
        hipMalloc(&d_mats, matsNum * sizeof(Material));
        hipMemcpy(d_mats, h_mats.data(), matsNum * sizeof(Material), hipMemcpyHostToDevice);
    }
}