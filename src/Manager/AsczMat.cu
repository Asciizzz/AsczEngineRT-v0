#include <AsczMat.cuh>
#include <hip/hip_runtime.h>

int AsczMat::appendMaterial(Material mat) {
    h_mats.push_back(mat);
    return matsNum++;
}

void AsczMat::freeDevice() {
    if (d_mats) {
        hipFree(d_mats);
        d_mats = nullptr;
    }
}

void AsczMat::toDevice() {
    freeDevice();
    if (matsNum) {
        hipMalloc(&d_mats, matsNum * sizeof(Material));
        hipMemcpy(d_mats, h_mats.data(), matsNum * sizeof(Material), hipMemcpyHostToDevice);
    }
}