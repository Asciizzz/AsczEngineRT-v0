#include "hip/hip_runtime.h"
#include <AsczMesh.cuh>
#include <hip/hip_runtime.h>
#include <omp.h>

void AsczMesh::appendMesh(MeshStruct mesh) {
    #pragma omp parallel for
    for (int i = 0; i < mesh.SOrF.size(); ++i) {
        SOrF.push_back(mesh.SOrF[i] + h_fv.size());
    }

    OrSO.push_back(SOrF.size());

    #pragma omp parallel for
    for (int i = 0; i < mesh.fv.size(); ++i) {
        // Offset the indices
        h_fv.push_back(mesh.fv[i] + h_v.size());
        h_ft.push_back(mesh.ft[i] + h_t.size());
        h_fn.push_back(mesh.fn[i] + h_n.size());
        h_fm.push_back(mesh.fm[i]);
    }

    h_v.insert(h_v.end(), mesh.v.begin(), mesh.v.end());
    h_t.insert(h_t.end(), mesh.t.begin(), mesh.t.end());
    h_n.insert(h_n.end(), mesh.n.begin(), mesh.n.end());

    vNum = h_v.size();
    tNum = h_t.size();
    nNum = h_n.size();
    fNum = h_fv.size();
}

void AsczMesh::freeDevice() {
    if (d_v) { hipFree(d_v); d_v = nullptr; }
    if (d_t) { hipFree(d_t); d_t = nullptr; }
    if (d_n) { hipFree(d_n); d_n = nullptr; }

    if (d_fv) { hipFree(d_fv); d_fv = nullptr; }
    if (d_ft) { hipFree(d_ft); d_ft = nullptr; }
    if (d_fn) { hipFree(d_fn); d_fn = nullptr; }
    if (d_fm) { hipFree(d_fm); d_fm = nullptr; }
}

void AsczMesh::computeData() {
    h_fABmin.resize(fNum);
    h_fABmax.resize(fNum);
    h_fABcen.resize(fNum);

    #pragma omp parallel for
    for (int i = 0; i < fNum; ++i) {
        Flt3 fABmin = Flt3(INFINITY);
        Flt3 fABmax = Flt3(-INFINITY);
        Flt3 fABcen = Flt3();

        for (int j = 0; j < 3; ++j) {
            Flt3 v = h_v[h_fv[i][j]];

            fABmin.x = fminf(fABmin.x, v.x);
            fABmin.y = fminf(fABmin.y, v.y);
            fABmin.z = fminf(fABmin.z, v.z);

            fABmax.x = fmaxf(fABmax.x, v.x);
            fABmax.y = fmaxf(fABmax.y, v.y);
            fABmax.z = fmaxf(fABmax.z, v.z);

            fABcen += v;
        }

        h_fABmin[i] = fABmin;
        h_fABmax[i] = fABmax;
        h_fABcen[i] = fABcen / 3;
    }
}

void AsczMesh::toDevice() {
    freeDevice();
    computeData();

    // -------------------------------------- //    

    hipMalloc(&d_v, vNum * sizeof(Flt3));
    hipMalloc(&d_t, tNum * sizeof(Flt2));
    hipMalloc(&d_n, nNum * sizeof(Flt3));

    hipMalloc(&d_fv, fNum * sizeof(Int3));
    hipMalloc(&d_ft, fNum * sizeof(Int3));
    hipMalloc(&d_fn, fNum * sizeof(Int3));
    hipMalloc(&d_fm, fNum * sizeof(int));

    hipMalloc(&d_fABmin, fNum * sizeof(Flt3));
    hipMalloc(&d_fABmax, fNum * sizeof(Flt3));
    hipMalloc(&d_fABcen, fNum * sizeof(Flt3));

    // -------------------------------------- //

    hipMemcpy(d_v, h_v.data(), vNum * sizeof(Flt3), hipMemcpyHostToDevice);
    hipMemcpy(d_t, h_t.data(), tNum * sizeof(Flt2), hipMemcpyHostToDevice);
    hipMemcpy(d_n, h_n.data(), nNum * sizeof(Flt3), hipMemcpyHostToDevice);

    hipMemcpy(d_fv, h_fv.data(), fNum * sizeof(Int3), hipMemcpyHostToDevice);
    hipMemcpy(d_ft, h_ft.data(), fNum * sizeof(Int3), hipMemcpyHostToDevice);
    hipMemcpy(d_fn, h_fn.data(), fNum * sizeof(Int3), hipMemcpyHostToDevice);
    hipMemcpy(d_fm, h_fm.data(), fNum * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_fABmin, h_fABmin.data(), fNum * sizeof(Flt3), hipMemcpyHostToDevice);
    hipMemcpy(d_fABmax, h_fABmax.data(), fNum * sizeof(Flt3), hipMemcpyHostToDevice);    
    hipMemcpy(d_fABcen, h_fABcen.data(), fNum * sizeof(Flt3), hipMemcpyHostToDevice);
}