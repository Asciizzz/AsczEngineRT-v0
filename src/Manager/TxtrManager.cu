#include <TxtrManager.cuh>
#include <SFML/Graphics.hpp>
#include <hip/hip_runtime.h>

int TxtrManager::appendTexture(const char *path) {
    sf::Image img;
    img.loadFromFile(path);

    int w = img.getSize().x;
    int h = img.getSize().y;

    TxtrPtr txtrPtr = {w, h, txtrSize};
    h_txtrPtr.push_back(txtrPtr);
    h_paths.push_back(path);

    txtrSize += w * h;

    for (int y = 0; y < h; ++y) {
        for (int x = 0; x < w; ++x) {
            sf::Color c = img.getPixel(x, h - y - 1);
            h_txtrFlat.push_back(Vec3f(c.r / 255.0f, c.g / 255.0f, c.b / 255.0f));
        }
    }

    // Return the index of the texture
    return txtrCount++;
}

void TxtrManager::freeDevice() {
    if (txtrSize == 0) return;

    hipFree(d_txtrFlat);
    hipFree(d_txtrPtr);
}

void TxtrManager::toDevice() {
    freeDevice();

    hipMalloc(&d_txtrFlat, txtrSize * sizeof(Vec3f));
    hipMalloc(&d_txtrPtr, txtrCount * sizeof(TxtrPtr));

    hipMemcpy(d_txtrFlat, h_txtrFlat.data(), txtrSize * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(d_txtrPtr, h_txtrPtr.data(), txtrCount * sizeof(TxtrPtr), hipMemcpyHostToDevice);
}