#include "hip/hip_runtime.h"
#include <MeshManager.cuh>
#include <hip/hip_runtime.h>
#include <omp.h>

void MeshManager::appendMesh(MeshStruct mesh) {
    #pragma omp parallel for
    for (int i = 0; i < mesh.SOrF.size(); i++) {
        SOrF.push_back(mesh.SOrF[i] + h_fv.size());
    }

    OrSO.push_back(SOrF.size());

    #pragma omp parallel for
    for (int i = 0; i < mesh.fv.size(); i++) {
        // Offset the indices
        h_fv.push_back(mesh.fv[i] + h_v.size());
        h_ft.push_back(mesh.ft[i] + h_t.size());
        h_fn.push_back(mesh.fn[i] + h_n.size());
        h_fm.push_back(mesh.fm[i]);
    }

    h_v.insert(h_v.end(), mesh.v.begin(), mesh.v.end());
    h_t.insert(h_t.end(), mesh.t.begin(), mesh.t.end());
    h_n.insert(h_n.end(), mesh.n.begin(), mesh.n.end());

    vNum = h_v.size();
    tNum = h_t.size();
    nNum = h_n.size();
    fNum = h_fv.size();
}

void MeshManager::freeDevice() {
    if (d_v) { hipFree(d_v); d_v = nullptr; }
    if (d_t) { hipFree(d_t); d_t = nullptr; }
    if (d_n) { hipFree(d_n); d_n = nullptr; }

    if (d_fv) { hipFree(d_fv); d_fv = nullptr; }
    if (d_ft) { hipFree(d_ft); d_ft = nullptr; }
    if (d_fn) { hipFree(d_fn); d_fn = nullptr; }
    if (d_fm) { hipFree(d_fm); d_fm = nullptr; }
}

void MeshManager::computeData() {
    // Debug: print OrSO and SOrF
    for (int i = 0; i < OrSO.size(); i++) {
        std::cout << OrSO[i] << std::endl;
    }

    std::cout << "----------------" << std::endl;

    for (int i = 0; i < SOrF.size(); i++) {
        std::cout << SOrF[i] << std::endl;
    }   


    // These data will be useful for BVH construction

    h_fABmin.resize(fNum);
    h_fABmax.resize(fNum);
    h_fABcen.resize(fNum);

    #pragma omp parallel for
    for (int i = 0; i < fNum; i++) {
        Vec3f fABmin = Vec3f(INFINITY);
        Vec3f fABmax = Vec3f(-INFINITY);
        Vec3f fABcen = Vec3f();

        for (int j = 0; j < 3; j++) {
            Vec3f v = h_v[h_fv[i][j]];

            fABmin.x = fminf(fABmin.x, v.x);
            fABmin.y = fminf(fABmin.y, v.y);
            fABmin.z = fminf(fABmin.z, v.z);

            fABmax.x = fmaxf(fABmax.x, v.x);
            fABmax.y = fmaxf(fABmax.y, v.y);
            fABmax.z = fmaxf(fABmax.z, v.z);

            fABcen += v;
        }

        h_fABmin[i] = fABmin;
        h_fABmax[i] = fABmax;
        h_fABcen[i] = fABcen / 3;
    }
}

void MeshManager::hostToDevice() {
    freeDevice();
    computeData();

    // -------------------------------------- //    

    hipMalloc(&d_v, vNum * sizeof(Vec3f));
    hipMalloc(&d_t, tNum * sizeof(Vec2f));
    hipMalloc(&d_n, nNum * sizeof(Vec3f));

    hipMalloc(&d_fv, fNum * sizeof(Vec3i));
    hipMalloc(&d_ft, fNum * sizeof(Vec3i));
    hipMalloc(&d_fn, fNum * sizeof(Vec3i));
    hipMalloc(&d_fm, fNum * sizeof(int));

    hipMalloc(&d_fABmin, fNum * sizeof(Vec3f));
    hipMalloc(&d_fABmax, fNum * sizeof(Vec3f));
    hipMalloc(&d_fABcen, fNum * sizeof(Vec3f));

    // -------------------------------------- //

    hipMemcpy(d_v, h_v.data(), vNum * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(d_t, h_t.data(), tNum * sizeof(Vec2f), hipMemcpyHostToDevice);
    hipMemcpy(d_n, h_n.data(), nNum * sizeof(Vec3f), hipMemcpyHostToDevice);

    hipMemcpy(d_fv, h_fv.data(), fNum * sizeof(Vec3i), hipMemcpyHostToDevice);
    hipMemcpy(d_ft, h_ft.data(), fNum * sizeof(Vec3i), hipMemcpyHostToDevice);
    hipMemcpy(d_fn, h_fn.data(), fNum * sizeof(Vec3i), hipMemcpyHostToDevice);
    hipMemcpy(d_fm, h_fm.data(), fNum * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_fABmin, h_fABmin.data(), fNum * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(d_fABmax, h_fABmax.data(), fNum * sizeof(Vec3f), hipMemcpyHostToDevice);    
    hipMemcpy(d_fABcen, h_fABcen.data(), fNum * sizeof(Vec3f), hipMemcpyHostToDevice);
}