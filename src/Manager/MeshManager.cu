#include <MeshManager.cuh>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <iostream>

void MeshManager::appendMesh(MeshStruct mesh) {
    #pragma omp parallel for
    for (int i = 0; i < mesh.fv.size(); i++) {
        // Offset the indices
        h_fv.push_back(mesh.fv[i] + h_v.size());
        h_ft.push_back(mesh.ft[i] + h_t.size());
        h_fn.push_back(mesh.fn[i] + h_n.size());
        h_fm.push_back(mesh.fm[i]);
    }

    h_v.insert(h_v.end(), mesh.v.begin(), mesh.v.end());
    h_t.insert(h_t.end(), mesh.t.begin(), mesh.t.end());
    h_n.insert(h_n.end(), mesh.n.begin(), mesh.n.end());

    vNum = h_v.size();
    tNum = h_t.size();
    nNum = h_n.size();
    fNum = h_fv.size();
}

void MeshManager::freeDevice() {
    if (d_v) { hipFree(d_v); d_v = nullptr; }
    if (d_t) { hipFree(d_t); d_t = nullptr; }
    if (d_n) { hipFree(d_n); d_n = nullptr; }

    if (d_fv) { hipFree(d_fv); d_fv = nullptr; }
    if (d_ft) { hipFree(d_ft); d_ft = nullptr; }
    if (d_fn) { hipFree(d_fn); d_fn = nullptr; }
    if (d_fm) { hipFree(d_fm); d_fm = nullptr; }
}

void MeshManager::hostToDevice() {
    freeDevice();

    hipMalloc(&d_v, vNum * sizeof(Vec3f));
    hipMalloc(&d_t, tNum * sizeof(Vec2f));
    hipMalloc(&d_n, nNum * sizeof(Vec3f));

    hipMalloc(&d_fv, fNum * sizeof(Vec3i));
    hipMalloc(&d_ft, fNum * sizeof(Vec3i));
    hipMalloc(&d_fn, fNum * sizeof(Vec3i));
    hipMalloc(&d_fm, fNum * sizeof(int));

    hipMemcpy(d_v, h_v.data(), vNum * sizeof(Vec3f), hipMemcpyHostToDevice);
    hipMemcpy(d_t, h_t.data(), tNum * sizeof(Vec2f), hipMemcpyHostToDevice);
    hipMemcpy(d_n, h_n.data(), nNum * sizeof(Vec3f), hipMemcpyHostToDevice);

    hipMemcpy(d_fv, h_fv.data(), fNum * sizeof(Vec3i), hipMemcpyHostToDevice);
    hipMemcpy(d_ft, h_ft.data(), fNum * sizeof(Vec3i), hipMemcpyHostToDevice);
    hipMemcpy(d_fn, h_fn.data(), fNum * sizeof(Vec3i), hipMemcpyHostToDevice);
    hipMemcpy(d_fm, h_fm.data(), fNum * sizeof(int), hipMemcpyHostToDevice);
}