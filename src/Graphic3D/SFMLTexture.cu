#include "hip/hip_runtime.h"
#include <SFMLTexture.cuh>

SFMLTexture::SFMLTexture(int width, int height) { resize(width, height); }

void SFMLTexture::free() {
    delete[] sfPixel;
    hipFree(d_sfPixel);
}

void SFMLTexture::resize(int width, int height) {
    texture.create(width, height);
    sprite.setTexture(texture);

    // Allocate memory for the Pixel buffer
    sfPixel = new sf::Uint8[width * height * 4];
    hipMalloc(&d_sfPixel, width * height * 4 * sizeof(sf::Uint8));

    pixelCount = width * height * 4;
    blockNum = (width * height + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
}

void SFMLTexture::updateTexture(Flt3 *frmbuffer, int b_w, int b_h) {
    int bCount = (b_w * b_h + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    updateTextureKernel<<<bCount, THREADS_PER_BLOCK>>>(
        d_sfPixel, frmbuffer, b_w, b_h
    );
    hipMemcpy(sfPixel, d_sfPixel, pixelCount * sizeof(sf::Uint8), hipMemcpyDeviceToHost);
    texture.update(sfPixel);
}

// Kernel for updating the texture
__global__ void updateTextureKernel(
    sf::Uint8 *d_sfPixel, Flt3 *frmbuffer, int b_w, int b_h
) {
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tIdx >= b_w * b_h) return;

    int x = tIdx % b_w;
    int y = tIdx / b_w;
    int b_i = x + y * b_w;

    // Limit the color to 0-255 
    frmbuffer[b_i].x = fminf(fmaxf(frmbuffer[b_i].x, 0.0f), 1.0f);
    frmbuffer[b_i].y = fminf(fmaxf(frmbuffer[b_i].y, 0.0f), 1.0f);
    frmbuffer[b_i].z = fminf(fmaxf(frmbuffer[b_i].z, 0.0f), 1.0f);

    int p_i = b_i * 4;
    d_sfPixel[p_i + 0] = (sf::Uint8)(frmbuffer[b_i].x * 255);
    d_sfPixel[p_i + 1] = (sf::Uint8)(frmbuffer[b_i].y * 255);
    d_sfPixel[p_i + 2] = (sf::Uint8)(frmbuffer[b_i].z * 255);
    d_sfPixel[p_i + 3] = 255;
}