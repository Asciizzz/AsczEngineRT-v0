#include "hip/hip_runtime.h"
#include <RayTrace.cuh>

__global__ void clearFrameBuffer(Vec3f *framebuffer, int frmW, int frmH) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < frmW * frmH) framebuffer[i] = Vec3f(0, 0, 0);
}

__global__ void iterativeRayTracing(
    Camera camera, Vec3f *framebuffer, int frmW, int frmH, // In-out
    Vec3f *txtrFlat, TxtrPtr *txtrPtr, // Textures
    Material *mats, // Materials
    // Mesh data
    Vec3f *mv, Vec2f *mt, Vec3f *mn, // Primitive data
    Vec3i *mfv, Vec3i *mft, Vec3i *mfn, int *mfm, // Face data
    int fNum, // Number of faces

    // BVH in the near future

    Vec3f lightSrc
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= frmW * frmH) return;

    int x = i % frmW;
    int y = i / frmW;

    Ray primaryRay = camera.castRay(x, y, frmW, frmH);

    const double EPSILON_1 = 0.001;
    const double EPSILON_2 = 0.00001;
    const int MAX_RAYS = 10;

    // Very important note:
    // If mfv.z = -2, the face is a Sphere!
    // The mv[mfv.x] is the center of the sphere
    // The mv[mfv.y].x is the radius of the sphere

    // Iterative ray tracing

    Ray rays[MAX_RAYS] = { primaryRay };
    RayHit hits[MAX_RAYS] = { RayHit() };
    float weights[MAX_RAYS] = { 1.0f };
    Vec3f vrtx[MAX_RAYS];
    Vec2f txtr[MAX_RAYS];
    Vec3f colr[MAX_RAYS];
    Vec3f nrml[MAX_RAYS];

    int rnum = 0;

    for (int r = 0; r < rnum + 1; r++) {
        if (rnum > MAX_RAYS - 4) break;

        Ray &ray = rays[r];
        RayHit &hit = hits[r];

        for (int i = 0; i < fNum; i++) {
            Vec3i &fv = mfv[i];

            // Get the vertices
            Vec3f v0 = mv[fv.x];
            Vec3f v1 = mv[fv.y];
            Vec3f v2 = mv[fv.z];

            if (fv.z == -2) {
                // Sphere
                Vec3f center = v0;
                float radius = v1.x;

                Vec3f l = center - ray.origin;
                float tca = l * ray.direction;
                float d2 = l * l - tca * tca;

                float rSq = radius * radius;

                if (d2 > rSq) continue;

                float thc = sqrt(rSq - d2);
                float t0 = tca - thc;
                float t1 = tca + thc;

                if (t0 < 0) t0 = t1;

                if (t0 > EPSILON_2 && t0 < hit.t) {
                    hit.hit = true;
                    hit.idx = i;
                    hit.t = t0;
                }

                continue;
            }


            Vec3f e1 = v1 - v0;
            Vec3f e2 = v2 - v0;
            Vec3f h = ray.direction & e2;
            float a = e1 * h;

            if (a > -EPSILON_2 && a < EPSILON_2) continue;

            float f = 1.0f / a;
            Vec3f s = ray.origin - v0;
            float u = f * (s * h);

            if (u < 0.0f || u > 1.0f) continue;

            Vec3f q = s & e1;
            float v = f * (ray.direction * q);

            if (v < 0.0f || u + v > 1.0f) continue;

            float t = f * (e2 * q);

            if (t > EPSILON_2 && t < hit.t) {
                hit.hit = true;
                hit.idx = i;
                hit.t = t;
                hit.u = u;
                hit.v = v;
            }
        }

    // =========================================================================
    // =========================================================================
    // =========================================================================

        if (!hit.hit) continue;

        // Get the face data
        int fIdx = hit.idx; int &fm = mfm[fIdx];
        Vec3i &ft = mft[fIdx]; Vec3i &fn = mfn[fIdx];

        const Material &mat = mats[fm];

        float w = 1 - hit.u - hit.v;
        vrtx[r] = ray.origin + ray.direction * hit.t;

        if (fn.x > -1) {
            Vec3f &n0 = mn[fn.x], &n1 = mn[fn.y], &n2 = mn[fn.z];
            nrml[r] = n0 * w + n1 * hit.u + n2 * hit.v;
        }

        if (mat.mapKd > -1) {
            Vec2f &t0 = mt[ft.x], &t1 = mt[ft.y], &t2 = mt[ft.z];
            txtr[r] = t0 * w + t1 * hit.u + t2 * hit.v;
            // Modulo 1
            txtr[r].x -= floor(txtr[r].x);
            txtr[r].y -= floor(txtr[r].y);

            int mapKd = mat.mapKd;
            int w = txtrPtr[mapKd].w;
            int h = txtrPtr[mapKd].h;
            int off = txtrPtr[mapKd].off;

            int txtrX = txtr[r].x * w;
            int txtrY = txtr[r].y * h;

            int mapKd2 = txtrX + txtrY * w + off;
            colr[r] = txtrFlat[mapKd2];
        } else {
            colr[r] = mat.Kd;
        }

        // Shadow ray
        Vec3f lightDir = lightSrc - vrtx[r]; lightDir.norm();
        Vec3f lightOrigin = vrtx[r] + lightDir * EPSILON_1;
        Ray shadowRay(lightOrigin, lightDir);
        bool shadow = false;

        for (int i = 0; i < fNum; i++) {
            if (i == fIdx) continue;

            Vec3i &fv = mfv[i];

            // Get the vertices
            Vec3f v0 = mv[fv.x];
            Vec3f v1 = mv[fv.y];
            Vec3f v2 = mv[fv.z];

            Vec3f e1 = v1 - v0;
            Vec3f e2 = v2 - v0;
            Vec3f h = shadowRay.direction & e2;
            float a = e1 * h;

            if (a > -EPSILON_2 && a < EPSILON_2) continue;

            float f = 1.0f / a;
            Vec3f s = shadowRay.origin - v0;
            float u = f * (s * h);

            if (u < 0.0f || u > 1.0f) continue;

            Vec3f q = s & e1;
            float v = f * (shadowRay.direction * q);

            if (v < 0.0f || u + v > 1.0f) continue;

            float t = f * (e2 * q);

            if (t > EPSILON_2) {
                shadow = true;
                break;
            }
        }

        if (shadow) weights[r] *= 0.3;

        // // Apply very basic lighting with light ray from the top
        // float diff = nrml[r] * lightDir;
        // if (diff < 0) diff = 0;

        // diff = 0.3 + diff * 0.7;
        // colr[r] *= diff;

        if (mat.reflect > 0.0f) {
            float weightLeft = weights[r] * mat.reflect;
            weights[r] *= (1 - mat.reflect);

            Vec3f reflDir = ray.reflect(nrml[r]);
            Vec3f reflOrigin = vrtx[r] + nrml[r] * EPSILON_1;

            rays[++rnum] = Ray(reflOrigin, reflDir);
            hits[rnum] = RayHit();
            weights[rnum] = weightLeft;
        } else if (mat.reflect == -1) {
            // Schlick's approximation
            float cosI = (-ray.direction) * nrml[r];
            if (cosI < 0) cosI = -cosI;

            // Find the fresnel coefficient
            float R = pow(1 - cosI, 5);

            float weightLeft = weights[r] * R;
            weights[r] *= (1 - R);

            Vec3f reflDir = ray.reflect(nrml[r]);
            Vec3f reflOrigin = vrtx[r] + nrml[r] * EPSILON_1;

            rays[++rnum] = Ray(reflOrigin, reflDir);
            hits[rnum] = RayHit();
            weights[rnum] = weightLeft;
        }
        else if (mat.transmit > 0.0f) {
            float weightLeft = weights[r] * mat.transmit;
            weights[r] *= (1 - mat.transmit);

            Vec3f transOrg = vrtx[r] + ray.direction * EPSILON_1;

            rays[++rnum] = Ray(transOrg, ray.direction);
            hits[rnum] = RayHit();
            weights[rnum] = weightLeft;
        }
        else if (mat.Fresnel > 0.0f) {
            float weightLeft = weights[r] * mat.Fresnel;
            weights[r] *= (1 - mat.Fresnel);

            // Schlick's approximation
            float cosI = (-ray.direction) * nrml[r];
            if (cosI < 0) cosI = -cosI;

            // Find the fresnel coefficient
            float R = pow(1 - cosI, 5);
            float Rrefl = R * weightLeft;
            float Rrefr = (1 - R) * weightLeft;

            // Refraction (for the time being just tranparent)
            Vec3f refrDir = ray.direction;
            Vec3f refrOrigin = vrtx[r] + refrDir * EPSILON_1;

            rays[++rnum] = Ray(refrOrigin, refrDir);
            hits[rnum] = RayHit();
            weights[rnum] = Rrefr;

            // Reflection
            Vec3f reflDir = ray.reflect(nrml[r]);
            Vec3f reflOrigin = vrtx[r] + nrml[r] * EPSILON_1;

            rays[++rnum] = Ray(reflOrigin, reflDir);
            hits[rnum] = RayHit();
            weights[rnum] = Rrefl;
        }
    }

    Vec3f finalColr(0, 0, 0);
    for (int i = 0; i <= rnum; i++) {
        finalColr += colr[i] * weights[i];
    }

    framebuffer[i] = finalColr;
}