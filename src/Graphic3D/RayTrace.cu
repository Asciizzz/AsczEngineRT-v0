#include "hip/hip_runtime.h"
#include <RayTrace.cuh>

#include <hiprand/hiprand_kernel.h>

__global__ void iterativeRayTracing(
    Camera camera, Flt3 *frmbuffer, int frmW, int frmH, // In-out
    Flt4 *txtrFlat, TxtrPtr *txtrPtr, // Textures
    Material *mats, // Materials
    // Mesh data
    Flt3 *mv, Flt2 *mt, Flt3 *mn, // Primitive data
    Int3 *mfv, Int3 *mft, Int3 *mfn, int *mfm, // Face data
    int fNum, // Number of faces

    // BVH data
    int *fidx, DevNode *nodes, int nNum,

    Flt3 lightSrc,

    hiprandState *randState
) {
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tIdx >= frmW * frmH) return;

    hiprandState localRand = randState[tIdx];

    int x = tIdx % frmW, y = tIdx / frmW;
    Ray primaryRay = camera.castRay(x, y, frmW, frmH);

    const int MAX_RAYS = 8;
    const int MAX_DEPTH = 32;

    Ray rstack[MAX_RAYS]; // Ray stack
    int rs_top = 0; // Ray stack top
    rstack[rs_top++] = primaryRay;

    int nstack[MAX_DEPTH]; // Node stack
    int ns_top = 0; // Node stack top

    int rnum = 0;
    Flt3 resultColr = Flt3(0, 0, 0);
    while (rs_top > 0) {
        Ray &ray = rstack[--rs_top];
        RayHit hit;

        // Ray with little contribution
        if (ray.w < 0.01f) continue;

        ns_top = 0;
        nstack[ns_top++] = 0; // Start with root

        while (ns_top > 0) {
            int nidx = nstack[--ns_top];
            DevNode &node = nodes[nidx];

            float hitDist = node.hitDist(ray.o, ray.invd);
            if (hitDist < 0 || hitDist > hit.t) continue;

            if (!node.leaf) {
                float ldist = nodes[node.l].hitDist(ray.o, ray.invd);
                float rdist = nodes[node.r].hitDist(ray.o, ray.invd);

                // Early exit
                if (ldist < 0 && rdist < 0) continue;
                // Push the valid node
                else if (ldist < 0) nstack[ns_top++] = node.r;
                else if (rdist < 0) nstack[ns_top++] = node.l;
                // Push the closest node first
                else {
                    nstack[ns_top++] = ldist < rdist ? node.r : node.l;
                    nstack[ns_top++] = ldist < rdist ? node.l : node.r;
                }

                continue;
            }

            for (int i = node.l; i < node.r; ++i) {
                int fi = fidx[i];
                Int3 &fv = mfv[fi];

                Flt3 v0 = mv[fv.x];
                Flt3 v1 = mv[fv.y];
                Flt3 v2 = mv[fv.z];

                Flt3 e1 = v1 - v0;
                Flt3 e2 = v2 - v0;
                Flt3 h = ray.d & e2;
                float a = e1 * h;

                if (a > -EPSILON_2 && a < EPSILON_2) continue;

                float f = 1.0f / a;
                Flt3 s = ray.o - v0;
                float u = f * (s * h);

                if (u < 0.0f || u > 1.0f) continue;

                Flt3 q = s & e1;
                float v = f * (ray.d * q);

                if (v < 0.0f || u + v > 1.0f) continue;

                float t = f * (e2 * q);

                if (t > EPSILON_2 && t < hit.t) {
                    hit.idx = fi;
                    hit.t = t;
                    hit.u = u;
                    hit.v = v;
                }
            }
        }

    // =========================================================================
    // =========================================================================
    // =========================================================================

        if (hit.idx == -1) continue;

        // Get the face data
        int hidx = hit.idx; int &fm = mfm[hidx];
        float hitw = 1 - hit.u - hit.v;
        const Material &mat = mats[fm];

        // Vertex interpolation
        Flt3 vrtx = ray.o + ray.d * hit.t;

        // Normal interpolation
        Int3 &fn = mfn[hidx];
        Flt3 nrml;
        if (fn.x > -1) {
            Flt3 &n0 = mn[fn.x], &n1 = mn[fn.y], &n2 = mn[fn.z];
            nrml = n0 * hitw + n1 * hit.u + n2 * hit.v;
            nrml.norm();
        }

        Flt3 colr;
        // Color/Texture interpolation
        if (mat.mapKd > -1) {
            Int3 &ft = mft[hidx];
            Flt2 &t0 = mt[ft.x], &t1 = mt[ft.y], &t2 = mt[ft.z];
            Flt2 txtr = t0 * hitw + t1 * hit.u + t2 * hit.v;
            // Modulo 1
            txtr.x -= floor(txtr.x);
            txtr.y -= floor(txtr.y);

            int mapKd = mat.mapKd;
            int tw = txtrPtr[mapKd].w;
            int th = txtrPtr[mapKd].h;
            int toff = txtrPtr[mapKd].off;

            int txtrX = txtr.x * tw;
            int txtrY = txtr.y * th;

            int mapKd2 = txtrX + txtrY * tw + toff;
            Flt4 tColr = txtrFlat[mapKd2];

            // if (tColr.w < 1.0f) {
            //     float wLeft = ray.w * (1 - tColr.w);
            //     ray.w *= tColr.w;

            //     // Create a new ray
            //     if (rs_top + 1 < MAX_RAYS) {
            //         Flt3 d = ray.d;
            //         Flt3 o = vrtx + d * EPSILON_2;

            //         rstack[rs_top++] = Ray(o, d, wLeft, ray.Ni);
            //     }
            // }

            colr = Flt3(tColr.x, tColr.y, tColr.z);
        } else {
            colr = mat.Kd;
        }

        // Light ray
        Flt3 lightDir = vrtx - lightSrc;
        float lightDist = lightDir.mag();
        lightDir /= lightDist;
        Flt3 lightDirInv = 1.0f / lightDir;

        Flt3 shadwColor(0, 0, 0);
        float lightIntens = 1.0f;
        int lightPass = 0;

        ns_top = 0;
        nstack[ns_top++] = 0; // Start with root

        while (ns_top > 0) {
            int idx = nstack[--ns_top];
            DevNode &node = nodes[idx];

            float hitDist = node.hitDist(lightSrc, lightDirInv);
            if (hitDist < 0 || hitDist > lightDist) continue;

            if (!node.leaf) {
                float ldist = nodes[node.l].hitDist(lightSrc, lightDirInv);
                float rdist = nodes[node.r].hitDist(lightSrc, lightDirInv);

                if (ldist < 0 && rdist < 0) continue;
                else if (ldist < 0) nstack[ns_top++] = node.r;
                else if (rdist < 0) nstack[ns_top++] = node.l;
                else {
                    nstack[ns_top++] = ldist < rdist ? node.r : node.l;
                    nstack[ns_top++] = ldist < rdist ? node.l : node.r;
                }

                continue;
            }

            for (int i = node.l; i < node.r; ++i) {
                int fi = fidx[i];
                if (fi == hit.idx) continue;

                Int3 &fv = mfv[fi];

                Flt3 v0 = mv[fv.x];
                Flt3 v1 = mv[fv.y];
                Flt3 v2 = mv[fv.z];

                Flt3 e1 = v1 - v0;
                Flt3 e2 = v2 - v0;
                Flt3 h = lightDir & e2;
                float a = e1 * h;

                if (a > -EPSILON_2 && a < EPSILON_2) continue;

                float f = 1.0f / a;
                Flt3 s = lightSrc - v0;
                float u = f * (s * h);

                if (u < 0.0f || u > 1.0f) continue;

                Flt3 q = s & e1;
                float v = f * (lightDir * q);

                if (v < 0.0f || u + v > 1.0f) continue;

                float t = f * (e2 * q);

                if (t > EPSILON_2 && t < lightDist) {
                    const Material &mat = mats[mfm[fi]];

                    if (mat.transmit > 0.0f) {
                        lightPass++;
                        lightIntens *= mat.transmit;

                        // Perform interpolation to get the color
                        if (mat.mapKd > -1) {
                            Int3 &ft = mft[fi];
                            float w = 1 - u - v;

                            Flt2 &t0 = mt[ft.x], &t1 = mt[ft.y], &t2 = mt[ft.z];
                            Flt2 txtr = t0 * w + t1 * u + t2 * v;
                            // Modulo 1
                            txtr.x -= floor(txtr.x);
                            txtr.y -= floor(txtr.y);

                            int mapKd = mat.mapKd;
                            int tw = txtrPtr[mapKd].w;
                            int th = txtrPtr[mapKd].h;
                            int toff = txtrPtr[mapKd].off;

                            int txtrX = txtr.x * tw;
                            int txtrY = txtr.y * th;

                            int mapKd2 = txtrX + txtrY * tw + toff;
                            Flt4 tColr = txtrFlat[mapKd2];
                            Flt3 sColr = Flt3(tColr.x, tColr.y, tColr.z);

                            shadwColor += sColr * mat.transmit;
                        } else {
                            shadwColor += mat.Kd * mat.transmit;
                        }
                    } else {
                        lightPass = 0;
                        lightIntens = 0.0f;
                        shadwColor = Flt3(0, 0, 0);
                        break;
                    }
                }
            }

            if (lightIntens < 0.01f) break;
        }

        if (lightPass > 0) shadwColor /= lightPass;

        if (mat.Phong) {
            float diff = -lightDir * nrml;
            diff = diff < 0 ? 0 : diff;
            diff = 0.3 + diff * 0.7;

            Flt3 refl = lightDir - nrml * 2 * (lightDir * nrml);
            float spec = lightIntens * pow(refl * ray.d, mat.Ns);
            spec = spec < 0 ? -spec : spec;

            colr *= diff + spec;
        }
        
        // Limit light intensity to 0.3 - 1.0
        lightIntens = 0.3 + lightIntens * 0.7;

        colr = colr * lightIntens + shadwColor * (1 - lightIntens);

        // Reflective
        if (mat.reflect > 0.0f && rs_top + 1 < MAX_RAYS) {
            float wLeft = ray.w * mat.reflect;
            ray.w *= (1 - mat.reflect);

            Flt3 reflDir = ray.reflect(nrml);
            Flt3 reflOrigin = vrtx + nrml * EPSILON_1;

            rstack[rs_top++] = Ray(reflOrigin, reflDir, wLeft, ray.Ni);
        }
        // Transparent
        else if (mat.transmit > 0.0f && rs_top + 1 < MAX_RAYS) {
            float wLeft = ray.w * mat.transmit;
            ray.w *= (1 - mat.transmit);

            Flt3 transOrg = vrtx + ray.d * EPSILON_1;

            rstack[rs_top++] = Ray(transOrg, ray.d, wLeft, mat.Ni);
        }
        // Fresnel effect
        else if (mat.Fresnel > 0.0f && rs_top + 2 < MAX_RAYS) {
            float wLeft = ray.w * mat.Fresnel;
            ray.w *= (1 - mat.Fresnel);

            // Schlick's approximation
            float cosI = (-ray.d) * nrml;
            cosI = cosI < 0 ? 0 : cosI;

            // Find the fresnel coefficient
            float R = pow(1 - cosI, 5);
            float Rrefl = R * wLeft;
            float Rrefr = (1 - R) * wLeft;

            // Refraction (for the time being just tranparent)
            Flt3 refrDir = ray.d;
            Flt3 refrOrigin = vrtx + refrDir * EPSILON_1;
            rstack[rs_top++] = Ray(refrOrigin, refrDir, Rrefr, ray.Ni);

            // Reflection
            Flt3 reflDir = ray.reflect(nrml);
            Flt3 reflOrigin = vrtx + nrml * EPSILON_1;
            rstack[rs_top++] = Ray(reflOrigin, reflDir, Rrefl, ray.Ni);
        }

        resultColr += colr * ray.w;
    }

    frmbuffer[tIdx] = resultColr;
}