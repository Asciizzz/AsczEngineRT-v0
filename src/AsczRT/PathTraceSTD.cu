#include "hip/hip_runtime.h"
#include <PathTraceSTD.cuh>
#include <AzDevMath.cuh>

__global__ void pathtraceSTDKernel(
    AsczCam camera, float *frmx, float *frmy, float *frmz, int frmw, int frmh,
    // Primitive data
    float *vx, float *vy, float *vz, float *tx, float *ty, float *nx, float *ny, float *nz,
    // Geometry data
    int *fv0, int *fv1, int *fv2, int *ft0, int *ft1, int *ft2, int *fn0, int *fn1, int *fn2, int *fm,
    // Materials
    AzMtl *mats, int *lsrc, int lNum,
    // Textures
    float *tr, float *tg, float *tb, float *ta, int *tw, int *th, int *toff,
    // BVH data
    float *mi_x, float *mi_y, float *mi_z, float *mx_x, float *mx_y, float *mx_z, int *pl, int *pr, bool *lf, int *gIdx,

    // Additional Debug Data
    hiprandState *rnd
) {
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tIdx >= frmw * frmh) return;

    const int MAX_BOUNCES = 4;
    const int MAX_NODES = 64;

    int tX = tIdx % frmw;
    int tY = tIdx / frmw;

    float R_rndA = hiprand_uniform(&rnd[tIdx]);
    float R_rndB = hiprand_uniform(&rnd[tIdx]);

    Ray R_cast = camera.castRay(tX, tY, frmw, frmh, R_rndA, R_rndB);

    float R_ox  = R_cast.ox,  R_oy  = R_cast.oy,  R_oz  = R_cast.oz;  // Origin
    float R_dx  = R_cast.dx,  R_dy  = R_cast.dy,  R_dz  = R_cast.dz;  // Direction
    float R_rdx = R_cast.rdx, R_rdy = R_cast.rdy, R_rdz = R_cast.rdz; // Inverse direction
    int RIgnore = R_cast.ignore; // Ignore face index
    // float RIor = R_cast.Ior;     // Index of refraction

    int nstack[MAX_NODES];
    int ns_top = 0;

    float THRU_x = 1.0f, THRU_y = 1.0f, THRU_z = 1.0f; // Throughput
    float RADI_x = 0.0f, RADI_y = 0.0f, RADI_z = 0.0f; // Radiance

    int R_bounce = 0;
    while (R_bounce < MAX_BOUNCES) {
        int H_Idx = -1;
        float H_t = 1e9f;
        float H_u = 0.0f;
        float H_v = 0.0f;
        float H_w = 0.0f;

        ns_top = 0;
        nstack[ns_top++] = 0;

        while (ns_top > 0) {
            int nidx = nstack[--ns_top];

            // Check if the ray is outside the bounding box
            float t1n = (mi_x[nidx] - R_ox) * R_rdx;
            float t2n = (mx_x[nidx] - R_ox) * R_rdx;
            float t3n = (mi_y[nidx] - R_oy) * R_rdy;
            float t4n = (mx_y[nidx] - R_oy) * R_rdy;
            float t5n = (mi_z[nidx] - R_oz) * R_rdz;
            float t6n = (mx_z[nidx] - R_oz) * R_rdz;

            float tminn1 = fminf(t1n, t2n), tmaxn1 = fmaxf(t1n, t2n);
            float tminn2 = fminf(t3n, t4n), tmaxn2 = fmaxf(t3n, t4n);
            float tminn3 = fminf(t5n, t6n), tmaxn3 = fmaxf(t5n, t6n);
    
            float tminn = fmaxf(fmaxf(tminn1, tminn2), tminn3);
            float tmaxn = fminf(fminf(tmaxn1, tmaxn2), tmaxn3);

            bool nOut = R_ox < mi_x[nidx] | R_ox > mx_x[nidx] |
                        R_oy < mi_y[nidx] | R_oy > mx_y[nidx] |
                        R_oz < mi_z[nidx] | R_oz > mx_z[nidx];
            bool nMiss = tmaxn < tminn | (tminn < 0 & nOut) | tminn > H_t;

            if (nMiss) continue;

            // If node is not a leaf:
            if (!lf[nidx]) {
                // Find the distance to the left child
                int tcl = pl[nidx];
                float t1l = (mi_x[tcl] - R_ox) * R_rdx;
                float t2l = (mx_x[tcl] - R_ox) * R_rdx;
                float t3l = (mi_y[tcl] - R_oy) * R_rdy;
                float t4l = (mx_y[tcl] - R_oy) * R_rdy;
                float t5l = (mi_z[tcl] - R_oz) * R_rdz;
                float t6l = (mx_z[tcl] - R_oz) * R_rdz;

                float tminl1 = fminf(t1l, t2l), tmaxl1 = fmaxf(t1l, t2l);
                float tminl2 = fminf(t3l, t4l), tmaxl2 = fmaxf(t3l, t4l);
                float tminl3 = fminf(t5l, t6l), tmaxl3 = fmaxf(t5l, t6l);
    
                float tminl = fmaxf(fmaxf(tminl1, tminl2), tminl3);
                float tmaxl = fminf(fminf(tmaxl1, tmaxl2), tmaxl3);

                bool lOut = R_ox < mi_x[tcl] | R_ox > mx_x[tcl] |
                            R_oy < mi_y[tcl] | R_oy > mx_y[tcl] |
                            R_oz < mi_z[tcl] | R_oz > mx_z[tcl];
                bool lMiss = tmaxl < tminl | tminl < 0;
                float lDist = (-lMiss + tminl * !lMiss) * lOut;

                // Find the distance to the right child
                int tcr = pr[nidx];
                float t1r = (mi_x[tcr] - R_ox) * R_rdx;
                float t2r = (mx_x[tcr] - R_ox) * R_rdx;
                float t3r = (mi_y[tcr] - R_oy) * R_rdy;
                float t4r = (mx_y[tcr] - R_oy) * R_rdy;
                float t5r = (mi_z[tcr] - R_oz) * R_rdz;
                float t6r = (mx_z[tcr] - R_oz) * R_rdz;

                float tminr1 = fminf(t1r, t2r), tmaxr1 = fmaxf(t1r, t2r);
                float tminr2 = fminf(t3r, t4r), tmaxr2 = fmaxf(t3r, t4r);
                float tminr3 = fminf(t5r, t6r), tmaxr3 = fmaxf(t5r, t6r);
    
                float tminr = fmaxf(fmaxf(tminr1, tminr2), tminr3);
                float tmaxr = fminf(fminf(tmaxr1, tmaxr2), tmaxr3);

                bool rOut = R_ox < mi_x[tcr] | R_ox > mx_x[tcr] |
                            R_oy < mi_y[tcr] | R_oy > mx_y[tcr] |
                            R_oz < mi_z[tcr] | R_oz > mx_z[tcr];
                bool rMiss = tmaxr < tminr | tminr < 0;
                float rDist = (-rMiss + tminr * !rMiss) * rOut;


                // Child ordering for closer intersection and early exit
                bool lcloser = lDist < rDist;

                nstack[ns_top] = tcr * lcloser + tcl * !lcloser;
                ns_top += (rDist >= 0) * lcloser + (lDist >= 0) * !lcloser;

                nstack[ns_top] = tcl * lcloser + tcr * !lcloser;
                ns_top += (lDist >= 0) * lcloser + (rDist >= 0) * !lcloser;

                continue;
            }

            for (int i = pl[nidx]; i < pr[nidx]; ++i) {
                int gi = gIdx[i];

                bool hit = gi != RIgnore;

                float e1x = vx[fv1[gi]] - vx[fv0[gi]];
                float e1y = vy[fv1[gi]] - vy[fv0[gi]];
                float e1z = vz[fv1[gi]] - vz[fv0[gi]];

                float e2x = vx[fv2[gi]] - vx[fv0[gi]];
                float e2y = vy[fv2[gi]] - vy[fv0[gi]];
                float e2z = vz[fv2[gi]] - vz[fv0[gi]];

                float hx = R_dy * e2z - R_dz * e2y;
                float hy = R_dz * e2x - R_dx * e2z;
                float hz = R_dx * e2y - R_dy * e2x;

                float a = e1x * hx + e1y * hy + e1z * hz;

                hit &= a != 0.0f;
                a = !hit + a;

                float sx = R_ox - vx[fv0[gi]];
                float sy = R_oy - vy[fv0[gi]];
                float sz = R_oz - vz[fv0[gi]];

                float f = 1.0f / a;

                float u = f * (sx * hx + sy * hy + sz * hz);

                hit &= u >= 0.0f & u <= 1.0f;

                float qx = sy * e1z - sz * e1y;
                float qy = sz * e1x - sx * e1z;
                float qz = sx * e1y - sy * e1x;

                float v = f * (R_dx * qx + R_dy * qy + R_dz * qz);
                float w = 1.0f - u - v;

                hit &= v >= 0.0f & w >= 0.0f;

                float t = f * (e2x * qx + e2y * qy + e2z * qz);

                hit &= t > 0.0f & t < H_t;

                H_t = t * hit + H_t * !hit;
                H_u = u * hit + H_u * !hit;
                H_v = v * hit + H_v * !hit;
                H_w = w * hit + H_w * !hit;
                H_Idx = gi * hit + H_Idx * !hit;
            }
        }

        if (H_Idx == -1) {
            // Mess around with these values for fun
            // float3 ground = { 0.01f, 0.01f, 0.03f };
            // float3 skyHorizon = { 0.01f, 0.01f, 0.03f };
            // float3 skyZenith = { 0.00f, 0.00f, 0.00f };
            // float3 sunDir = { -1, -1, 1 };
            // float sunFocus = 169.0f, sunIntensity = 0.6f;

            float3 ground = { 1.00f, 1.00f, 1.00f };
            float3 skyHorizon = { 1.00f, 1.00f, 1.00f };
            float3 skyZenith = { 0.20f, 0.30f, 1.00f };
            float3 sunDir = { -1, -1, 1 };
            float sunFocus = 100.0f, sunIntensity = 8.0f;

            // float sunMag = sqrtf(sunDir.x * sunDir.x + sunDir.y * sunDir.y + sunDir.z * sunDir.z);
            float rsunMag = AzDevMath::rsqrt(sunDir.x * sunDir.x + sunDir.y * sunDir.y + sunDir.z * sunDir.z);
            sunDir.x *= rsunMag; sunDir.y *= rsunMag; sunDir.z *= rsunMag;

            // Sky calculation
            float sky_t = R_dy * 2.2f;
            sky_t = fmaxf(0.0f, fminf(1.0f, sky_t));
            float skyGradT = powf(sky_t, 0.35f);
            float skyGradR = skyHorizon.x * (1.0f - skyGradT) + skyZenith.x * skyGradT;
            float skyGradG = skyHorizon.y * (1.0f - skyGradT) + skyZenith.y * skyGradT;
            float skyGradB = skyHorizon.z * (1.0f - skyGradT) + skyZenith.z * skyGradT;

            // Sun calculation
            float SdotR = sunDir.x * R_dx + sunDir.y * R_dy + sunDir.z * R_dz;
            SdotR *= -(SdotR < 0.0f);
            float sun_t = powf(SdotR, sunFocus) * sunIntensity;
            bool sky_mask = R_dy > 0.0f;

            // Final color calculation
            float final_r = ground.x * !sky_mask + (skyGradR + sun_t) * sky_mask;
            float final_g = ground.y * !sky_mask + (skyGradG + sun_t) * sky_mask;
            float final_b = ground.z * !sky_mask + (skyGradB + sun_t) * sky_mask;

            RADI_x += final_r * THRU_x;
            RADI_y += final_g * THRU_y;
            RADI_z += final_b * THRU_z;

            break;
        }

        // Get the face data
        const AzMtl &H_m = mats[fm[H_Idx]];

        // Texture interpolation (if available)
        int ht0 = ft0[H_Idx], ht1 = ft1[H_Idx], ht2 = ft2[H_Idx];
        float H_tu = tx[ht0] * H_w + tx[ht1] * H_u + tx[ht2] * H_v;
        float H_tv = ty[ht0] * H_w + ty[ht1] * H_u + ty[ht2] * H_v;
        H_tu -= floor(H_tu); H_tv -= floor(H_tv);

        int H_alb_map = H_m.AlbMap;
        int H_tw = tw[H_alb_map];
        int H_th = th[H_alb_map];
        int H_toff = toff[H_alb_map];

        int H_tx = (int)(H_tu * H_tw);
        int H_ty = (int)(H_tv * H_th);
        int H_tidx = H_toff + H_ty * H_tw + H_tx;

        bool H_hasT = H_m.AlbMap > 0; // T mask
        float H_alb_x = tr[H_tidx] * H_hasT + H_m.Alb_r * !H_hasT;
        float H_alb_y = tg[H_tidx] * H_hasT + H_m.Alb_g * !H_hasT;
        float H_alb_z = tb[H_tidx] * H_hasT + H_m.Alb_b * !H_hasT;

        // Vertex linear interpolation
        float H_vx = R_ox + R_dx * H_t;
        float H_vy = R_oy + R_dy * H_t;
        float H_vz = R_oz + R_dz * H_t;

        // Normal interpolation
        int hn0 = fn0[H_Idx], hn1 = fn1[H_Idx], hn2 = fn2[H_Idx];
        float H_nx = nx[hn0] * H_w + nx[hn1] * H_u + nx[hn2] * H_v;
        float H_ny = ny[hn0] * H_w + ny[hn1] * H_u + ny[hn2] * H_v;
        float H_nz = nz[hn0] * H_w + nz[hn1] * H_u + nz[hn2] * H_v;
        bool H_hasN = hn0 > 0; // Quite important later on

// ================== Light contribution =========================

        float H_NdotR_D = H_nx * R_dx + H_ny * R_dy + H_nz * R_dz;
        H_NdotR_D = H_NdotR_D * H_NdotR_D + !H_hasN;

        float RADI_i = H_NdotR_D * H_m.Ems_i;
        RADI_x += THRU_x * H_m.Ems_r * RADI_i;
        RADI_y += THRU_y * H_m.Ems_g * RADI_i;
        RADI_z += THRU_z * H_m.Ems_b * RADI_i;

        THRU_x *= H_alb_x * (1.0f - H_m.Tr) + H_m.Tr;
        THRU_y *= H_alb_y * (1.0f - H_m.Tr) + H_m.Tr;
        THRU_z *= H_alb_z * (1.0f - H_m.Tr) + H_m.Tr;

// =================== Indirect lighting =========================

    // Random diffuse direction
        float IL_rndA = hiprand_uniform(&rnd[tIdx]);
        float IL_rndB = hiprand_uniform(&rnd[tIdx]);

        float IL_theta1 = acosf(sqrtf(1.0f - IL_rndA));
        float IL_phi = M_PIx2 * IL_rndB;

        // Cosine weighted hemisphere
        float IL_rnd_x = sinf(IL_theta1) * cosf(IL_phi);
        float IL_rnd_y = sinf(IL_theta1) * sinf(IL_phi);
        float IL_rnd_z = cosf(IL_theta1);

        // Truly random direction
        float IL_theta2 = acosf(1.0f - 2.0f * IL_rndA);
        float IL_truly_rnd_x = sinf(IL_theta2) * cosf(IL_phi);
        float IL_truly_rnd_y = sinf(IL_theta2) * sinf(IL_phi);
        float IL_truly_rnd_z = cosf(IL_theta2);

        // Construct a coordinate system
        bool IL_xGreater = fabsf(H_nx) > 0.9;
        float IL_ta_x = !IL_xGreater;
        float IL_ta_y = IL_xGreater;

        // Tangent vector
        // There supposed to also be a ta_z, but since its = 0,
        // you can ignore it in the cross product calculation
        float IL_tang_x =  IL_ta_y * H_nz;
        float IL_tang_y = -IL_ta_x * H_nz;
        float IL_tang_z = IL_ta_x * H_ny - IL_ta_y * H_nx;

        // Bitangent vector
        float IL_bitang_x = IL_tang_y * H_nz - IL_tang_z * H_ny;
        float IL_bitang_y = IL_tang_z * H_nx - IL_tang_x * H_nz;
        float IL_bitang_z = IL_tang_x * H_ny - IL_tang_y * H_nx;

        // Transform the vector to the normal space
        float IL_diff_x = IL_rnd_x * IL_tang_x + IL_rnd_y * IL_bitang_x + IL_rnd_z * H_nx;
        float IL_diff_y = IL_rnd_x * IL_tang_y + IL_rnd_y * IL_bitang_y + IL_rnd_z * H_ny;
        float IL_diff_z = IL_rnd_x * IL_tang_z + IL_rnd_y * IL_bitang_z + IL_rnd_z * H_nz;

    // Specular direction (a.k.a. reflection)
        float IL_spec_x = R_dx - H_nx * 2.0f * (H_nx * R_dx);
        float IL_spec_y = R_dy - H_ny * 2.0f * (H_ny * R_dy);
        float IL_spec_z = R_dz - H_nz * 2.0f * (H_nz * R_dz);

    // Lerp diffuse and specular from roughness/smoothness
        float IL_smooth = 1.0f - H_m.Rough;
        float IL_r_dx = IL_diff_x * H_m.Rough + IL_spec_x * IL_smooth;
        float IL_r_dy = IL_diff_y * H_m.Rough + IL_spec_y * IL_smooth;
        float IL_r_dz = IL_diff_z * H_m.Rough + IL_spec_z * IL_smooth;

        bool IL_hasTr = IL_rndA < H_m.Tr;
        IL_r_dx = IL_r_dx * !IL_hasTr + R_dx * IL_hasTr;
        IL_r_dy = IL_r_dy * !IL_hasTr + R_dy * IL_hasTr;
        IL_r_dz = IL_r_dz * !IL_hasTr + R_dz * IL_hasTr;

// =================== Construct new ray =========================
        // Origin (truly random for non-normal surfaces)
        R_ox = H_vx;
        R_oy = H_vy;
        R_oz = H_vz;
        // Direction
        R_dx = IL_r_dx * H_hasN + IL_truly_rnd_x * !H_hasN;
        R_dy = IL_r_dy * H_hasN + IL_truly_rnd_y * !H_hasN;
        R_dz = IL_r_dz * H_hasN + IL_truly_rnd_z * !H_hasN;
        // Inverse direction
        R_rdx = 1.0f / R_dx;
        R_rdy = 1.0f / R_dy;
        R_rdz = 1.0f / R_dz;
        // Other ray properties
        RIgnore = H_Idx;
        // RIor = H_m.Ior;

// =================== RUSSIAN ROULETTE TERMINATION =========================

        float THRU_lumi = 0.2126f * THRU_x + 0.7152f * THRU_y + 0.0722f * THRU_z;

        float R_survival = fminf(1.0f, THRU_lumi);
        float R_rsurvival = 1.0f / R_survival;

        bool R_survived = hiprand_uniform(&rnd[tIdx]) < R_survival;

        R_bounce += 1 + !R_survived * MAX_BOUNCES;

        // Boost for the surviving ray
        THRU_x *= R_rsurvival;
        THRU_y *= R_rsurvival;
        THRU_z *= R_rsurvival;
    }

    frmx[tIdx] = RADI_x;
    frmy[tIdx] = RADI_y;
    frmz[tIdx] = RADI_z;
}
