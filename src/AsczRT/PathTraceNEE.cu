#include "hip/hip_runtime.h"
#include <PathTraceNEE.cuh>
#include <AzDevMath.cuh>

__global__ void pathtraceNEEKernel(
    AsczCam camera, float *frmx, float *frmy, float *frmz, int frmw, int frmh,
    // Primitive data
    float *vx, float *vy, float *vz, float *tx, float *ty, float *nx, float *ny, float *nz,
    // Geometry data
    int *fv0, int *fv1, int *fv2, int *ft0, int *ft1, int *ft2, int *fn0, int *fn1, int *fn2, int *fm,
    // Materials
    AzMtl *mats, int *lsrc, int lNum,
    // Textures
    float *tr, float *tg, float *tb, float *ta, int *tw, int *th, int *toff,
    // BVH data
    float *mi_x, float *mi_y, float *mi_z, float *mx_x, float *mx_y, float *mx_z, int *pl, int *pr, bool *lf, int *gIdx,

    // Additional Debug Data
    hiprandState *rnd
) {
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tIdx >= frmw * frmh) return;

    const int MAX_BOUNCES = 4;
    const int MAX_NODES = 64;

    int tX = tIdx % frmw;
    int tY = tIdx / frmw;

    float R_rndA = hiprand_uniform(&rnd[tIdx]);
    float R_rndB = hiprand_uniform(&rnd[tIdx]);

    Ray ray = camera.castRay(tX, tY, frmw, frmh, R_rndA, R_rndB);

    float R_ox  = ray.ox,  R_oy  = ray.oy,  R_oz  = ray.oz;  // Origin
    float R_dx  = ray.dx,  R_dy  = ray.dy,  R_dz  = ray.dz;  // Direction
    float R_rdx = ray.rdx, R_rdy = ray.rdy, R_rdz = ray.rdz; // Inverse direction
    int RIgnore = ray.ignore; // Ignore face index
    // float RIor = ray.Ior;     // Index of refraction

    int nstack[MAX_NODES];
    int ns_top = 0;

    float THRU_x = 1.0f, THRU_y = 1.0f, THRU_z = 1.0f; // Throughput
    float RADI_x = 0.0f, RADI_y = 0.0f, RADI_z = 0.0f; // Radiance

    int R_bounce = 0;
    while (R_bounce < MAX_BOUNCES) {
        int H_Idx = -1;
        float H_t = 1e9f;
        float H_u = 0.0f;
        float H_v = 0.0f;
        float H_w = 0.0f;

        ns_top = 0;
        nstack[ns_top++] = 0;

        while (ns_top > 0) {
            int nidx = nstack[--ns_top];

            // Check if the ray is outside the bounding box
            float t1n = (mi_x[nidx] - R_ox) * R_rdx;
            float t2n = (mx_x[nidx] - R_ox) * R_rdx;
            float t3n = (mi_y[nidx] - R_oy) * R_rdy;
            float t4n = (mx_y[nidx] - R_oy) * R_rdy;
            float t5n = (mi_z[nidx] - R_oz) * R_rdz;
            float t6n = (mx_z[nidx] - R_oz) * R_rdz;

            float tminn = fminf(t1n, t2n), tmaxn = fmaxf(t1n, t2n);
            tminn = fmaxf(tminn, fminf(t3n, t4n)); tmaxn = fminf(tmaxn, fmaxf(t3n, t4n));
            tminn = fmaxf(tminn, fminf(t5n, t6n)); tmaxn = fminf(tmaxn, fmaxf(t5n, t6n));

            bool nOut = R_ox < mi_x[nidx] | R_ox > mx_x[nidx] |
                        R_oy < mi_y[nidx] | R_oy > mx_y[nidx] |
                        R_oz < mi_z[nidx] | R_oz > mx_z[nidx];
            float nDist = ((tmaxn < tminn | tminn < 0) ? -1 : tminn) * nOut;

            if (nDist < 0 | nDist > H_t) continue;

            // If node is not a leaf:
            if (!lf[nidx]) {
                // Find the distance to the left child
                int tcl = pl[nidx];
                float t1l = (mi_x[tcl] - R_ox) * R_rdx;
                float t2l = (mx_x[tcl] - R_ox) * R_rdx;
                float t3l = (mi_y[tcl] - R_oy) * R_rdy;
                float t4l = (mx_y[tcl] - R_oy) * R_rdy;
                float t5l = (mi_z[tcl] - R_oz) * R_rdz;
                float t6l = (mx_z[tcl] - R_oz) * R_rdz;

                float tminl = fminf(t1l, t2l), tmaxl = fmaxf(t1l, t2l);
                tminl = fmaxf(tminl, fminf(t3l, t4l)); tmaxl = fminf(tmaxl, fmaxf(t3l, t4l));
                tminl = fmaxf(tminl, fminf(t5l, t6l)); tmaxl = fminf(tmaxl, fmaxf(t5l, t6l));

                bool lOut = R_ox < mi_x[tcl] | R_ox > mx_x[tcl] |
                            R_oy < mi_y[tcl] | R_oy > mx_y[tcl] |
                            R_oz < mi_z[tcl] | R_oz > mx_z[tcl];
                float Ldist = ((tmaxl < tminl | tminl < 0) ? -1 : tminl) * lOut;

                // Find the distance to the right child
                int tcr = pr[nidx];
                float t1r = (mi_x[tcr] - R_ox) * R_rdx;
                float t2r = (mx_x[tcr] - R_ox) * R_rdx;
                float t3r = (mi_y[tcr] - R_oy) * R_rdy;
                float t4r = (mx_y[tcr] - R_oy) * R_rdy;
                float t5r = (mi_z[tcr] - R_oz) * R_rdz;
                float t6r = (mx_z[tcr] - R_oz) * R_rdz;

                float tminr = fminf(t1r, t2r), tmaxr = fmaxf(t1r, t2r);
                tminr = fmaxf(tminr, fminf(t3r, t4r)); tmaxr = fminf(tmaxr, fmaxf(t3r, t4r));
                tminr = fmaxf(tminr, fminf(t5r, t6r)); tmaxr = fminf(tmaxr, fmaxf(t5r, t6r));

                bool rOut = R_ox < mi_x[tcr] | R_ox > mx_x[tcr] |
                            R_oy < mi_y[tcr] | R_oy > mx_y[tcr] |
                            R_oz < mi_z[tcr] | R_oz > mx_z[tcr];
                float rdist = ((tmaxr < tminr | tminr < 0) ? -1 : tminr) * rOut;


                // Child ordering for closer intersection and early exit
                bool lcloser = Ldist < rdist;

                nstack[ns_top] = tcr * lcloser + tcl * !lcloser;
                ns_top += (rdist >= 0) * lcloser + (Ldist >= 0) * !lcloser;

                nstack[ns_top] = tcl * lcloser + tcr * !lcloser;
                ns_top += (Ldist >= 0) * lcloser + (rdist >= 0) * !lcloser;

                continue;
            }

            for (int i = pl[nidx]; i < pr[nidx]; ++i) {
                int gi = gIdx[i];

                bool hit = gi != RIgnore;

                float e1x = vx[fv1[gi]] - vx[fv0[gi]];
                float e1y = vy[fv1[gi]] - vy[fv0[gi]];
                float e1z = vz[fv1[gi]] - vz[fv0[gi]];

                float e2x = vx[fv2[gi]] - vx[fv0[gi]];
                float e2y = vy[fv2[gi]] - vy[fv0[gi]];
                float e2z = vz[fv2[gi]] - vz[fv0[gi]];

                float hx = R_dy * e2z - R_dz * e2y;
                float hy = R_dz * e2x - R_dx * e2z;
                float hz = R_dx * e2y - R_dy * e2x;

                float a = e1x * hx + e1y * hy + e1z * hz;

                hit &= a != 0.0f;
                a = !hit + a;

                float sx = R_ox - vx[fv0[gi]];
                float sy = R_oy - vy[fv0[gi]];
                float sz = R_oz - vz[fv0[gi]];

                float f = 1.0f / a;

                float u = f * (sx * hx + sy * hy + sz * hz);

                hit &= u >= 0.0f & u <= 1.0f;

                float qx = sy * e1z - sz * e1y;
                float qy = sz * e1x - sx * e1z;
                float qz = sx * e1y - sy * e1x;

                float v = f * (R_dx * qx + R_dy * qy + R_dz * qz);
                float w = 1.0f - u - v;

                hit &= v >= 0.0f & w >= 0.0f;

                float t = f * (e2x * qx + e2y * qy + e2z * qz);

                hit &= t > 0.0f & t < H_t;

                H_t = t * hit + H_t * !hit;
                H_u = u * hit + H_u * !hit;
                H_v = v * hit + H_v * !hit;
                H_w = w * hit + H_w * !hit;
                H_Idx = gi * hit + H_Idx * !hit;
            }
        }

        if (H_Idx == -1) {
            // Mess around with these values for fun
            // float3 ground = { 0.01f, 0.01f, 0.03f };
            // float3 skyHorizon = { 0.01f, 0.01f, 0.03f };
            // float3 skyZenith = { 0.00f, 0.00f, 0.00f };
            // float3 sunDir = { -1, -1, 1 };
            // float sunFocus = 169.0f, sunIntensity = 0.6f;

            float3 ground = { 1.00f, 1.00f, 1.00f };
            float3 skyHorizon = { 1.00f, 1.00f, 1.00f };
            float3 skyZenith = { 0.20f, 0.30f, 1.00f };
            float3 sunDir = { -1, -1, 1 };
            float sunFocus = 100.0f, sunIntensity = 8.0f;

            // float sunMag = sqrtf(sunDir.x * sunDir.x + sunDir.y * sunDir.y + sunDir.z * sunDir.z);
            float rsunMag = AzDevMath::rsqrt(sunDir.x * sunDir.x + sunDir.y * sunDir.y + sunDir.z * sunDir.z);
            sunDir.x *= rsunMag; sunDir.y *= rsunMag; sunDir.z *= rsunMag;

            // Sky calculation
            float sky_t = R_dy * 2.2f;
            sky_t = fmaxf(0.0f, fminf(1.0f, sky_t));
            float skyGradT = powf(sky_t, 0.35f);
            float skyGradR = skyHorizon.x * (1.0f - skyGradT) + skyZenith.x * skyGradT;
            float skyGradG = skyHorizon.y * (1.0f - skyGradT) + skyZenith.y * skyGradT;
            float skyGradB = skyHorizon.z * (1.0f - skyGradT) + skyZenith.z * skyGradT;

            // Sun calculation
            float SdotR = sunDir.x * R_dx + sunDir.y * R_dy + sunDir.z * R_dz;
            SdotR *= -(SdotR < 0.0f);
            float sun_t = powf(SdotR, sunFocus) * sunIntensity;
            bool sky_mask = R_dy > 0.0f;

            // // Star calculation
            // float theta = atan2f(R_dz, R_dx);
            // float phi = acosf(R_dy);

            float final_r = ground.x * !sky_mask + (skyGradR + sun_t) * sky_mask;
            float final_g = ground.y * !sky_mask + (skyGradG + sun_t) * sky_mask;
            float final_b = ground.z * !sky_mask + (skyGradB + sun_t) * sky_mask;

            RADI_x += final_r * THRU_x;
            RADI_y += final_g * THRU_y;
            RADI_z += final_b * THRU_z;

            break;
        }

/* Guide:

H_: hit point
DL_: direct light
IL_: indirect light

*/

        // Get the face material
        const AzMtl &H_m = mats[fm[H_Idx]];

        // Vertex linear interpolation
        float H_vx = R_ox + R_dx * H_t;
        float H_vy = R_oy + R_dy * H_t;
        float H_vz = R_oz + R_dz * H_t;

        // Texture interpolation (if available)
        int ht0 = ft0[H_Idx], ht1 = ft1[H_Idx], ht2 = ft2[H_Idx];
        float H_tu = tx[ht0] * H_w + tx[ht1] * H_u + tx[ht2] * H_v;
        float H_tv = ty[ht0] * H_w + ty[ht1] * H_u + ty[ht2] * H_v;
        H_tu -= floor(H_tu); H_tv -= floor(H_tv);

        int H_alb_map = H_m.AlbMap;
        int H_tw = tw[H_alb_map];
        int H_th = th[H_alb_map];
        int H_toff = toff[H_alb_map];

        int H_tx = (int)(H_tu * H_tw);
        int H_ty = (int)(H_tv * H_th);
        int H_tIdx = H_toff + H_ty * H_tw + H_tx;

        bool H_hasT = H_m.AlbMap > 0;
        float H_alb_x = tr[H_tIdx] * H_hasT + H_m.Alb_r * !H_hasT;
        float H_alb_y = tg[H_tIdx] * H_hasT + H_m.Alb_g * !H_hasT;
        float H_alb_z = tb[H_tIdx] * H_hasT + H_m.Alb_b * !H_hasT;

        // Normal interpolation
        int hn0 = fn0[H_Idx], hn1 = fn1[H_Idx], hn2 = fn2[H_Idx];
        float H_nx = nx[hn0] * H_w + nx[hn1] * H_u + nx[hn2] * H_v;
        float H_ny = ny[hn0] * H_w + ny[hn1] * H_u + ny[hn2] * H_v;
        float H_nz = nz[hn0] * H_w + nz[hn1] * H_u + nz[hn2] * H_v;
        bool H_hasN = hn0 > 0;

// =================== Direct lighting =========================

        // Retrieve the light source and it's informations

        // Sample random light source
        int DL_Idx = lNum ? lsrc[(int)(lNum * hiprand_uniform(&rnd[tIdx]))] : 0;
        const AzMtl &DL_m = mats[fm[DL_Idx]];

        // Sample random point on the light source
        float DL_u = hiprand_uniform(&rnd[tIdx]);
        float DL_v = hiprand_uniform(&rnd[tIdx]);
        bool DL_uv_valid = DL_u + DL_v < 1.0f;

        DL_u = DL_u * DL_uv_valid + (1.0f - DL_u) * !DL_uv_valid;
        DL_v = DL_v * DL_uv_valid + (1.0f - DL_v) * !DL_uv_valid;
        float DL_w = 1.0f - DL_u - DL_v;

        // Sample light's vertex
        int lv0 = fv0[DL_Idx], lv1 = fv1[DL_Idx], lv2 = fv2[DL_Idx];
        float DL_vx = vx[lv0] * DL_u + vx[lv1] * DL_v + vx[lv2] * DL_w;
        float DL_vy = vy[lv0] * DL_u + vy[lv1] * DL_v + vy[lv2] * DL_w;
        float DL_vz = vz[lv0] * DL_u + vz[lv1] * DL_v + vz[lv2] * DL_w;

        // // Sample light's normal
        // int ln0 = fn0[DL_Idx], ln1 = fn1[DL_Idx], ln2 = fn2[DL_Idx];
        // float DL_nx = nx[ln0] * DL_u + nx[ln1] * DL_v + nx[ln2] * DL_w;
        // float DL_ny = ny[ln0] * DL_u + ny[ln1] * DL_v + ny[ln2] * DL_w;
        // float DL_nz = nz[ln0] * DL_u + nz[ln1] * DL_v + nz[ln2] * DL_w;

        // Sample light's direction (not normalized)
        float DL_dx = H_vx - DL_vx;
        float DL_dy = H_vy - DL_vy;
        float DL_dz = H_vz - DL_vz;

        // Sample light's distance
        float LdistSqr = DL_dx * DL_dx + DL_dy * DL_dy + DL_dz * DL_dz;
        float LdistRsqrt = AzDevMath::rsqrt(LdistSqr + !LdistSqr); // Avoid zero division
        float Ldist = LdistSqr * LdistRsqrt;

        // Normalize light direction
        DL_dx *= LdistRsqrt;
        DL_dy *= LdistRsqrt;
        DL_dz *= LdistRsqrt;

        // Sample light's inverse direction (for traversal)
        float DL_rdx = 1.0f / DL_dx;
        float DL_rdy = 1.0f / DL_dy;
        float DL_rdz = 1.0f / DL_dz;

        // Get relevant data
        float DL_NdotH_N = DL_dx * H_nx + DL_dy * H_ny + DL_dz * H_nz;
        DL_NdotH_N = -DL_NdotH_N * (DL_NdotH_N < 0.0f) + !H_hasN;

        // Check for occlusion
        nstack[0] = 0;
        ns_top = DL_Idx > 0;
        bool occluded = false;

        while (ns_top > 0) {
            int nidx = nstack[--ns_top];

            // Check if the ray is outside the bounding box
            float t1n = (mi_x[nidx] - DL_vx) * DL_rdx;
            float t2n = (mx_x[nidx] - DL_vx) * DL_rdx;
            float t3n = (mi_y[nidx] - DL_vy) * DL_rdy;
            float t4n = (mx_y[nidx] - DL_vy) * DL_rdy;
            float t5n = (mi_z[nidx] - DL_vz) * DL_rdz;
            float t6n = (mx_z[nidx] - DL_vz) * DL_rdz;

            float tminn = fminf(t1n, t2n), tmaxn = fmaxf(t1n, t2n);
            tminn = fmaxf(tminn, fminf(t3n, t4n)); tmaxn = fminf(tmaxn, fmaxf(t3n, t4n));
            tminn = fmaxf(tminn, fminf(t5n, t6n)); tmaxn = fminf(tmaxn, fmaxf(t5n, t6n));

            bool nOut = DL_vx < mi_x[nidx] | DL_vx > mx_x[nidx] |
                        DL_vy < mi_y[nidx] | DL_vy > mx_y[nidx] |
                        DL_vz < mi_z[nidx] | DL_vz > mx_z[nidx];
            float nDist = ((tmaxn < tminn | tminn < 0) ? -1 : tminn) * nOut;

            if (nDist < 0 | nDist > Ldist) continue;

            // If node is not a leaf:
            if (!lf[nidx]) {
                // Find the distance to the left child
                int tcl = pl[nidx];
                float t1l = (mi_x[tcl] - DL_vx) * DL_rdx;
                float t2l = (mx_x[tcl] - DL_vx) * DL_rdx;
                float t3l = (mi_y[tcl] - DL_vy) * DL_rdy;
                float t4l = (mx_y[tcl] - DL_vy) * DL_rdy;
                float t5l = (mi_z[tcl] - DL_vz) * DL_rdz;
                float t6l = (mx_z[tcl] - DL_vz) * DL_rdz;

                float tminl = fminf(t1l, t2l), tmaxl = fmaxf(t1l, t2l);
                tminl = fmaxf(tminl, fminf(t3l, t4l)); tmaxl = fminf(tmaxl, fmaxf(t3l, t4l));
                tminl = fmaxf(tminl, fminf(t5l, t6l)); tmaxl = fminf(tmaxl, fmaxf(t5l, t6l));

                bool lOut = DL_vx < mi_x[tcl] | DL_vx > mx_x[tcl] |
                            DL_vy < mi_y[tcl] | DL_vy > mx_y[tcl] |
                            DL_vz < mi_z[tcl] | DL_vz > mx_z[tcl];
                float Ldist = ((tmaxl < tminl | tminl < 0) ? -1 : tminl) * lOut;

                // Find the distance to the right child
                int tcr = pr[nidx];
                float t1r = (mi_x[tcr] - DL_vx) * DL_rdx;
                float t2r = (mx_x[tcr] - DL_vx) * DL_rdx;
                float t3r = (mi_y[tcr] - DL_vy) * DL_rdy;
                float t4r = (mx_y[tcr] - DL_vy) * DL_rdy;
                float t5r = (mi_z[tcr] - DL_vz) * DL_rdz;
                float t6r = (mx_z[tcr] - DL_vz) * DL_rdz;

                float tminr = fminf(t1r, t2r), tmaxr = fmaxf(t1r, t2r);
                tminr = fmaxf(tminr, fminf(t3r, t4r)); tmaxr = fminf(tmaxr, fmaxf(t3r, t4r));
                tminr = fmaxf(tminr, fminf(t5r, t6r)); tmaxr = fminf(tmaxr, fmaxf(t5r, t6r));

                bool rOut = DL_vx < mi_x[tcr] | DL_vx > mx_x[tcr] |
                            DL_vy < mi_y[tcr] | DL_vy > mx_y[tcr] |
                            DL_vz < mi_z[tcr] | DL_vz > mx_z[tcr];
                float rdist = ((tmaxr < tminr | tminr < 0) ? -1 : tminr) * rOut;


                // Child ordering for closer intersection and early exit
                bool lcloser = Ldist < rdist;

                nstack[ns_top] = tcr * lcloser + tcl * !lcloser;
                ns_top += (rdist >= 0) * lcloser + (Ldist >= 0) * !lcloser;

                nstack[ns_top] = tcl * lcloser + tcr * !lcloser;
                ns_top += (Ldist >= 0) * lcloser + (rdist >= 0) * !lcloser;

                continue;
            }

            for (int i = pl[nidx]; i < pr[nidx]; ++i) {
                int gi = gIdx[i];

                bool hit = gi != RIgnore & gi != H_Idx;

                int f0 = fv0[gi], f1 = fv1[gi], f2 = fv2[gi];

                float e1x = vx[f1] - vx[f0];
                float e1y = vy[f1] - vy[f0];
                float e1z = vz[f1] - vz[f0];

                float e2x = vx[f2] - vx[f0];
                float e2y = vy[f2] - vy[f0];
                float e2z = vz[f2] - vz[f0];

                float hx = DL_dy * e2z - DL_dz * e2y;
                float hy = DL_dz * e2x - DL_dx * e2z;
                float hz = DL_dx * e2y - DL_dy * e2x;

                float a = e1x * hx + e1y * hy + e1z * hz;

                hit &= a != 0.0f;
                a = !hit + a;

                float sx = DL_vx - vx[f0];
                float sy = DL_vy - vy[f0];
                float sz = DL_vz - vz[f0];

                float f = 1.0f / a;

                float u = f * (sx * hx + sy * hy + sz * hz);

                hit &= u >= 0.0f & u <= 1.0f;

                float qx = sy * e1z - sz * e1y;
                float qy = sz * e1x - sx * e1z;
                float qz = sx * e1y - sy * e1x;

                float v = f * (DL_dx * qx + DL_dy * qy + DL_dz * qz);
                float w = 1.0f - u - v;

                hit &= v >= 0.0f & w >= 0.0f;

                float t = f * (e2x * qx + e2y * qy + e2z * qz);

                hit &= t > 0.0f & t < Ldist;

                occluded |= hit;
                ns_top *= !hit;
            }
        }

        float RADI_i = DL_NdotH_N * DL_m.Ems_i * !occluded;
        RADI_x += THRU_x * DL_m.Ems_r * H_alb_x * RADI_i + THRU_x * H_m.Ems_r * H_m.Ems_i;
        RADI_y += THRU_y * DL_m.Ems_g * H_alb_y * RADI_i + THRU_x * H_m.Ems_g * H_m.Ems_i;
        RADI_z += THRU_z * DL_m.Ems_b * H_alb_z * RADI_i + THRU_x * H_m.Ems_b * H_m.Ems_i;

        THRU_x *= H_alb_x * (1.0f - H_m.Tr) + H_m.Tr;
        THRU_y *= H_alb_y * (1.0f - H_m.Tr) + H_m.Tr;
        THRU_z *= H_alb_z * (1.0f - H_m.Tr) + H_m.Tr;


// =================== Indirect lighting =========================

        // Random diffuse lighting
        float IL_rndA = hiprand_uniform(&rnd[tIdx]);
        float IL_rndB = hiprand_uniform(&rnd[tIdx]);

        float IL_theta1 = acosf(sqrtf(1.0f - IL_rndA));
        float IL_phi = M_PIx2 * IL_rndB;

        // Cosine weighted hemisphere
        float IL_rnd_x = sinf(IL_theta1) * cosf(IL_phi);
        float IL_rnd_y = sinf(IL_theta1) * sinf(IL_phi);
        float IL_rnd_z = cosf(IL_theta1);

        // Truly random direction
        float IL_theta2 = acosf(1.0f - 2.0f * IL_rndA);
        float IL_truly_rnd_x = sinf(IL_theta2) * cosf(IL_phi);
        float IL_truly_rnd_y = sinf(IL_theta2) * sinf(IL_phi);
        float IL_truly_rnd_z = cosf(IL_theta2);

        // Construct a coordinate system
        bool IL_xGreater = fabsf(H_nx) > 0.9;
        float IL_ta_x = !IL_xGreater;
        float IL_ta_y = IL_xGreater;

        // Tangent vector
        // There supposed to also be a ta_z, but since its = 0,
        // you can ignore it in the cross product calculation
        float IL_tang_x =  IL_ta_y * H_nz;
        float IL_tang_y = -IL_ta_x * H_nz;
        float IL_tang_z = IL_ta_x * H_ny - IL_ta_y * H_nx;

        // Bitangent vector
        float IL_bitang_x = IL_tang_y * H_nz - IL_tang_z * H_ny;
        float IL_bitang_y = IL_tang_z * H_nx - IL_tang_x * H_nz;
        float IL_bitang_z = IL_tang_x * H_ny - IL_tang_y * H_nx;

        // Transform the vector to the normal space
        float IL_diff_x = IL_rnd_x * IL_tang_x + IL_rnd_y * IL_bitang_x + IL_rnd_z * H_nx;
        float IL_diff_y = IL_rnd_x * IL_tang_y + IL_rnd_y * IL_bitang_y + IL_rnd_z * H_ny;
        float IL_diff_z = IL_rnd_x * IL_tang_z + IL_rnd_y * IL_bitang_z + IL_rnd_z * H_nz;

    // Specular direction (a.k.a. reflection)
        float IL_spec_x = R_dx - H_nx * 2.0f * (H_nx * R_dx);
        float IL_spec_y = R_dy - H_ny * 2.0f * (H_ny * R_dy);
        float IL_spec_z = R_dz - H_nz * 2.0f * (H_nz * R_dz);

    // Lerp diffuse and specular from roughness/smoothness
        float IL_smooth = 1.0f - H_m.Rough;
        float IL_r_dx = IL_diff_x * H_m.Rough + IL_spec_x * IL_smooth;
        float IL_r_dy = IL_diff_y * H_m.Rough + IL_spec_y * IL_smooth;
        float IL_r_dz = IL_diff_z * H_m.Rough + IL_spec_z * IL_smooth;

        bool IL_hasTr = IL_rndA < H_m.Tr;
        IL_r_dx = IL_r_dx * !IL_hasTr + R_dx * IL_hasTr;
        IL_r_dy = IL_r_dy * !IL_hasTr + R_dy * IL_hasTr;
        IL_r_dz = IL_r_dz * !IL_hasTr + R_dz * IL_hasTr;

// =================== Construct new ray =========================

        // Origin (truly random for non-normal surfaces)
        R_ox = H_vx;
        R_oy = H_vy;
        R_oz = H_vz;
        // Direction
        R_dx = IL_r_dx * H_hasN + IL_truly_rnd_x * !H_hasN;
        R_dy = IL_r_dy * H_hasN + IL_truly_rnd_y * !H_hasN;
        R_dz = IL_r_dz * H_hasN + IL_truly_rnd_z * !H_hasN;
        // Inverse direction
        R_rdx = 1.0f / R_dx;
        R_rdy = 1.0f / R_dy;
        R_rdz = 1.0f / R_dz;
        // Other ray properties
        RIgnore = H_Idx;
        // RIor = H_m.Ior;

// =================== RUSSIAN ROULETTE TERMINATION =========================

        float THRU_lumi = 0.2126f * THRU_x + 0.7152f * THRU_y + 0.0722f * THRU_z;

        float R_survival = fmaxf(1.0f, THRU_lumi);
        float R_rsurvival = 1.0f / R_survival;

        bool R_survived = hiprand_uniform(&rnd[tIdx]) < R_survival;

        R_bounce += 1 + !R_survived * MAX_BOUNCES;

        // Boost for the surviving ray
        THRU_x *= R_rsurvival;
        THRU_y *= R_rsurvival;
        THRU_z *= R_rsurvival;
    }

    frmx[tIdx] = RADI_x;
    frmy[tIdx] = RADI_y;
    frmz[tIdx] = RADI_z;
}
