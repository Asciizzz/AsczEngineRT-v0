#include "hip/hip_runtime.h"
#include <RayCast.cuh>

__global__ void raycastKernel(
    AsczCam camera, float *frmx, float *frmy, float *frmz, int frmw, int frmh,
    // Primitive data
    float *vx, float *vy, float *vz, float *tx, float *ty, float *nx, float *ny, float *nz,
    // Geometry data
    int *fv0, int *fv1, int *fv2, int *ft0, int *ft1, int *ft2, int *fn0, int *fn1, int *fn2, int *fm,
    // Materials
    AzMtl *mats, 
    // Textures
    float *tr, float *tg, float *tb, float *ta, int *tw, int *th, int *toff,
    // BVH data
    float *mi_x, float *mi_y, float *mi_z, float *mx_x, float *mx_y, float *mx_z, int *pl, int *pr, bool *lf, int *gIdx,
    // Fake shading (for better feel since you can get lost in the scene)
    bool fakeShading,
    // Debugging
    float *frmdepth, int *frmmat
) {
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tIdx >= frmw * frmh) return;

    int x = tIdx % frmw;
    int y = tIdx / frmw;
    Ray ray = camera.castRay(x, y, frmw, frmh);

    // Hit info
    int hidx = -1;
    float ht = 1e9f;
    float hu = 0.0f;
    float hv = 0.0f;
    float hw = 0.0f;

    const int MAX_NODES = 64;
    int nstack[MAX_NODES] = { 0 };
    int ns_top = 1;

    while (ns_top > 0) {
        int nidx = nstack[--ns_top];

        // Check if the ray is outside the bounding box
        float t1n = (mi_x[nidx] - ray.ox) * ray.rdx;
        float t2n = (mx_x[nidx] - ray.ox) * ray.rdx;
        float t3n = (mi_y[nidx] - ray.oy) * ray.rdy;
        float t4n = (mx_y[nidx] - ray.oy) * ray.rdy;
        float t5n = (mi_z[nidx] - ray.oz) * ray.rdz;
        float t6n = (mx_z[nidx] - ray.oz) * ray.rdz;

        float tminn = fminf(t1n, t2n), tmaxn = fmaxf(t1n, t2n);
        tminn = fmaxf(tminn, fminf(t3n, t4n)); tmaxn = fminf(tmaxn, fmaxf(t3n, t4n));
        tminn = fmaxf(tminn, fminf(t5n, t6n)); tmaxn = fminf(tmaxn, fmaxf(t5n, t6n));

        bool nOut = ray.ox < mi_x[nidx] | ray.ox > mx_x[nidx] |
                    ray.oy < mi_y[nidx] | ray.oy > mx_y[nidx] |
                    ray.oz < mi_z[nidx] | ray.oz > mx_z[nidx];
        float nDist = ((tmaxn < tminn | tminn < 0) ? -1 : tminn) * nOut;

        if (nDist < 0 | nDist > ht) continue;

        // If node is not a leaf:
        if (!lf[nidx]) {
            // Find the distance to the left child
            int tcl = pl[nidx];
            float t1l = (mi_x[tcl] - ray.ox) * ray.rdx;
            float t2l = (mx_x[tcl] - ray.ox) * ray.rdx;
            float t3l = (mi_y[tcl] - ray.oy) * ray.rdy;
            float t4l = (mx_y[tcl] - ray.oy) * ray.rdy;
            float t5l = (mi_z[tcl] - ray.oz) * ray.rdz;
            float t6l = (mx_z[tcl] - ray.oz) * ray.rdz;

            float tminl = fminf(t1l, t2l), tmaxl = fmaxf(t1l, t2l);
            tminl = fmaxf(tminl, fminf(t3l, t4l)); tmaxl = fminf(tmaxl, fmaxf(t3l, t4l));
            tminl = fmaxf(tminl, fminf(t5l, t6l)); tmaxl = fminf(tmaxl, fmaxf(t5l, t6l));

            bool lOut = ray.ox < mi_x[tcl] | ray.ox > mx_x[tcl] |
                        ray.oy < mi_y[tcl] | ray.oy > mx_y[tcl] |
                        ray.oz < mi_z[tcl] | ray.oz > mx_z[tcl];
            float ldist = ((tmaxl < tminl | tminl < 0) ? -1 : tminl) * lOut;

            // Find the distance to the right child
            int tcr = pr[nidx];
            float t1r = (mi_x[tcr] - ray.ox) * ray.rdx;
            float t2r = (mx_x[tcr] - ray.ox) * ray.rdx;
            float t3r = (mi_y[tcr] - ray.oy) * ray.rdy;
            float t4r = (mx_y[tcr] - ray.oy) * ray.rdy;
            float t5r = (mi_z[tcr] - ray.oz) * ray.rdz;
            float t6r = (mx_z[tcr] - ray.oz) * ray.rdz;

            float tminr = fminf(t1r, t2r), tmaxr = fmaxf(t1r, t2r);
            tminr = fmaxf(tminr, fminf(t3r, t4r)); tmaxr = fminf(tmaxr, fmaxf(t3r, t4r));
            tminr = fmaxf(tminr, fminf(t5r, t6r)); tmaxr = fminf(tmaxr, fmaxf(t5r, t6r));

            bool rOut = ray.ox < mi_x[tcr] | ray.ox > mx_x[tcr] |
                        ray.oy < mi_y[tcr] | ray.oy > mx_y[tcr] |
                        ray.oz < mi_z[tcr] | ray.oz > mx_z[tcr];
            float rdist = ((tmaxr < tminr | tminr < 0) ? -1 : tminr) * rOut;


            // Child ordering for closer intersection and early exit
            bool lcloser = ldist < rdist;

            nstack[ns_top] = tcr * lcloser + tcl * !lcloser;
            ns_top += (rdist >= 0) * lcloser + (ldist >= 0) * !lcloser;

            nstack[ns_top] = tcl * lcloser + tcr * !lcloser;
            ns_top += (ldist >= 0) * lcloser + (rdist >= 0) * !lcloser;

            continue;
        }

        for (int i = pl[nidx]; i < pr[nidx]; ++i) {
            int gi = gIdx[i];

            bool hit = true;

            int f0 = fv0[gi];
            int f1 = fv1[gi];
            int f2 = fv2[gi];

            float e1x = vx[f1] - vx[f0];
            float e1y = vy[f1] - vy[f0];
            float e1z = vz[f1] - vz[f0];

            float e2x = vx[f2] - vx[f0];
            float e2y = vy[f2] - vy[f0];
            float e2z = vz[f2] - vz[f0];

            float hx = ray.dy * e2z - ray.dz * e2y;
            float hy = ray.dz * e2x - ray.dx * e2z;
            float hz = ray.dx * e2y - ray.dy * e2x;

            float a = e1x * hx + e1y * hy + e1z * hz;

            hit &= a != 0.0f;
            a = a == 0.0f ? 1.0f : a;

            float f = 1.0f / a;

            float sx = ray.ox - vx[f0];
            float sy = ray.oy - vy[f0];
            float sz = ray.oz - vz[f0];

            float u = f * (sx * hx + sy * hy + sz * hz);

            hit &= u >= 0.0f & u <= 1.0f;

            float qx = sy * e1z - sz * e1y;
            float qy = sz * e1x - sx * e1z;
            float qz = sx * e1y - sy * e1x;

            float v = f * (ray.dx * qx + ray.dy * qy + ray.dz * qz);
            
            float w = 1.0f - u - v;

            hit &= v >= 0.0f & w >= 0.0f;

            float t = f * (e2x * qx + e2y * qy + e2z * qz);

            hit &= t > 0.0f & t < ht;

            ht = t * hit + ht * !hit;
            hu = u * hit + hu * !hit;
            hv = v * hit + hv * !hit;
            hw = w * hit + hw * !hit;
            hidx = gi * hit + hidx * !hit;
        }
    }

    if (hidx == -1) {
        frmx[tIdx] = 0.0f;
        frmy[tIdx] = 0.0f;
        frmz[tIdx] = 0.0f;
        frmdepth[tIdx] = -1.0f;
        frmmat[tIdx] = -1;
        return;
    }

    const AzMtl &hm = mats[fm[hidx]];

    // Normal interpolation
    int n0 = fn0[hidx], n1 = fn1[hidx], n2 = fn2[hidx];
    float nrml_x = nx[n0] * hw + nx[n1] * hu + nx[n2] * hv;
    float nrml_y = ny[n0] * hw + ny[n1] * hu + ny[n2] * hv;
    float nrml_z = nz[n0] * hw + nz[n1] * hu + nz[n2] * hv;
    bool hasNrml = n0 > 0;

    // Texture interpolation (if available)
    int t0 = ft0[hidx], t1 = ft1[hidx], t2 = ft2[hidx];
    float t_u = tx[t0] * hw + tx[t1] * hu + tx[t2] * hv;
    float t_v = ty[t0] * hw + ty[t1] * hu + ty[t2] * hv;
    t_u -= floor(t_u); t_v -= floor(t_v);

    int alb_map = hm.AlbMap;
    int t_w = tw[alb_map];
    int t_h = th[alb_map];
    int t_off = toff[alb_map];

    int t_x = (int)(t_u * t_w);
    int t_y = (int)(t_v * t_h);
    int t_idx = t_off + t_y * t_w + t_x;

    bool hasTxtr = hm.AlbMap > 0;
    float alb_x = tr[t_idx] * hasTxtr + hm.Alb_r * !hasTxtr;
    float alb_y = tg[t_idx] * hasTxtr + hm.Alb_g * !hasTxtr;
    float alb_z = tb[t_idx] * hasTxtr + hm.Alb_b * !hasTxtr;

    // Fake shading
    bool fShade = fakeShading && hasNrml;
    float NdotL = nrml_x * ray.dx + nrml_y * ray.dy + nrml_z * ray.dz;
    NdotL *= NdotL;

    alb_x *= NdotL * fShade + !fShade;
    alb_y *= NdotL * fShade + !fShade;
    alb_z *= NdotL * fShade + !fShade;

    frmx[tIdx] = alb_x;
    frmy[tIdx] = alb_y;
    frmz[tIdx] = alb_z;
    frmdepth[tIdx] = ht;
    frmmat[tIdx] = fm[hidx];
};