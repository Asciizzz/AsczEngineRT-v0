#include "hip/hip_runtime.h"
#include <AsczBvh.cuh>

#include <ToDevice.cuh>
#include <algorithm>
#include <execution>
#include <omp.h>

__global__ void toSoAKernel(
    float *BV_min_x, float *BV_min_y, float *BV_min_z,
    float *BV_max_x, float *BV_max_y, float *BV_max_z,
    int *pl, int *pr, bool *lf,
    DevNode *nodes, int nNum
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= nNum) return;

    BV_min_x[idx] = nodes[idx].min_x;
    BV_min_y[idx] = nodes[idx].min_y;
    BV_min_z[idx] = nodes[idx].min_z;

    BV_max_x[idx] = nodes[idx].max_x;
    BV_max_y[idx] = nodes[idx].max_y;
    BV_max_z[idx] = nodes[idx].max_z;

    lf[idx] = nodes[idx].cl == -1;

    pl[idx] = lf[idx] ? nodes[idx].ll : nodes[idx].cl;
    pr[idx] = lf[idx] ? nodes[idx].lr : nodes[idx].cr;
}

AsczBvh::~AsczBvh() {
    hipFree(d_nodes);
    hipFree(d_min_x);
    hipFree(d_min_y);
    hipFree(d_min_z);
    hipFree(d_max_x);
    hipFree(d_max_y);
    hipFree(d_max_z);
    hipFree(d_pl);
    hipFree(d_pr);
    hipFree(d_lf);
    hipFree(d_fIdx);
}

void AsczBvh::toDevice() {
    nNum = h_nodes.size();

    hipMalloc(&d_nodes, nNum * sizeof(DevNode));
    hipMalloc(&d_min_x, nNum * sizeof(float));
    hipMalloc(&d_min_y, nNum * sizeof(float));
    hipMalloc(&d_min_z, nNum * sizeof(float));
    hipMalloc(&d_max_x, nNum * sizeof(float));
    hipMalloc(&d_max_y, nNum * sizeof(float));
    hipMalloc(&d_max_z, nNum * sizeof(float));
    hipMalloc(&d_pl, nNum * sizeof(int));
    hipMalloc(&d_pr, nNum * sizeof(int));
    hipMalloc(&d_lf, nNum * sizeof(bool));

    hipMemcpy(d_nodes, h_nodes.data(), nNum * sizeof(DevNode), hipMemcpyHostToDevice);

    toSoAKernel<<<nNum / 256 + 1, 256>>>(
        d_min_x, d_min_y, d_min_z,
        d_max_x, d_max_y, d_max_z,
        d_pl, d_pr, d_lf,
        d_nodes, nNum
    );

    ToDevice::I(h_fIdx, d_fIdx);
}

void AsczBvh::designBVH(AsczMesh &meshMgr) {
    int gNum = meshMgr.gNum;
    const AABB &GlbAB = meshMgr.GlbAB;
    // const std::vector<AABB> &O_AB = meshMgr.O_AB;
    // const std::vector<AABB> &SO_AB = meshMgr.SO_AB;
    const std::vector<AABB> &G_AB = meshMgr.G_AB;

    // Initialize h_fIdx
    h_fIdx.resize(gNum);
    #pragma omp parallel
    for (int i = 0; i < gNum; ++i) h_fIdx[i] = i;

    h_nodes.push_back({
        GlbAB.min.x, GlbAB.min.y, GlbAB.min.z,
        GlbAB.max.x, GlbAB.max.y, GlbAB.max.z,
        -1, -1, 0, gNum, 0
    });

    buildBvhTest(
        h_nodes, h_fIdx, G_AB, MAX_DEPTH, NODE_FACES, BIN_COUNT
    );
}



int AsczBvh::buildBvhTest(
    VecNode &nodes, std::vector<int> &fIdxs, const std::vector<AABB> &fABs,
    const int MAX_DEPTH, const int NODE_FACES, const int BIN_COUNT
) {
    std::queue<int> queue;
    queue.push(0);

    while (!queue.empty()) {
        int nIdx = queue.front();
        queue.pop();

        DevNode nd = nodes[nIdx];

        int nF = nd.lr - nd.ll;
        if (nF <= NODE_FACES || nd.depth >= MAX_DEPTH) {
            continue;
        }

        float nLn_x = nd.max_x - nd.min_x;
        float nLn_y = nd.max_y - nd.min_y;
        float nLn_z = nd.max_z - nd.min_z;

        int bestAxis = -1;
        int bestSplit = -1;

        float bestLab_min_x, bestLab_min_y, bestLab_min_z;
        float bestLab_max_x, bestLab_max_y, bestLab_max_z;

        float bestRab_min_x, bestRab_min_y, bestRab_min_z;
        float bestRab_max_x, bestRab_max_y, bestRab_max_z;

        float bestCost = (nLn_x * nLn_x + nLn_y * nLn_y + nLn_z * nLn_z) * nF;

        for (int a = 0; a < 3; ++a) {
            std::sort(std::execution::par,
            fIdxs.begin() + nd.ll, fIdxs.begin() + nd.lr,
            [&](int i1, int i2) {
                return fABs[i1].cent()[a] < fABs[i2].cent()[a];
            });

            for (int b = 0; b < BIN_COUNT; ++b) {
                DevNode cl, cr;

                float s1 = nd.min_x * (a == 0) + nd.min_y * (a == 1) + nd.min_z * (a == 2);
                float s2 = nLn_x * (a == 0) + nLn_y * (a == 1) + nLn_z * (a == 2);
                float splitPoint = s1 + s2 * (b + 1) / BIN_COUNT;

                int splitIdx = nd.ll;

                for (int g = nd.ll; g < nd.lr; ++g) {
                    int i = fIdxs[g];

                    float cent = fABs[i].cent()[a];

                    if (cent < splitPoint) {
                        cl.min_x = fminf(cl.min_x, fABs[i].min.x);
                        cl.min_y = fminf(cl.min_y, fABs[i].min.y);
                        cl.min_z = fminf(cl.min_z, fABs[i].min.z);

                        cl.max_x = fmaxf(cl.max_x, fABs[i].max.x);
                        cl.max_y = fmaxf(cl.max_y, fABs[i].max.y);
                        cl.max_z = fmaxf(cl.max_z, fABs[i].max.z);

                        splitIdx++;
                    }
                    else {
                        cr.min_x = fminf(cr.min_x, fABs[i].min.x);
                        cr.min_y = fminf(cr.min_y, fABs[i].min.y);
                        cr.min_z = fminf(cr.min_z, fABs[i].min.z);

                        cr.max_x = fmaxf(cr.max_x, fABs[i].max.x);
                        cr.max_y = fmaxf(cr.max_y, fABs[i].max.y);
                        cr.max_z = fmaxf(cr.max_z, fABs[i].max.z);
                    }
                }

                float lCost = (cl.max_x - cl.min_x) * (cl.max_y - cl.min_y) * (cl.max_z - cl.min_z) * (splitIdx - nd.ll);
                float rCost = (cr.max_x - cr.min_x) * (cr.max_y - cr.min_y) * (cr.max_z - cr.min_z) * (nd.lr - splitIdx);
                float cost = lCost + rCost;

                if (cost < bestCost) {
                    bestCost = cost;
                    bestAxis = a;
                    bestSplit = splitIdx;

                    bestLab_min_x = cl.min_x;
                    bestLab_min_y = cl.min_y;
                    bestLab_min_z = cl.min_z;

                    bestLab_max_x = cl.max_x;
                    bestLab_max_y = cl.max_y;
                    bestLab_max_z = cl.max_z;

                    bestRab_min_x = cr.min_x;
                    bestRab_min_y = cr.min_y;
                    bestRab_min_z = cr.min_z;

                    bestRab_max_x = cr.max_x;
                    bestRab_max_y = cr.max_y;
                    bestRab_max_z = cr.max_z;
                }
            }
        }

        if (bestAxis == -1) {
            continue;
        }

        std::sort(std::execution::par,
        fIdxs.begin() + nd.ll, fIdxs.begin() + nd.lr,
        [&](int i1, int i2) {
            return fABs[i1].cent()[bestAxis] < fABs[i2].cent()[bestAxis];
        });

        // Create left and right node
        DevNode nl = {
            bestLab_min_x, bestLab_min_y, bestLab_min_z,
            bestLab_max_x, bestLab_max_y, bestLab_max_z,
            -1, -1, nd.ll, bestSplit, nd.depth + 1
        };

        DevNode nr = {
            bestRab_min_x, bestRab_min_y, bestRab_min_z,
            bestRab_max_x, bestRab_max_y, bestRab_max_z,
            -1, -1, bestSplit, nd.lr, nd.depth + 1
        };

        int lIdx = nodes.size();
        nodes.push_back(nl);

        int rIdx = nodes.size();
        nodes.push_back(nr);

        nd.cl = lIdx;
        nd.cr = rIdx;
        nodes[nIdx] = nd;

        queue.push(lIdx);
        queue.push(rIdx);
    }

    return 0;
}