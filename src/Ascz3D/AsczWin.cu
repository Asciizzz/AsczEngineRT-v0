#include "hip/hip_runtime.h"
#include <AsczWin.cuh>
#include <AzDevMath.cuh>
#include <hip/hip_runtime.h>

#include <string>

#define _GAMMA 1.0f/2.2f

__global__ void copyToDrawBuffer(Flt3 *frmbuffer, unsigned int *drawbuffer, int width, int height, bool toneMap) {
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tIdx < width * height) {
        int x = tIdx % width;
        int y = tIdx / width;
        int i = y * width + x;

        Flt3 color = frmbuffer[i];
        color.clamp(0.0f, 1.0f);

        color.x = AzDevMath::ACESFilm(powf(color.x, _GAMMA)) * toneMap + color.x * !toneMap;
        color.y = AzDevMath::ACESFilm(powf(color.y, _GAMMA)) * toneMap + color.y * !toneMap;
        color.z = AzDevMath::ACESFilm(powf(color.z, _GAMMA)) * toneMap + color.z * !toneMap;

        drawbuffer[i] = (int(color.x * 255) << 16) | (int(color.y * 255) << 8) | int(color.z * 255);
    }
}

// Constructor
AsczWin::AsczWin(int w, int h, std::wstring t) : width(w), height(h), title(t) {
    InitWindow();
    InitGDI();

    blockCount = (width * height + threadCount - 1) / threadCount;

    h_drawbuffer = new unsigned int[width * height];
    hipMalloc(&d_drawbuffer, width * height * sizeof(unsigned int));

    hipMalloc(&d_frmbuffer1, width * height * sizeof(Flt3));
    hipMalloc(&d_frmbuffer2, width * height * sizeof(Flt3));
    hipMalloc(&d_frmbuffer3, width * height * sizeof(Flt3));
}

void AsczWin::InitWindow() {
    WNDCLASS wc = { 0 };
    wc.lpfnWndProc = WindowProc;
    wc.hInstance = GetModuleHandle(nullptr);
    wc.lpszClassName = L"Win32App";
    wc.cbWndExtra = sizeof(AsczWin*);  // Store pointer to our Window instance
    RegisterClass(&wc);

    hwnd = CreateWindowEx(
        0, L"Win32App", title.c_str(),
        WS_OVERLAPPEDWINDOW | WS_VISIBLE,
        CW_USEDEFAULT, CW_USEDEFAULT, width, height, nullptr, nullptr, GetModuleHandle(nullptr), this
    );

    if (!hwnd) {
        std::cerr << "Failed to create window!\n";
        exit(1);
    }

    hdc = GetDC(hwnd);
}

void AsczWin::InitGDI() {
    bmi = {};
    bmi.bmiHeader.biSize = sizeof(BITMAPINFOHEADER);
    bmi.bmiHeader.biWidth = width;
    bmi.bmiHeader.biHeight = -height;  // Negative to ensure top-down DIB
    bmi.bmiHeader.biPlanes = 1;
    bmi.bmiHeader.biBitCount = 32;
    bmi.bmiHeader.biCompression = BI_RGB;
}


// Debug
void AsczWin::DrawText(HDC hdc, int x, int y, const AsczDebug &db) {
    SetBkMode(hdc, TRANSPARENT);
    SetTextColor(hdc, RGB(db.color.x, db.color.y, db.color.z));
    TextOut(hdc, x, y, db.text.c_str(), db.text.length());
}
void AsczWin::appendDebug(std::wstring text, Int3 color) {
    AsczDebug db;
    db.text = text;
    db.color = color;
    debugs.push_back(db);
}
void AsczWin::appendDebug(std::string text, Int3 color) {
    appendDebug(std::wstring(text.begin(), text.end()), color);
}

// Framebuffer
void AsczWin::DrawFramebuffer(int buffer) {
    copyToDrawBuffer<<<blockCount, threadCount>>>(
        buffer == 1 ? d_frmbuffer1 :
        buffer == 2 ? d_frmbuffer2 : d_frmbuffer3,
        d_drawbuffer, width, height, buffer > 1
    );

    hipMemcpy(h_drawbuffer, d_drawbuffer, width * height * sizeof(unsigned int), hipMemcpyDeviceToHost);
    StretchDIBits(hdc, 0, 0, width, height, 0, 0, width, height, h_drawbuffer, &bmi, DIB_RGB_COLORS, SRCCOPY);
}

// Draw everything
void AsczWin::Draw(int buffer, bool debug) {
    DrawFramebuffer(buffer);

    if (!debug) return;

    for (int i = 0; i < debugs.size(); i++) {
        DrawText(hdc, 10, 10 + i * 20, debugs[i]);
    }
    debugs.clear();
}


// Clear everything
void AsczWin::Terminate() {
    delete[] h_drawbuffer;
    hipFree(d_drawbuffer);

    hipFree(d_frmbuffer1);
    hipFree(d_frmbuffer2);
    hipFree(d_frmbuffer3);

    ReleaseDC(hwnd, hdc);
    DestroyWindow(hwnd);
    UnregisterClass(L"Win32App", GetModuleHandle(nullptr));
}


// 📦 Static Window Procedure
LRESULT CALLBACK AsczWin::WindowProc(HWND hwnd, UINT uMsg, WPARAM wParam, LPARAM lParam) {
    AsczWin* self = nullptr;
    if (uMsg == WM_NCCREATE) {
        self = static_cast<AsczWin*>(((CREATESTRUCT*)lParam)->lpCreateParams);
        SetWindowLongPtr(hwnd, GWLP_USERDATA, (LONG_PTR)self);
    } else {
        self = (AsczWin*)GetWindowLongPtr(hwnd, GWLP_USERDATA);
    }

    if (self) {
        switch (uMsg) {
            case WM_DESTROY: PostQuitMessage(0); return 0;

            // Mouse input
            case WM_MOUSEMOVE:
                self->mousePos.x = LOWORD(lParam);
                self->mousePos.y = HIWORD(lParam);
                return 0;

            case WM_LBUTTONDOWN: self->leftMouseDown = true; return 0;
            case WM_LBUTTONUP: self->leftMouseDown = false; return 0;
            case WM_RBUTTONDOWN: self->rightMouseDown = true; return 0;
            case WM_RBUTTONUP: self->rightMouseDown = false; return 0;
            
            // Keyboard input
            case WM_KEYDOWN: self->keys[wParam] = true; return 0;
            case WM_KEYUP: self->keys[wParam] = false; return 0;

            case WM_SETCURSOR:
                SetCursor(LoadCursor(NULL, IDC_ARROW)); 
                return TRUE;

            case WM_MOUSEWHEEL:
                self->scroll = GET_WHEEL_DELTA_WPARAM(wParam) / WHEEL_DELTA;
                return 0;
        }
    }

    return DefWindowProc(hwnd, uMsg, wParam, lParam);
}