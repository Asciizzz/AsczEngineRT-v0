#define STB_IMAGE_IMPLEMENTATION

#include <AsczTxtr.cuh>
#include <stb_image.h>
#include <hip/hip_runtime.h>

AsczTxtr::~AsczTxtr() {
    hipFree(d_tr);
    hipFree(d_tg);
    hipFree(d_tb);
    hipFree(d_ta);

    hipFree(d_tw);
    hipFree(d_th);
    hipFree(d_toff);
}

int AsczTxtr::appendTexture(const char *path) {
    int w, h, n;
    unsigned char *data = stbi_load(path, &w, &h, &n, 4);

    if (data == nullptr) return -1;

    h_tw.push_back(w);
    h_th.push_back(h);
    h_toff.push_back(size);

    for (int y = h - 1; y > -1; y--) {
        for (int x = 0; x < w; x++) {
            int i = (y * w + x) * 4;

            h_tr.push_back(data[i + 0] / 255.0f);
            h_tg.push_back(data[i + 1] / 255.0f);
            h_tb.push_back(data[i + 2] / 255.0f);
            h_ta.push_back(data[i + 3] / 255.0f);
        }
    }

    size += w * h;

    return count++;
}

void AsczTxtr::toDevice() {
    hipMalloc(&d_tr, size * sizeof(float));
    hipMalloc(&d_tg, size * sizeof(float));
    hipMalloc(&d_tb, size * sizeof(float));
    hipMalloc(&d_ta, size * sizeof(float));

    hipMemcpy(d_tr, h_tr.data(), size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tg, h_tg.data(), size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_tb, h_tb.data(), size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_ta, h_ta.data(), size * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc(&d_tw, count * sizeof(int));
    hipMalloc(&d_th, count * sizeof(int));
    hipMalloc(&d_toff, count * sizeof(int));

    hipMemcpy(d_tw, h_tw.data(), count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_th, h_th.data(), count * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_toff, h_toff.data(), count * sizeof(int), hipMemcpyHostToDevice);
}