#include "hip/hip_runtime.h"
#include <AsczMesh.cuh>
#include <ToDevice.cuh>
#include <omp.h>

AsczMesh::~AsczMesh() {
    if (d_vx) { hipFree(d_vx); d_vx = nullptr; }
    if (d_vy) { hipFree(d_vy); d_vy = nullptr; }
    if (d_vz) { hipFree(d_vz); d_vz = nullptr; }
    if (d_nx) { hipFree(d_nx); d_nx = nullptr; }
    if (d_ny) { hipFree(d_ny); d_ny = nullptr; }
    if (d_nz) { hipFree(d_nz); d_nz = nullptr; }
    if (d_tx) { hipFree(d_tx); d_tx = nullptr; }
    if (d_ty) { hipFree(d_ty); d_ty = nullptr; }

    if (d_fv0) { hipFree(d_fv0); d_fv0 = nullptr; }
    if (d_fv1) { hipFree(d_fv1); d_fv1 = nullptr; }
    if (d_fv2) { hipFree(d_fv2); d_fv2 = nullptr; }
    if (d_fn0) { hipFree(d_fn0); d_fn0 = nullptr; }
    if (d_fn1) { hipFree(d_fn1); d_fn1 = nullptr; }
    if (d_fn2) { hipFree(d_fn2); d_fn2 = nullptr; }
    if (d_ft0) { hipFree(d_ft0); d_ft0 = nullptr; }
    if (d_ft1) { hipFree(d_ft1); d_ft1 = nullptr; }
    if (d_ft2) { hipFree(d_ft2); d_ft2 = nullptr; }
    if (d_fm)  { hipFree(d_fm);  d_fm  = nullptr; }
}

void AsczMesh::append(MeshStruct mesh) {
    // #pragma omp parallel for
    // for (int i = 0; i < mesh.SOrF.size(); ++i) {
    //     SOrF.push_back(mesh.SOrF[i] + h_fv0.size());
    // }
    // OrSO.push_back(SOrF.size());
    // oNum++;

    // Update global AABB
    GLB_min_x = fminf(GLB_min_x, mesh.O_AB_min_x);
    GLB_min_y = fminf(GLB_min_y, mesh.O_AB_min_y);
    GLB_min_z = fminf(GLB_min_z, mesh.O_AB_min_z);
    GLB_max_x = fmaxf(GLB_max_x, mesh.O_AB_max_x);
    GLB_max_y = fmaxf(GLB_max_y, mesh.O_AB_max_y);
    GLB_max_z = fmaxf(GLB_max_z, mesh.O_AB_max_z);

    int vPrev = h_vx.size();
    int nPrev = h_nx.size();
    int tPrev = h_tx.size();

    h_vx.insert(h_vx.end(), mesh.vx.begin(), mesh.vx.end());
    h_vy.insert(h_vy.end(), mesh.vy.begin(), mesh.vy.end());
    h_vz.insert(h_vz.end(), mesh.vz.begin(), mesh.vz.end());
    h_nx.insert(h_nx.end(), mesh.nx.begin(), mesh.nx.end());
    h_ny.insert(h_ny.end(), mesh.ny.begin(), mesh.ny.end());
    h_nz.insert(h_nz.end(), mesh.nz.begin(), mesh.nz.end());
    h_tx.insert(h_tx.end(), mesh.tx.begin(), mesh.tx.end());
    h_ty.insert(h_ty.end(), mesh.ty.begin(), mesh.ty.end());

    // Append light sources and offset
    #pragma omp parallel
    for (int i = 0; i < mesh.lsrc.size(); ++i) {
        h_lsrc.push_back(mesh.lsrc[i] + h_fv0.size());
    }

    #pragma omp parallel for
    for (int i = 0; i < mesh.fv0.size(); ++i) {
        // Offset the indices and append
        h_fv0.push_back(mesh.fv0[i] + vPrev);
        h_fv1.push_back(mesh.fv1[i] + vPrev);
        h_fv2.push_back(mesh.fv2[i] + vPrev);

        bool hasN = mesh.fn0[i] != -1;
        int offsetN = nPrev * hasN + !hasN;
        h_fn0.push_back(mesh.fn0[i] + offsetN);
        h_fn1.push_back(mesh.fn1[i] + offsetN);
        h_fn2.push_back(mesh.fn2[i] + offsetN);

        bool hasT = mesh.ft0[i] != -1;
        int offsetT = tPrev * hasT + !hasT;
        h_ft0.push_back(mesh.ft0[i] + offsetT);
        h_ft1.push_back(mesh.ft1[i] + offsetT);
        h_ft2.push_back(mesh.ft2[i] + offsetT);

        h_fm .push_back(mesh.fm[i]);

        int fv0 = h_fv0.back();
        int fv1 = h_fv1.back();
        int fv2 = h_fv2.back();

        AB_min_x.push_back(fminf(h_vx[fv0], fminf(h_vx[fv1], h_vx[fv2])));
        AB_min_y.push_back(fminf(h_vy[fv0], fminf(h_vy[fv1], h_vy[fv2])));
        AB_min_z.push_back(fminf(h_vz[fv0], fminf(h_vz[fv1], h_vz[fv2])));
        AB_max_x.push_back(fmaxf(h_vx[fv0], fmaxf(h_vx[fv1], h_vx[fv2])));
        AB_max_y.push_back(fmaxf(h_vy[fv0], fmaxf(h_vy[fv1], h_vy[fv2])));
        AB_max_z.push_back(fmaxf(h_vz[fv0], fmaxf(h_vz[fv1], h_vz[fv2])));

        AB_cx.push_back((AB_min_x.back() + AB_max_x.back()) * 0.5f);
        AB_cy.push_back((AB_min_y.back() + AB_max_y.back()) * 0.5f);
        AB_cz.push_back((AB_min_z.back() + AB_max_z.back()) * 0.5f);
    }

    vNum = h_vx.size();
    tNum = h_tx.size();
    nNum = h_nx.size();

    fNum = h_fv0.size();
    lNum = h_lsrc.size();
}

void AsczMesh::toDevice() {
    ToDevice::F(h_vx, d_vx); ToDevice::F(h_vy, d_vy); ToDevice::F(h_vz, d_vz);
    ToDevice::F(h_nx, d_nx); ToDevice::F(h_ny, d_ny); ToDevice::F(h_nz, d_nz);
    ToDevice::F(h_tx, d_tx); ToDevice::F(h_ty, d_ty);

    ToDevice::I(h_fv0, d_fv0); ToDevice::I(h_fv1, d_fv1); ToDevice::I(h_fv2, d_fv2);
    ToDevice::I(h_fn0, d_fn0); ToDevice::I(h_fn1, d_fn1); ToDevice::I(h_fn2, d_fn2);
    ToDevice::I(h_ft0, d_ft0); ToDevice::I(h_ft1, d_ft1); ToDevice::I(h_ft2, d_ft2);
    ToDevice::I(h_fm,  d_fm);
    ToDevice::I(h_lsrc,d_lsrc);
}