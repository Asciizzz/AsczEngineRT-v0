#include "hip/hip_runtime.h"
#include <AsczMesh.cuh>
#include <hip/hip_runtime.h>
#include <omp.h>

void AsczMesh::appendMesh(MeshStruct mesh) {
    #pragma omp parallel for
    for (int i = 0; i < mesh.SOrF.size(); ++i) {
        SOrF.push_back(mesh.SOrF[i] + h_fv.size());
    }

    OrSO.push_back(SOrF.size());

    #pragma omp parallel for
    for (int i = 0; i < mesh.geom.size(); ++i) {
        // Offset the indices
        h_fv.push_back(mesh.geom[i].tri.v + h_v.size());
        h_ft.push_back(mesh.geom[i].tri.t + h_t.size());
        h_fn.push_back(mesh.geom[i].tri.n + h_n.size());
        h_fm.push_back(mesh.geom[i].m);
    }

    h_v.insert(h_v.end(), mesh.v.begin(), mesh.v.end());
    h_t.insert(h_t.end(), mesh.t.begin(), mesh.t.end());
    h_n.insert(h_n.end(), mesh.n.begin(), mesh.n.end());

    vNum = h_v.size();
    tNum = h_t.size();
    nNum = h_n.size();
    fNum = h_fv.size();
}

void AsczMesh::freeDevice() {
    if (d_v) { hipFree(d_v); d_v = nullptr; }
    if (d_t) { hipFree(d_t); d_t = nullptr; }
    if (d_n) { hipFree(d_n); d_n = nullptr; }

    if (d_fv) { hipFree(d_fv); d_fv = nullptr; }
    if (d_ft) { hipFree(d_ft); d_ft = nullptr; }
    if (d_fn) { hipFree(d_fn); d_fn = nullptr; }
    if (d_fm) { hipFree(d_fm); d_fm = nullptr; }
}

void AsczMesh::computeData() {
    h_ABmin.resize(fNum);
    h_ABmax.resize(fNum);
    h_ABcen.resize(fNum);

    #pragma omp parallel for
    for (int i = 0; i < fNum; ++i) {
        Flt3 ABmin = Flt3(INFINITY);
        Flt3 ABmax = Flt3(-INFINITY);

        for (int j = 0; j < 3; ++j) {
            Flt3 v = h_v[h_fv[i][j]];

            ABmin.x = fminf(ABmin.x, v.x);
            ABmin.y = fminf(ABmin.y, v.y);
            ABmin.z = fminf(ABmin.z, v.z);

            ABmax.x = fmaxf(ABmax.x, v.x);
            ABmax.y = fmaxf(ABmax.y, v.y);
            ABmax.z = fmaxf(ABmax.z, v.z);
        }

        h_ABmin[i] = ABmin;
        h_ABmax[i] = ABmax;
        h_ABcen[i] = (ABmin + ABmax) * 0.5f;
    }
}

void AsczMesh::toDevice() {
    freeDevice();
    computeData();

    // -------------------------------------- //    

    hipMalloc(&d_v, vNum * sizeof(Flt3));
    hipMalloc(&d_t, tNum * sizeof(Flt2));
    hipMalloc(&d_n, nNum * sizeof(Flt3));

    hipMalloc(&d_fv, fNum * sizeof(Int3));
    hipMalloc(&d_ft, fNum * sizeof(Int3));
    hipMalloc(&d_fn, fNum * sizeof(Int3));
    hipMalloc(&d_fm, fNum * sizeof(int));

    hipMalloc(&d_ABmin, fNum * sizeof(Flt3));
    hipMalloc(&d_ABmax, fNum * sizeof(Flt3));
    hipMalloc(&d_ABcen, fNum * sizeof(Flt3));

    // -------------------------------------- //

    hipMemcpy(d_v, h_v.data(), vNum * sizeof(Flt3), hipMemcpyHostToDevice);
    hipMemcpy(d_t, h_t.data(), tNum * sizeof(Flt2), hipMemcpyHostToDevice);
    hipMemcpy(d_n, h_n.data(), nNum * sizeof(Flt3), hipMemcpyHostToDevice);

    hipMemcpy(d_fv, h_fv.data(), fNum * sizeof(Int3), hipMemcpyHostToDevice);
    hipMemcpy(d_ft, h_ft.data(), fNum * sizeof(Int3), hipMemcpyHostToDevice);
    hipMemcpy(d_fn, h_fn.data(), fNum * sizeof(Int3), hipMemcpyHostToDevice);
    hipMemcpy(d_fm, h_fm.data(), fNum * sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(d_ABmin, h_ABmin.data(), fNum * sizeof(Flt3), hipMemcpyHostToDevice);
    hipMemcpy(d_ABmax, h_ABmax.data(), fNum * sizeof(Flt3), hipMemcpyHostToDevice);    
    hipMemcpy(d_ABcen, h_ABcen.data(), fNum * sizeof(Flt3), hipMemcpyHostToDevice);
}