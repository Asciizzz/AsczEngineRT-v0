#include <AsczMesh.cuh>
#include <hip/hip_runtime.h>
#include <omp.h>

int AsczMesh::appendVrtx(Flt3 v) {
    h_v.push_back(v); return vNum++;
}
int AsczMesh::appendTxtr(Flt2 t) {
    h_t.push_back(t); return tNum++;
}
int AsczMesh::appendNrml(Flt3 n) {
    h_n.push_back(n); return nNum++;
}



void AsczMesh::appendMesh(MeshStruct mesh) {
    #pragma omp parallel for
    for (int i = 0; i < mesh.SOrF.size(); ++i) {
        SOrF.push_back(mesh.SOrF[i] + h_geom.size());
    }

    OrSO.push_back(SOrF.size());

    #pragma omp parallel for
    for (int i = 0; i < mesh.geom.size(); ++i) {
        // Offset the indices

        AzGeom &g = mesh.geom[i];
        if (g.type == AzGeom::TRIANGLE) {
            g.tri.v += h_v.size();
            g.tri.t += h_t.size();
            g.tri.n += h_n.size();
        }
        else if (g.type == AzGeom::SPHERE) {
            g.sph.c += h_v.size();
        }

        h_geom.push_back(g);
    }

    h_v.insert(h_v.end(), mesh.v.begin(), mesh.v.end());
    h_t.insert(h_t.end(), mesh.t.begin(), mesh.t.end());
    h_n.insert(h_n.end(), mesh.n.begin(), mesh.n.end());

    vNum = h_v.size();
    tNum = h_t.size();
    nNum = h_n.size();

    gNum = h_geom.size();
}

void AsczMesh::freeDevice() {
    if (d_v) { hipFree(d_v); d_v = nullptr; }
    if (d_t) { hipFree(d_t); d_t = nullptr; }
    if (d_n) { hipFree(d_n); d_n = nullptr; }

    if (d_geom) { hipFree(d_geom); d_geom = nullptr; }
}

void AsczMesh::toDevice() {
    freeDevice();

    hipMalloc(&d_v, vNum * sizeof(Flt3));
    hipMalloc(&d_t, tNum * sizeof(Flt2));
    hipMalloc(&d_n, nNum * sizeof(Flt3));
    hipMalloc(&d_geom, gNum * sizeof(AzGeom));

    hipMemcpy(d_v, h_v.data(), vNum * sizeof(Flt3), hipMemcpyHostToDevice);
    hipMemcpy(d_t, h_t.data(), tNum * sizeof(Flt2), hipMemcpyHostToDevice);
    hipMemcpy(d_n, h_n.data(), nNum * sizeof(Flt3), hipMemcpyHostToDevice);
    hipMemcpy(d_geom, h_geom.data(), gNum * sizeof(AzGeom), hipMemcpyHostToDevice);
}