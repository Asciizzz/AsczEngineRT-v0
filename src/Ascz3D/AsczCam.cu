#include "hip/hip_runtime.h"
#include <AsczCam.cuh>

#include <string>
#include <iostream>

// ================================ Ray ================================

Ray::Ray() {};
Ray::Ray(Flt3 o, Flt3 d, float w, float Ior, int ignore) :
    o(o), d(d), invd(1.0f / d), w(w), Ior(Ior), ignore(ignore) {}

// Reflection + refraction
Flt3 Ray::reflect(const Flt3 &n) {
    return d - n * (2.0f * (d * n));
}

Flt3 Ray::refract(const Flt3 &n, float Ior2) {
    float Ior1 = Ior;
    float cosI = -n * d;
    float cosT2 = 1.0f - Ior1 * Ior1 * (1.0f - cosI * cosI) / (Ior2 * Ior2);

    if (cosT2 < 0.0f) return Flt3(); // Total internal reflection

    return d * Ior1 / Ior2 + n * (Ior1 * cosI / Ior2 - sqrt(cosT2));
}

Flt3 Ray::reflect(const Flt3 &d, const Flt3 &n) {
    return d - n * (2.0f * (d * n));
}

// ================================ Camera ================================

void AsczCam::restrictRot() {
    if (rot.x <= -M_PI_2) rot.x = -M_PI_2 + 0.001;
    else if (rot.x >= M_PI_2) rot.x = M_PI_2 - 0.001;

    if (rot.y > M_2_PI) rot.y -= M_2_PI;
    else if (rot.y < 0) rot.y += M_2_PI;
}

void AsczCam::updateView() {
    forward.x = sin(rot.y) * cos(rot.x);
    forward.y = sin(rot.x);
    forward.z = cos(rot.y) * cos(rot.x);
    forward.norm();

    right = Flt3(0, 1, 0) ^ forward;
    right.norm();

    up = forward ^ right;
    up.norm();
}


Flt2 AsczCam::getScrnNDC(float x, float y, float width, float height) const {
    // Note: w/2 and h/2 are used to center the screen space coordinates
    return Flt2((2 * x - width) / width, (height - 2 * y) / height);
}

Ray AsczCam::castRay(float x, float y, float width, float height, float dx, float dy) const {
    // Step 1: Convert screen space coordinates to NDC
    Flt2 ndc = getScrnNDC(x, y, width, height);

    // Step 2: Calculate the direction vector for the ray
    // tan(fov / 2) scales the direction based on the field of view.
    // Aspect ratio adjusts the direction for non-square screens (i.e., when width != height).
    float tanFov = tan(fov / 2);
    Flt3 rayDir = forward + right * ndc.x * tanFov * width / height + up * ndc.y * tanFov;
    rayDir += right * dx + up * dy;

    // Step 3: Normalize the direction vector
    rayDir.norm();

    // Step 4: Create and return the ray from the camera's position and the calculated direction
    return Ray(pos, rayDir);
}


void AsczCam::update() {
    restrictRot();
    updateView();
}