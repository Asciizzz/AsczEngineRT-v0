#include "hip/hip_runtime.h"
#include <RayTrace.cuh>

#include <hiprand/hiprand_kernel.h>

struct RayHit {
    int idx = -1;
    float t = 1e9;
    float u = 0;
    float v = 0;
};

__device__ Flt4 getTextureColor(
    float u, float v,
    float *tr, float *tg, float *tb, float *ta,
    int *tw, int *th, int *toff, int AlbMap
) {
    u -= floor(u);
    v -= floor(v);

    int w = tw[AlbMap];
    int h = th[AlbMap];
    int off = toff[AlbMap];

    int tx = (int)(u * w);
    int ty = (int)(v * h);

    int t = off + ty * w + tx;
    return Flt4(tr[t], tg[t], tb[t], ta[t]);
}

__device__ Flt3 ASESFilm(const Flt3 &P) {
    const float a = 2.51f;
    const float b = 0.03f;
    const float c = 2.43f;
    const float d = 0.59f;
    const float e = 0.14f;

    Flt3 y = Flt3(
        (P.x * (a * P.x + b)) / (P.x * (c * P.x + d) + e),
        (P.y * (a * P.y + b)) / (P.y * (c * P.y + d) + e),
        (P.z * (a * P.z + b)) / (P.z * (c * P.z + d) + e)
    ).clamp(0.0f, 1.0f);

    return y;
}

__device__ Flt3 randomHemisphereSample(hiprandState *rnd, const Flt3 &n) {
    float r1 = hiprand_uniform(rnd);  // Random number [0,1]
    float r2 = hiprand_uniform(rnd);

    float theta = acos(sqrt(1.0f - r1));  // Importance sampling (cosine-weighted)
    float phi = 2.0f * M_PI * r2;         // Uniform azimuthal angle

    // Convert to Cartesian coordinates
    float x = sin(theta) * cos(phi);
    float y = sin(theta) * sin(phi);
    float z = cos(theta);

    // Construct a coordinate system
    Flt3 tangent, bitangent;

    tangent = n.x > 0.9f || n.x < -0.9f ? Flt3(0, 1, 0) : Flt3(1, 0, 0);
    tangent = (tangent ^ n).norm();
    bitangent = n ^ tangent;

    // Transform to world space
    return tangent * x + bitangent * y + n * z;
}



__global__ void raytraceKernel(
    AsczCam camera, unsigned int *frmbuffer, int frmW, int frmH, // In-out
    // Primitive data
    float *vx, float *vy, float *vz, float *tx, float *ty, float *nx, float *ny, float *nz,
    // Materials
    AzMtl *mats,
    // Textures
    float *tr, float *tg, float *tb, float *ta, int *tw, int *th, int *toff,
    // Geometry data
    int *fv0, int *fv1, int *fv2, int *ft0, int *ft1, int *ft2, int *fn0, int *fn1, int *fn2, int *fm,
    // Light data
    int *lSrc, int lNum, 
    // BVH data
    float *mi_x, float *mi_y, float *mi_z, float *mx_x, float *mx_y, float *mx_z, int *cl, int *cr, int *ll, int *lr, int *gIdx,
    // Additional Debug Data
    bool falseAmbient
) {
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (tIdx >= frmW * frmH) return;

    int x = tIdx % frmW, y = tIdx / frmW;
    Ray primaryRay = camera.castRay(x, y, frmW, frmH);

    const int MAX_RAYS = 8;
    const int MAX_NODES = 64;

    Ray rstack[MAX_RAYS] = { primaryRay };
    int rs_top = 1;

    int nstack[MAX_NODES];
    int ns_top = 0;

    Flt3 resultColr;
    while (rs_top > 0) {
        // Copy before pop since there's high chance of overwriting
        Ray ray = rstack[--rs_top];
        RayHit rhit;

        ns_top = 0;
        nstack[ns_top++] = 0;

        while (ns_top > 0) {
            int nidx = nstack[--ns_top];

            float t1n = (mi_x[nidx] - ray.o.x) * ray.invd.x;
            float t2n = (mx_x[nidx] - ray.o.x) * ray.invd.x;
            float t3n = (mi_y[nidx] - ray.o.y) * ray.invd.y;
            float t4n = (mx_y[nidx] - ray.o.y) * ray.invd.y;
            float t5n = (mi_z[nidx] - ray.o.z) * ray.invd.z;
            float t6n = (mx_z[nidx] - ray.o.z) * ray.invd.z;

            float tminn = fmaxf(fmaxf(fminf(t1n, t2n), fminf(t3n, t4n)), fminf(t5n, t6n));
            float tmaxn = fminf(fminf(fmaxf(t1n, t2n), fmaxf(t3n, t4n)), fmaxf(t5n, t6n));

            bool nOut = ray.o.x < mi_x[nidx] | ray.o.x > mx_x[nidx] |
                        ray.o.y < mi_y[nidx] | ray.o.y > mx_y[nidx] |
                        ray.o.z < mi_z[nidx] | ray.o.z > mx_z[nidx];
            float nDist = ((tmaxn < tminn | tminn < 0) ? -1 : tminn) * nOut;

            if (nDist < 0 | nDist > rhit.t) continue;

            if (cl[nidx] > -1) {
                float t1l = (mi_x[cl[nidx]] - ray.o.x) * ray.invd.x;
                float t2l = (mx_x[cl[nidx]] - ray.o.x) * ray.invd.x;
                float t3l = (mi_y[cl[nidx]] - ray.o.y) * ray.invd.y;
                float t4l = (mx_y[cl[nidx]] - ray.o.y) * ray.invd.y;
                float t5l = (mi_z[cl[nidx]] - ray.o.z) * ray.invd.z;
                float t6l = (mx_z[cl[nidx]] - ray.o.z) * ray.invd.z;

                float tminl = fmaxf(fmaxf(fminf(t1l, t2l), fminf(t3l, t4l)), fminf(t5l, t6l));
                float tmaxl = fminf(fminf(fmaxf(t1l, t2l), fmaxf(t3l, t4l)), fmaxf(t5l, t6l));

                bool lOut = ray.o.x < mi_x[cl[nidx]] | ray.o.x > mx_x[cl[nidx]] |
                            ray.o.y < mi_y[cl[nidx]] | ray.o.y > mx_y[cl[nidx]] |
                            ray.o.z < mi_z[cl[nidx]] | ray.o.z > mx_z[cl[nidx]];
                float ldist = ((tmaxl < tminl | tminl < 0) ? -1 : tminl) * lOut;


                float t1r = (mi_x[cr[nidx]] - ray.o.x) * ray.invd.x;
                float t2r = (mx_x[cr[nidx]] - ray.o.x) * ray.invd.x;
                float t3r = (mi_y[cr[nidx]] - ray.o.y) * ray.invd.y;
                float t4r = (mx_y[cr[nidx]] - ray.o.y) * ray.invd.y;
                float t5r = (mi_z[cr[nidx]] - ray.o.z) * ray.invd.z;
                float t6r = (mx_z[cr[nidx]] - ray.o.z) * ray.invd.z;

                float tminr = fmaxf(fmaxf(fminf(t1r, t2r), fminf(t3r, t4r)), fminf(t5r, t6r));
                float tmaxr = fminf(fminf(fmaxf(t1r, t2r), fmaxf(t3r, t4r)), fmaxf(t5r, t6r));

                bool rOut = ray.o.x < mi_x[cr[nidx]] | ray.o.x > mx_x[cr[nidx]] |
                            ray.o.y < mi_y[cr[nidx]] | ray.o.y > mx_y[cr[nidx]] |
                            ray.o.z < mi_z[cr[nidx]] | ray.o.z > mx_z[cr[nidx]];
                float rdist = ((tmaxr < tminr | tminr < 0) ? -1 : tminr) * rOut;


                bool lcloser = ldist < rdist;

                nstack[ns_top] = cr[nidx] * lcloser + cl[nidx] * !lcloser;
                ns_top += (rdist >= 0) * lcloser + (ldist >= 0) * !lcloser;

                nstack[ns_top] = cl[nidx] * lcloser + cr[nidx] * !lcloser;
                ns_top += (ldist >= 0) * lcloser + (rdist >= 0) * !lcloser;

                continue;
            }

            for (int i = ll[nidx]; i < lr[nidx]; ++i) {
                int gi = gIdx[i];

                bool hit = gi != rhit.idx;

                float e1x = vx[fv1[gi]] - vx[fv0[gi]];
                float e1y = vy[fv1[gi]] - vy[fv0[gi]];
                float e1z = vz[fv1[gi]] - vz[fv0[gi]];

                float e2x = vx[fv2[gi]] - vx[fv0[gi]];
                float e2y = vy[fv2[gi]] - vy[fv0[gi]];
                float e2z = vz[fv2[gi]] - vz[fv0[gi]];

                float hx = ray.d.y * e2z - ray.d.z * e2y;
                float hy = ray.d.z * e2x - ray.d.x * e2z;
                float hz = ray.d.x * e2y - ray.d.y * e2x;

                float a = e1x * hx + e1y * hy + e1z * hz;

                hit &= a != 0.0f;
                a = a == 0.0f ? 1.0f : a;

                float f = 1.0f / a;

                float sx = ray.o.x - vx[fv0[gi]];
                float sy = ray.o.y - vy[fv0[gi]];
                float sz = ray.o.z - vz[fv0[gi]];

                float u = f * (sx * hx + sy * hy + sz * hz);

                hit &= u >= 0.0f & u <= 1.0f;

                float qx = sy * e1z - sz * e1y;
                float qy = sz * e1x - sx * e1z;
                float qz = sx * e1y - sy * e1x;

                float v = f * (ray.d.x * qx + ray.d.y * qy + ray.d.z * qz);

                hit &= v >= 0.0f & u + v <= 1.0f;

                float t = f * (e2x * qx + e2y * qy + e2z * qz);

                hit &= t > 0.0f & t < rhit.t;

                rhit.t = t * hit + rhit.t * !hit;
                rhit.u = u * hit + rhit.u * !hit;
                rhit.v = v * hit + rhit.v * !hit;
                rhit.idx = gi * hit + rhit.idx * !hit;
            }
        }

        int hIdx = rhit.idx;
        if (hIdx == -1) continue;

        // Get the face data
        const AzMtl &hm = mats[fm[hIdx]];

        float rhitw = 1 - rhit.u - rhit.v;

        Flt3 vrtx = ray.o + ray.d * rhit.t;

        Flt3 nrml = {
            nx[fn0[hIdx]] * rhitw + nx[fn1[hIdx]] * rhit.u + nx[fn2[hIdx]] * rhit.v,
            ny[fn0[hIdx]] * rhitw + ny[fn1[hIdx]] * rhit.u + ny[fn2[hIdx]] * rhit.v,
            nz[fn0[hIdx]] * rhitw + nz[fn1[hIdx]] * rhit.u + nz[fn2[hIdx]] * rhit.v
        };

        Flt3 alb;
        if (hm.AlbMap > -1) {
            Int3 tt = Int3(ft0[hIdx], ft1[hIdx], ft2[hIdx]);
            float tu = tx[tt.x] * rhitw + tx[tt.y] * rhit.u + tx[tt.z] * rhit.v;
            float tv = ty[tt.x] * rhitw + ty[tt.y] * rhit.u + ty[tt.z] * rhit.v;

            Flt4 txColr = getTextureColor(tu, tv, tr, tg, tb, ta, tw, th, toff, hm.AlbMap);
            alb = txColr.f3();
        } else {
            alb = hm.Alb;
        }

        // Lighting and shading
        float NdotL = falseAmbient ? nrml * ray.d : 0.0f;
        Flt3 finalColr = alb * 0.02f * NdotL * NdotL;

        if (!hm.Ems.isZero()) {
            resultColr += hm.Ems * ray.w;
            continue;
        }

        // Direct lighting
        for (int l = 0; l < lNum; ++l) {
            // Get material and geometry data of light
            int lIdx = lSrc[l];
            const AzMtl &lMat = mats[fm[lIdx]];

            // Get position based on the geometry type
            float lpx = (vx[fv0[lIdx]] + vx[fv1[lIdx]] + vx[fv2[lIdx]]) / 3.0f;
            float lpy = (vy[fv0[lIdx]] + vy[fv1[lIdx]] + vy[fv2[lIdx]]) / 3.0f;
            float lpz = (vz[fv0[lIdx]] + vz[fv1[lIdx]] + vz[fv2[lIdx]]) / 3.0f;

            // Flt3 lDir = vrtx - Flt3(lpx, lpy, lpz);
            float ldx = vrtx.x - lpx;
            float ldy = vrtx.y - lpy;
            float ldz = vrtx.z - lpz;

            float ldst = sqrt(ldx * ldx + ldy * ldy + ldz * ldz);
            if (ldst < 0.01f) continue;

            ldx /= ldst;
            ldy /= ldst;
            ldz /= ldst;

            float linvx = 1.0f / ldx;
            float linvy = 1.0f / ldy;
            float linvz = 1.0f / ldz;

            // Reset the stack
            ns_top = 0;
            nstack[ns_top++] = 0;

            bool shadow = false;
            while (ns_top > 0) {
                int nidx = nstack[--ns_top];

                float t1n = (mi_x[nidx] - lpx) * linvx;
                float t2n = (mx_x[nidx] - lpx) * linvx;
                float t3n = (mi_y[nidx] - lpy) * linvy;
                float t4n = (mx_y[nidx] - lpy) * linvy;
                float t5n = (mi_z[nidx] - lpz) * linvz;
                float t6n = (mx_z[nidx] - lpz) * linvz;

                float tminn = fmaxf(fmaxf(fminf(t1n, t2n), fminf(t3n, t4n)), fminf(t5n, t6n));
                float tmaxn = fminf(fminf(fmaxf(t1n, t2n), fmaxf(t3n, t4n)), fmaxf(t5n, t6n));
                
                bool nOut = lpx < mi_x[nidx] | lpx > mx_x[nidx] |
                            lpy < mi_y[nidx] | lpy > mx_y[nidx] |
                            lpz < mi_z[nidx] | lpz > mx_z[nidx];
                float nDist = ((tmaxn < tminn | tminn < 0) ? -1 : tminn) * nOut;

                if (nDist < 0 | nDist > ldst) continue;

                if (cl[nidx] > -1) {
                    float t1l = (mi_x[cl[nidx]] - lpx) * linvx;
                    float t2l = (mx_x[cl[nidx]] - lpx) * linvx;
                    float t3l = (mi_y[cl[nidx]] - lpy) * linvy;
                    float t4l = (mx_y[cl[nidx]] - lpy) * linvy;
                    float t5l = (mi_z[cl[nidx]] - lpz) * linvz;
                    float t6l = (mx_z[cl[nidx]] - lpz) * linvz;

                    float tminl = fmaxf(fmaxf(fminf(t1l, t2l), fminf(t3l, t4l)), fminf(t5l, t6l));
                    float tmaxl = fminf(fminf(fmaxf(t1l, t2l), fmaxf(t3l, t4l)), fmaxf(t5l, t6l));

                    bool lOut = lpx < mi_x[cl[nidx]] | lpx > mx_x[cl[nidx]] |
                                lpy < mi_y[cl[nidx]] | lpy > mx_y[cl[nidx]] |
                                lpz < mi_z[cl[nidx]] | lpz > mx_z[cl[nidx]];
                    float ldist = ((tmaxl < tminl | tminl < 0) ? -1 : tminl) * lOut;


                    float t1r = (mi_x[cr[nidx]] - lpx) * linvx;
                    float t2r = (mx_x[cr[nidx]] - lpx) * linvx;
                    float t3r = (mi_y[cr[nidx]] - lpy) * linvy;
                    float t4r = (mx_y[cr[nidx]] - lpy) * linvy;
                    float t5r = (mi_z[cr[nidx]] - lpz) * linvz;
                    float t6r = (mx_z[cr[nidx]] - lpz) * linvz;

                    float tminr = fmaxf(fmaxf(fminf(t1r, t2r), fminf(t3r, t4r)), fminf(t5r, t6r));
                    float tmaxr = fminf(fminf(fmaxf(t1r, t2r), fmaxf(t3r, t4r)), fmaxf(t5r, t6r));

                    bool rOut = lpx < mi_x[cr[nidx]] | lpx > mx_x[cr[nidx]] |
                                lpy < mi_y[cr[nidx]] | lpy > mx_y[cr[nidx]] |
                                lpz < mi_z[cr[nidx]] | lpz > mx_z[cr[nidx]];
                    float rdist = ((tmaxr < tminr | tminr < 0) ? -1 : tminr) * rOut;


                    nstack[ns_top] = cl[nidx];
                    ns_top += (ldist >= 0);

                    nstack[ns_top] = cr[nidx];
                    ns_top += (rdist >= 0);
    
                    continue;
                }
    
                for (int i = ll[nidx]; i < lr[nidx]; ++i) {
                    int gi = gIdx[i];

                    bool hit = gi != hIdx & gi != lIdx;

                    float e1x = vx[fv1[gi]] - vx[fv0[gi]];
                    float e1y = vy[fv1[gi]] - vy[fv0[gi]];
                    float e1z = vz[fv1[gi]] - vz[fv0[gi]];

                    float e2x = vx[fv2[gi]] - vx[fv0[gi]];
                    float e2y = vy[fv2[gi]] - vy[fv0[gi]];
                    float e2z = vz[fv2[gi]] - vz[fv0[gi]];

                    float hx = ldy * e2z - ldz * e2y;
                    float hy = ldz * e2x - ldx * e2z;
                    float hz = ldx * e2y - ldy * e2x;

                    float a = e1x * hx + e1y * hy + e1z * hz;

                    hit &= a != 0;
                    a = a == 0 ? 1 : a;

                    float f = 1.0f / a;

                    float sx = lpx - vx[fv0[gi]];
                    float sy = lpy - vy[fv0[gi]];
                    float sz = lpz - vz[fv0[gi]];

                    float u = f * (sx * hx + sy * hy + sz * hz);

                    hit &= u >= 0 & u <= 1;

                    float qx = sy * e1z - sz * e1y;
                    float qy = sz * e1x - sx * e1z;
                    float qz = sx * e1y - sy * e1x;

                    float v = f * (ldx * qx + ldy * qy + ldz * qz);

                    hit &= v >= 0 & u + v <= 1;

                    float t = f * (e2x * qx + e2y * qy + e2z * qz);

                    hit &= t > 0 & t < ldst;

                    shadow |= hit;
                    ns_top *= !shadow;
                }
            }

            float NdotL = nrml.x * ldx + nrml.y * ldy + nrml.z * ldz;
            NdotL *= (NdotL < 0) * -1;
            Flt3 diff = alb * NdotL;

            finalColr += shadow ? 0 : lMat.Ems & diff;
        }

        // ======== Additional rays ========

        // Transparent
        if (hm.Tr > 0.0f & rs_top + 2 < MAX_RAYS) {
            float wLeft = ray.w * hm.Tr;
            ray.w *= (1 - hm.Tr);

            Flt3 rO = vrtx + ray.d * EPSILON_1;
            rstack[rs_top++] = Ray(rO, ray.d, wLeft, hm.Ior, hIdx);
        }

        resultColr += finalColr * ray.w;
    }

    // Tone mapping
    resultColr = ASESFilm(resultColr);

    float _gamma = 1.0f / 2.2f;
    resultColr = resultColr.pow(_gamma);

    int r = (int)(resultColr.x * 255);
    int g = (int)(resultColr.y * 255);
    int b = (int)(resultColr.z * 255);

    frmbuffer[tIdx] = (r << 16) | (g << 8) | b;
}
