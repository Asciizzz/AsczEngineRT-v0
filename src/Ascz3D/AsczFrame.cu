#include "hip/hip_runtime.h"
#include <AsczFrame.cuh>
#include <AzDevMath.cuh>
#include <iostream>
#define _GAMMA 1.0f/2.2f

__global__ void initRandState(hiprandState *state, int width, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) hiprand_init(1234, idx, 0, &state[idx]);
}


AsczFrame::AsczFrame(int w, int h) : width(w), height(h), size(w * h) {
    blockCount = (size + blockSize - 1) / blockSize;
    // Buffers initialization

    hipMalloc(&d_fx0, size * sizeof(float));
    hipMalloc(&d_fy0, size * sizeof(float));
    hipMalloc(&d_fz0, size * sizeof(float));

    hipMalloc(&d_fx1, size * sizeof(float));
    hipMalloc(&d_fy1, size * sizeof(float));
    hipMalloc(&d_fz1, size * sizeof(float));

    hipMalloc(&d_fx2, size * sizeof(float));
    hipMalloc(&d_fy2, size * sizeof(float));
    hipMalloc(&d_fz2, size * sizeof(float));

    hipMalloc(&d_draw, size * sizeof(unsigned int));
    h_draw = new unsigned int[size];

    hipMalloc(&d_rand, size * sizeof(hiprandState));
    initRandState<<<blockCount, blockSize>>>(d_rand, width, size);

    hipMalloc(&d_depth, size * sizeof(float));
    h_depth = new float[size];

    hipMalloc(&d_mat, size * sizeof(int));
    h_mat = new int[size];
}

AsczFrame::~AsczFrame() {
    hipFree(d_fx0); hipFree(d_fy0); hipFree(d_fz0);
    hipFree(d_fx1); hipFree(d_fy1); hipFree(d_fz1);
    hipFree(d_fx2); hipFree(d_fy2); hipFree(d_fz2);
    
    hipFree(d_draw); delete[] h_draw;

    hipFree(d_rand);

    hipFree(d_depth); delete[] h_depth;
    hipFree(d_mat); delete[] h_mat;
}



__global__ void toDrawBuffer(float *fx, float *fy, float *fz, unsigned int *draw, int width, int size, bool toneMap, bool crosshair=false) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    int x = i % width;
    int y = i / width;
    int height = size / width;
    int h_width = width / 2;
    int h_height = height / 2;
    int cr_thick = 1;
    int cr_length = 5;

    float r = AzDevMath::ACESFilm(powf(fx[i], _GAMMA)) * toneMap + fx[i] * !toneMap;
    float g = AzDevMath::ACESFilm(powf(fy[i], _GAMMA)) * toneMap + fy[i] * !toneMap;
    float b = AzDevMath::ACESFilm(powf(fz[i], _GAMMA)) * toneMap + fz[i] * !toneMap;

    // Draw a crosshair
    int x_sub_hw = x - h_width;
    int y_sub_hh = y - h_height;

    bool isCrosshair = crosshair &
        (x_sub_hw >= -cr_thick & x_sub_hw <= cr_thick & y_sub_hh >= -cr_length & y_sub_hh <= cr_length) |
        (y_sub_hh >= -cr_thick & y_sub_hh <= cr_thick & x_sub_hw >= -cr_length & x_sub_hw <= cr_length);

    int ir = r * 255;
    int ig = g * 255;
    int ib = b * 255;

    // This to ensure the cross hair will never share the same color as the background
    int truer = ir * !isCrosshair + ((ir + 128) % 255) * isCrosshair;
    int trueg = ig * !isCrosshair + ((ig + 128) % 255) * isCrosshair;
    int trueb = ib * !isCrosshair + ((ib + 128) % 255) * isCrosshair;

    draw[i] = (truer << 16) | (trueg << 8) | trueb;
}

void AsczFrame::toDraw0(bool toneMap, bool crosshair) {
    toDrawBuffer<<<blockCount, blockSize>>>(d_fx0, d_fy0, d_fz0, d_draw, width, size, toneMap, crosshair);
    hipMemcpy(h_draw, d_draw, size * sizeof(unsigned int), hipMemcpyDeviceToHost);
}

void AsczFrame::toDraw1(bool toneMap, bool crosshair) {
    toDrawBuffer<<<blockCount, blockSize>>>(d_fx1, d_fy1, d_fz1, d_draw, width, size, toneMap, crosshair);
    hipMemcpy(h_draw, d_draw, size * sizeof(unsigned int), hipMemcpyDeviceToHost);
}


__global__ void toDrawAccumulatedBuffer(float *fx, float *fy, float *fz, unsigned int *draw, int width, int size, int acc, bool toneMap) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    float rdiv = fx[i] / acc;
    float gdiv = fy[i] / acc;
    float bdiv = fz[i] / acc;

    float r = AzDevMath::ACESFilm(powf(rdiv, _GAMMA)) * toneMap + rdiv * !toneMap;
    float g = AzDevMath::ACESFilm(powf(gdiv, _GAMMA)) * toneMap + gdiv * !toneMap;
    float b = AzDevMath::ACESFilm(powf(bdiv, _GAMMA)) * toneMap + bdiv * !toneMap;

    draw[i] = (int(r * 255) << 16) | (int(g * 255) << 8) | int(b * 255);
}

void AsczFrame::toDraw2(bool toneMap) {
    toDrawAccumulatedBuffer<<<blockCount, blockSize>>>(d_fx2, d_fy2, d_fz2, d_draw, width, size, f_acc, toneMap);
    hipMemcpy(h_draw, d_draw, size * sizeof(unsigned int), hipMemcpyDeviceToHost);
}



__global__ void addKernel(float *fx, float *fy, float *fz, float *fx2, float *fy2, float *fz2, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size) return;

    fx[i] += fx2[i];
    fy[i] += fy2[i];
    fz[i] += fz2[i];
}

void AsczFrame::add0() {
    addKernel<<<blockCount, blockSize>>>(d_fx2, d_fy2, d_fz2, d_fx0, d_fy0, d_fz0, size);
    f_acc++;
}
void AsczFrame::add1() {
    addKernel<<<blockCount, blockSize>>>(d_fx2, d_fy2, d_fz2, d_fx1, d_fy1, d_fz1, size);
    f_acc++;
}


__global__ void resetKernel(float *fx, float *fy, float *fz, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        fx[i] = 0.0f; fy[i] = 0.0f; fz[i] = 0.0f;
    }
}

void AsczFrame::reset2() {
    resetKernel<<<blockCount, blockSize>>>(d_fx2, d_fy2, d_fz2, size);
    f_acc = 0;
}


__global__ void bilateralFilter(
    float *fx0, float *fy0, float *fz0,  // Input frame
    float *fx1, float *fy1, float *fz1,  // Output frame
    float *depth,                         // Depth buffer
    int width, int height,                // Image dimensions
    float sigmaSpatial, float sigmaRange, // Filter parameters
    int radius                            // Filter radius
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx % width, y = idx / width;

    if (x >= width || y >= height) return;

    float centerDepth = depth[idx];
    float centerR = fx0[idx];
    float centerG = fy0[idx];
    float centerB = fz0[idx];

    float sumR = 0.0f, sumG = 0.0f, sumB = 0.0f;
    float totalWeight = 0.0f;

    for (int dy = -radius; dy <= radius; dy++) {
        for (int dx = -radius; dx <= radius; dx++) {
            int nx = x + dx, ny = y + dy;
            if (nx < 0 || nx >= width || ny < 0 || ny >= height) continue;

            int nIdx = ny * width + nx;
            float neighborDepth = depth[nIdx];
            float neighborR = fx0[nIdx];
            float neighborG = fy0[nIdx];
            float neighborB = fz0[nIdx];

            // Compute spatial weight (Gaussian blur based on distance)
            float spatialWeight = expf(-(dx * dx + dy * dy) / (2.0f * sigmaSpatial * sigmaSpatial));

            // Compute range weight (Gaussian depth similarity)
            float rangeWeight = expf(-((neighborDepth - centerDepth) * (neighborDepth - centerDepth)) / (2.0f * sigmaRange * sigmaRange));

            // Final weight
            float weight = spatialWeight * rangeWeight;
            totalWeight += weight;

            // Apply weighted sum
            sumR += neighborR * weight;
            sumG += neighborG * weight;
            sumB += neighborB * weight;
        }
    }

    // Normalize and write back
    fx1[idx] = sumR / totalWeight;
    fy1[idx] = sumG / totalWeight;
    fz1[idx] = sumB / totalWeight;
}

void AsczFrame::biliFilter0() {
    bilateralFilter<<<blockCount, blockSize>>>(
        d_fx0, d_fy0, d_fz0,
        d_fx1, d_fy1, d_fz1,
        d_depth, width, height,
        1.0f, 0.1f, 3
    );
}

void AsczFrame::biliFilter1() {
    bilateralFilter<<<blockCount, blockSize>>>(
        d_fx1, d_fy1, d_fz1,
        d_fx0, d_fy0, d_fz0,
        d_depth, width, height,
        1.0f, 0.1f, 3
    );
}