#include <AsczMat.cuh>
#include <hip/hip_runtime.h>

AsczMat::~AsczMat() {
    if (d_mtls) {
        hipFree(d_mtls);
        d_mtls = nullptr;
    }
}

int AsczMat::append(AzMtl mtl, std::wstring name, std::wstring path) {
    h_mtls.push_back(mtl);
    names.push_back(name);
    paths.push_back(path);
    return mtlsNum++;
}

void AsczMat::toDevice() {
    if (mtlsNum) {
        hipMalloc(&d_mtls, mtlsNum * sizeof(AzMtl));
        hipMemcpy(d_mtls, h_mtls.data(), mtlsNum * sizeof(AzMtl), hipMemcpyHostToDevice);
    }
}