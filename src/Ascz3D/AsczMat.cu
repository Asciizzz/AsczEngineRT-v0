#include <AsczMat.cuh>
#include <hip/hip_runtime.h>

int AsczMat::appendMaterial(AzMtl mtl) {
    h_mtls.push_back(mtl);
    return mtlsNum++;
}

void AsczMat::freeDevice() {
    if (d_mtls) {
        hipFree(d_mtls);
        d_mtls = nullptr;
    }
}

void AsczMat::toDevice() {
    freeDevice();
    if (mtlsNum) {
        hipMalloc(&d_mtls, mtlsNum * sizeof(AzMtl));
        hipMemcpy(d_mtls, h_mtls.data(), mtlsNum * sizeof(AzMtl), hipMemcpyHostToDevice);
    }
}