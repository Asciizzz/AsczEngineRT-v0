#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <Utility.cuh>

#include <AsczTxtr.cuh>
#include <AsczMtl.cuh>
#include <AsczMesh.cuh>
#include <AsczBvh.cuh>
#include <AsczLight.cuh>
#include <AsczWin.cuh>

#include <FXAA.cuh>

#include <RayTrace.cuh>

int main() {
    // =================== Initialize FPS and Window ==============
    FpsHandler &FPS = FpsHandler::instance();
    AsczWin WinMgr(1280, 720, L"AsczEngineRT");

    // =================== Initialize window ===================

    // =============== Initialize Important Managers ================

    // All managers
    AsczTxtr TxtrMgr;
    AsczMtl MtlMgr;
    AsczMesh MeshMgr;
    AsczBvh BvhMgr;
    AsczLight LightMgr;

    // Create Camera
    // By logic, then this is CameraManager?
    // Idk, just a funny thought
    Camera CAMERA;

    // ====================== Some very scuffed init ==========================
    
    std::ifstream cfgFile(".cfg");
    std::string cfgLine;
    while (std::getline(cfgFile, cfgLine)) {
        if (cfgLine.size() == 0 || cfgLine[0] == '#') continue;

        std::stringstream ss(cfgLine);
        std::string type; ss >> type;

        if (type == "CameraPos")
            ss >> CAMERA.pos.x >> CAMERA.pos.y >> CAMERA.pos.z;
        else if (type == "CameraRot")
            ss >> CAMERA.rot.x >> CAMERA.rot.y >> CAMERA.rot.z;
        else if (type == "CameraFov")
            ss >> CAMERA.fov;
        else if (type == "VelSpec")
            ss >> CAMERA.velSpec;
        else if (type == "SlowFactor")
            ss >> CAMERA.slowFactor;
        else if (type == "FastFactor")
            ss >> CAMERA.fastFactor;

        if (type == "LightSrc") {
            LightSrc lSrc; ss >>
                lSrc.pos.x >> lSrc.pos.y >> lSrc.pos.z >>
                lSrc.colr.x >> lSrc.colr.y >> lSrc.colr.z >>
                lSrc.intens >>
                lSrc.falloff >> lSrc.bias >> lSrc.exp >> lSrc.falloffDist;

            LightMgr.appendLight(lSrc);
        }

        if (type == "MaxDepth")
            ss >> BvhMgr.MAX_DEPTH;
        else if (type == "BinCount")
            ss >> BvhMgr.BIN_COUNT;
    };

    // ========================================================================
    // ========================= Buffer Allocation ============================
    // ========================================================================

    // Allocate frame buffers

    // ========================================================================
    // ======================= Some test geometries ===========================
    // ========================================================================

    // Test object loading
    // Load object file from .model
    std::ifstream objsFile(".model");
    std::string objLine;

    while (std::getline(objsFile, objLine)) {
        if (objLine.size() == 0 || objLine[0] == '#') continue;
        if (objLine[0] == '~') break;

        std::stringstream ss(objLine);

        std::string objPath;
        short objPlacement = 0;
        float objScale = 1.0f;

        ss >> objPath >> objPlacement >> objScale;

        Utils::appendObj(
            MeshMgr, MtlMgr, TxtrMgr,
            objPath.c_str(), objPlacement, objScale
        );
    }

    // ======================= Copy to device memory ==========================

    // Copy to device memory
    TxtrMgr.toDevice();
    MtlMgr.toDevice();
    MeshMgr.toDevice();

    BvhMgr.designBVH(MeshMgr);
    BvhMgr.toDevice();

    LightMgr.toDevice();

    // ========================================================================
    // ========================================================================

    MSG msg = { 0 };
    while (msg.message != WM_QUIT) {
        FPS.startFrame();

        if (PeekMessage(&msg, nullptr, 0, 0, PM_REMOVE)) {
            TranslateMessage(&msg);
            DispatchMessage(&msg);
        }

        if (CAMERA.focus) {
            // Get previous cursor position
            POINT prev;
            GetCursorPos(&prev);

            // Set cursor position to the center of the window
            POINT center = { WinMgr.width / 2, WinMgr.height / 2 };
            ClientToScreen(WinMgr.hwnd, &center);
            SetCursorPos(center.x, center.y);

            float dx = prev.x - center.x;
            float dy = center.y - prev.y;

            // Update camera rotation
            CAMERA.rot.y += dx * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.x += dy * CAMERA.mSens * FPS.dTimeSec;

            // For the time being, press the arrow keys to look around
            bool k_up = WinMgr.keys[VK_UP];
            bool k_dw = WinMgr.keys[VK_DOWN];
            bool k_lf = WinMgr.keys[VK_LEFT];
            bool k_rt = WinMgr.keys[VK_RIGHT];

            if (k_up && !k_dw) CAMERA.rot.x += CAMERA.mSens * FPS.dTimeSec;
            if (k_dw && !k_up) CAMERA.rot.x -= CAMERA.mSens * FPS.dTimeSec;
            if (k_lf && !k_rt) CAMERA.rot.y -= CAMERA.mSens * FPS.dTimeSec;
            if (k_rt && !k_lf) CAMERA.rot.y += CAMERA.mSens * FPS.dTimeSec;

            // CSGO perspective movement
            float vel = CAMERA.velSpec;
            bool k_w = WinMgr.keys['W'];
            bool k_a = WinMgr.keys['A'];
            bool k_s = WinMgr.keys['S'];
            bool k_d = WinMgr.keys['D'];
            bool k_ctrl = WinMgr.keys[VK_LCONTROL];
            bool k_shift = WinMgr.keys[VK_LSHIFT];

            // Hold ctrl to go slow, hold shift to go fast
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;

            // Press W/S to move forward/backward
            if (k_w && !k_s) CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
            if (k_s && !k_w) CAMERA.pos -= CAMERA.forward * vel * FPS.dTimeSec;

            // Press A/D to move left/right
            if (k_a && !k_d) CAMERA.pos -= CAMERA.right * vel * FPS.dTimeSec;
            if (k_d && !k_a) CAMERA.pos += CAMERA.right * vel * FPS.dTimeSec;

            // Update camera
            CAMERA.update();
        } else {
            ClipCursor(nullptr);
        }

        // Render frmbuffer
        raytraceKernel<<<WinMgr.blockCount, WinMgr.threadCount>>>(
            CAMERA, WinMgr.d_framebuffer, WinMgr.width, WinMgr.height,
            TxtrMgr.d_txtrFlat, TxtrMgr.d_txtrPtr, MtlMgr.d_mtls,
            MeshMgr.d_v, MeshMgr.d_t, MeshMgr.d_n, MeshMgr.d_geom, MeshMgr.gNum,
            BvhMgr.d_gIdx, BvhMgr.d_nodes, BvhMgr.nNum,
            LightMgr.d_lSrc, LightMgr.num
        );

        WinMgr.Draw();

        FPS.endFrame();
    }

    // ========================================================================
    // ========================================================================

    // Free everything
    TxtrMgr.freeDevice();
    MtlMgr.freeDevice();
    MeshMgr.freeDevice();
    BvhMgr.freeDevice();

    WinMgr.Clear();

    return 0;
}
