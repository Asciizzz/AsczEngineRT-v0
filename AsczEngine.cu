#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>

#include <hip/hip_runtime.h>

#include <Camera.cuh>
#include <SFMLTexture.cuh>
#include <Utility.cuh>

/* Goal:

Ray intersection with AABB: Done
Ray reflection: Done
Create rays from camera: Done
Ray casting: Done
Ray recursion: Done

Additional notes:

During ray recursion, the color will become darker
since mirror irl get darker as it reflects more
due to the loss of light energy.

*/

__device__ float shadowMultiplier(Vec3f color) {
    // real color = base color * shadowMultiplier
    // Darker colors are less affected by shadows

    float a = 0.1;
    float b = 0.8;
    float k = 5.0;

    float avg = (color.x + color.y + color.z) / 3.0f;
    float multiplier = a + (b - a) / (1 + expf(-k * (avg - 0.5f)));
    return 1 - multiplier;
}

__global__ void clearFramebuffer(Vec3f *framebuffer, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    framebuffer[idx] = Vec3f(0, 0, 0);
}

__global__ void resetRecursive(bool *raycursive, int *recursionidx, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    raycursive[idx] = true; // To kickstart the first iteration
    recursionidx[idx] = idx;
}

__global__ void generateRays(Camera camera, Ray *rays, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    int x = idx % width;
    int y = idx / width;

    rays[y * width + x] = camera.castRay(x, y, width, height);
}

__global__ void castRays(
    Vec3f *framebuffer, Vec3f *vertexbuffer, Vec3f *normalbuffer,
    Ray *rays, bool *raycursive, int *recursionidx, bool *hasrecursive,
    Vec3f lightPos,
    Triangle *triangles, int width, int height, int triangleCount
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    if (!raycursive[idx]) return;

    Ray ray = rays[idx];

    int curTri = -1;
    float curZ = 1000000.0f;
    // This will soon be replaced with BVH traversal method
    for (int i = 0; i < triangleCount; i++) {
        if (!triangles[i].display) continue;

        Vec3f A = triangles[i].v0;
        Vec3f B = triangles[i].v1;
        Vec3f C = triangles[i].v2;

        Vec3f e1 = B - A;
        Vec3f e2 = C - A;

        Vec3f h = ray.direction & e2;
        float a = e1 * h;

        // Ray is parallel to the triangle
        if (a > -0.00001 && a < 0.00001) continue;

        float f = 1.0f / a;
        Vec3f s = ray.origin - A;
        float u = f * (s * h);

        if (u < 0.0f || u > 1.0f) continue;

        Vec3f q = s & e1;
        float v = f * (ray.direction * q);

        if (v < 0.0f || u + v > 1.0f) continue;

        float t = f * (e2 * q);

        if (t > 0.00001 && t < curZ) {
            curZ = t;

            // Saving relevant data to avoid recalculating  
            curTri = i;
        }
    }

    if (curTri == recursionidx[idx] && curTri != -1) {
        raycursive[idx] = false;
        return;
    }

    bool recursive = triangles[curTri].reflect;

    Vec3f vertex = ray.origin + ray.direction * curZ;
    Vec3f bary = Vec3f::bary(vertex, triangles[curTri].v0, triangles[curTri].v1, triangles[curTri].v2);
    float u = bary.x;
    float v = bary.y;
    float w = bary.z;

    Vec3f normal = triangles[curTri].n0 * w
                + triangles[curTri].n1 * u
                + triangles[curTri].n2 * v;
    normal.norm();
    vertexbuffer[idx] = vertex;
    normalbuffer[idx] = normal;

    if (recursive) {
        // Set the recursive ray
        Ray recursiveRay;
        recursiveRay.direction = ray.reflect(normal);
        recursiveRay.origin = vertex + normal * 0.001f; // To avoid self-intersection
        rays[idx] = recursiveRay;

        raycursive[idx] = true;
        recursionidx[idx] = curTri;
        *hasrecursive = true;
    } else {
        raycursive[idx] = false;

        // Interpolate color
        Vec3f color = triangles[curTri].c0 * w
                    + triangles[curTri].c1 * u
                    + triangles[curTri].c2 * v;

        // Lighting (with ambient, diffuse, specular and shininess)
        Vec3f lightDir = lightPos - vertex;
        lightDir.norm();

        float ka = triangles[curTri].ambient;
        float kd = triangles[curTri].diffuse;
        float ks = triangles[curTri].specular;
        float shine = triangles[curTri].shininess;

        float ambient = ka;
        float diffuse = kd * fmaxf(0.0f, normal * lightDir);
        Vec3f reflectDir = lightDir - normal * (2.0f * (lightDir * normal));
        float specular = ks * pow(fmaxf(0.0f, ray.direction * reflectDir), shine);

        framebuffer[idx] = color * (ambient + diffuse + specular);
    }
}

__global__ void applyShadow(
    Vec3f *framebuffer, Vec3f *vertexbuffer, Vec3f *normalbuffer,
    Vec3f lightPos,
    Triangle *triangles, int width, int height, int triangleCount
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    Ray ray;
    ray.origin = vertexbuffer[idx] + normalbuffer[idx] * 0.1f; // To avoid self-intersection
    ray.direction = lightPos - vertexbuffer[idx];

    // Check if the line connecting the point and the light source intersects with any triangle
    for (int i = 0; i < triangleCount; i++) {
        if (!triangles[i].display) continue;

        Vec3f A = triangles[i].v0;
        Vec3f B = triangles[i].v1;
        Vec3f C = triangles[i].v2;

        Vec3f e1 = B - A;
        Vec3f e2 = C - A;

        Vec3f h = ray.direction & e2;
        float a = e1 * h;

        // Ray is parallel to the triangle
        if (a > -0.00001 && a < 0.00001) continue;

        float f = 1.0f / a;
        Vec3f s = ray.origin - A;
        float u = f * (s * h);

        if (u < 0.0f || u > 1.0f) continue;

        Vec3f q = s & e1;
        float v = f * (ray.direction * q);

        if (v < 0.0f || u + v > 1.0f) continue;

        float t = f * (e2 * q);

        if (t > 0.00001 && t < 1.0f) {
            // The point is in shadow
            framebuffer[idx] *= shadowMultiplier(framebuffer[idx]);
            break;
        }
    }
}

__global__ void calcLuminance(float *lumabuffer, Vec3f *framebuffer, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    Vec3f color = framebuffer[idx];
    lumabuffer[idx] = 0.299f * color.x + 0.587f * color.y + 0.114f * color.z;
}

__global__ void maskEdge(bool *edgebuffer, float *lumabuffer, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    int x = idx % width;
    int y = idx / width;

    // Ignore the window edges
    if (x == 0 || x == width - 1 || y == 0 || y == height - 1) {
        edgebuffer[idx] = false;
        return;
    }

    // Find the 12 surrounding pixels

    float luma = lumabuffer[y * width + x];
    // Adjacent pixels
    float lumaL = lumabuffer[y * width + x - 1];
    float lumaU = lumabuffer[(y - 1) * width + x];
    float lumaD = lumabuffer[(y + 1) * width + x];
    float lumaR = lumabuffer[y * width + x + 1];
    // Diagonal pixels
    float lumaLU = lumabuffer[(y - 1) * width + x - 1];
    float lumaRU = lumabuffer[(y - 1) * width + x + 1];
    float lumaLD = lumabuffer[(y + 1) * width + x - 1];
    float lumaRD = lumabuffer[(y + 1) * width + x + 1];
    // Adjacent*2 pixels
    float lumaLL = lumabuffer[y * width + x - 2];
    float lumaUU = lumabuffer[(y - 2) * width + x];
    float lumaDD = lumabuffer[(y + 2) * width + x];
    float lumaRR = lumabuffer[y * width + x + 2];

    float contrast = abs(luma * 4 - lumaL - lumaU - lumaD - lumaR)
                   + abs(luma * 4 - lumaLU - lumaRU - lumaLD - lumaRD)
                   + abs(luma * 4 - lumaLL - lumaUU - lumaDD - lumaRR);

    float edgeThreshold = 0.01f;
    edgebuffer[idx] = contrast > edgeThreshold;
}

__global__ void FXAA(Vec3f *framebuffer, bool *edgebuffer, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    if (!edgebuffer[idx]) return;

    int x = idx % width;
    int y = idx / width;

    if (x == 0 || x == width - 1 || y == 0 || y == height - 1) return;

    Vec3f color = framebuffer[idx];

    // Get the average of the 4 neighbors
    Vec3f colorSum = color;
    int count = 1;

    // Adjacent pixels
    if (edgebuffer[y * width + x - 1]) {
        colorSum += framebuffer[y * width + x - 1];
        count++;
    }
    if (edgebuffer[y * width + x + 1]) {
        colorSum += framebuffer[y * width + x + 1];
        count++;
    }
    if (edgebuffer[(y - 1) * width + x]) {
        colorSum += framebuffer[(y - 1) * width + x];
        count++;
    }
    if (edgebuffer[(y + 1) * width + x]) {
        colorSum += framebuffer[(y + 1) * width + x];
        count++;
    }
    // Diagonal pixels
    if (edgebuffer[(y - 1) * width + x - 1]) {
        colorSum += framebuffer[(y - 1) * width + x - 1];
        count++;
    }
    if (edgebuffer[(y - 1) * width + x + 1]) {
        colorSum += framebuffer[(y - 1) * width + x + 1];
        count++;
    }
    if (edgebuffer[(y + 1) * width + x - 1]) {
        colorSum += framebuffer[(y + 1) * width + x - 1];
        count++;
    }
    if (edgebuffer[(y + 1) * width + x + 1]) {
        colorSum += framebuffer[(y + 1) * width + x + 1];
        count++;
    }
    // Adjacent*2 pixels
    if (edgebuffer[y * width + x - 2]) {
        colorSum += framebuffer[y * width + x - 2];
        count++;
    }
    if (edgebuffer[y * width + x + 2]) {
        colorSum += framebuffer[y * width + x + 2];
        count++;
    }
    if (edgebuffer[(y - 2) * width + x]) {
        colorSum += framebuffer[(y - 2) * width + x];
        count++;
    }
    if (edgebuffer[(y + 2) * width + x]) {
        colorSum += framebuffer[(y + 2) * width + x];
        count++;
    }

    framebuffer[idx] = colorSum / count;
}

int main() {
    /*
    Isn't it funny how these things have been here
    For 3 entire versions of the engine?
    They been through 3 "code wars"
    */
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();

    // Create SFMLTexture
    int width = 1600;
    int height = 900;
    SFMLTexture SFTex(width, height);

    int threads = 256;
    int blocks = (width * height + threads - 1) / threads;

    // Test camera
    Camera CAMERA;
    CAMERA.pos = Vec3f(0, 0, -10);
    CAMERA.rot = Vec3f(0, 0, 0);
    CAMERA.updateView();

    // Set up buffers
    Vec3f *d_framebuffer; // or colorbuffer
    Vec3f *d_vertexbuffer;
    Vec3f *d_normalbuffer;
    hipMalloc(&d_framebuffer, width * height * sizeof(Vec3f));
    hipMalloc(&d_vertexbuffer, width * height * sizeof(Vec3f));
    hipMalloc(&d_normalbuffer, width * height * sizeof(Vec3f));
    // Buffers for FXAA
    float *d_lumabuffer;
    bool *d_edgebuffer;
    hipMalloc(&d_lumabuffer, width * height * sizeof(float));
    hipMalloc(&d_edgebuffer, width * height * sizeof(bool));

    // Set up rays
    Ray *d_rays;
    bool *d_raycursive; // Pun intended
    int *d_recursionidx; // The origin of the recursive ray
    bool *d_hasrecursive;
    hipMalloc(&d_rays, width * height * sizeof(Ray));
    hipMalloc(&d_raycursive, width * height * sizeof(bool));
    hipMalloc(&d_recursionidx, width * height * sizeof(int));
    hipMalloc(&d_hasrecursive, sizeof(bool));

    // Set the hasrecursive true
    hipMemcpy(d_hasrecursive, new bool(true), sizeof(bool), hipMemcpyHostToDevice);

    // Creating some test triangles
    std::vector<Triangle> shape0 = Utils::readObjFile("test", "assets/Models/Shapes/Test/test0.obj");
    #pragma omp parallel
    for (int i = 0; i < shape0.size(); i++) {
        int scaleFac = 40;
        shape0[i].v0.scale(Vec3f(), scaleFac);
        shape0[i].v1.scale(Vec3f(), scaleFac);
        shape0[i].v2.scale(Vec3f(), scaleFac);
    }

    std::vector<Triangle> shape1 = Utils::readObjFile("test", "assets/Models/Shapes/Test/test1.obj");
    #pragma omp parallel
    for (Triangle &t : shape1) {
        t.reflect = true;
        // t.display = false;

        int scaleFac = 16;
        t.scale(Vec3f(), scaleFac);
        t.translate(Vec3f(0, 0, 39.8));
    }

    std::vector<Triangle> shape2 = Utils::readObjFile("test1", "assets/Models/Shapes/Test/test2.obj");
    #pragma omp parallel
    for (Triangle &t : shape2) {
        t.reflect = true;

        int scaleFac = 16;
        t.scale(Vec3f(), scaleFac);
        t.translate(Vec3f(0, 0, -39.8));
    }

    std::vector<Triangle> shape3 = Utils::readObjFile("test2", "assets/Models/Shapes/Test/test3.obj");
    #pragma omp parallel
    for (Triangle &t : shape3) {
        t.reflect = true;

        float scaleFac = 4;
        t.scale(Vec3f(), scaleFac);

        // // Invert normals
        // t.n0 = -t.n0;
        // t.n1 = -t.n1;
        // t.n2 = -t.n2;
    }

    std::vector<Triangle> triangles = shape0;
    triangles.insert(triangles.end(), shape1.begin(), shape1.end());
    triangles.insert(triangles.end(), shape2.begin(), shape2.end());
    triangles.insert(triangles.end(), shape3.begin(), shape3.end());

    // Copy to device
    Triangle *d_triangles;
    int tc = triangles.size();
    hipMalloc(&d_triangles, tc * sizeof(Triangle));
    hipMemcpy(d_triangles, triangles.data(), tc * sizeof(Triangle), hipMemcpyHostToDevice);

    // Test light   
    Vec3f lightSrc = Vec3f(0, 0, 0);

    // Create window
    sf::RenderWindow window(sf::VideoMode(width, height), "AsczEngine");
    sf::Mouse::setPosition(sf::Vector2i(width / 2, height / 2), window);
    window.setMouseCursorVisible(!CAMERA.focus);

    // Fun settings
    bool followLight = false;
    bool blackenScreen = false;
    bool hasAntiAliasing = true;
    bool hasShadow = true;

    // Crosshair
    int crosshairSize = 10;
    int crosshairThick = 2;
    sf::Color crosshairColor = sf::Color::Green;
    sf::RectangleShape crosshair1(
        sf::Vector2f(crosshairSize + crosshairThick, crosshairThick)
    );
    crosshair1.setPosition(width / 2 - crosshairSize / 2, height / 2);
    crosshair1.setFillColor(crosshairColor);
    sf::RectangleShape crosshair2(
        sf::Vector2f(crosshairThick, crosshairSize + crosshairThick)
    );
    crosshair2.setPosition(width / 2, height / 2 - crosshairSize / 2);
    crosshair2.setFillColor(crosshairColor);

    // Black opaque rectangle
    sf::RectangleShape blackScreen(sf::Vector2f(width, height));
    blackScreen.setFillColor(sf::Color(0, 0, 0, 180));

    // Main loop
    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {      
                window.close();
            }

            // Press f1 to toggle camera focus
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::F1) {
                CAMERA.focus = !CAMERA.focus;
                sf::Mouse::setPosition(sf::Vector2i(width / 2, height / 2), window);

                // Hide cursor
                window.setMouseCursorVisible(!CAMERA.focus);
            }

            // Press L to toggle light follow
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::L) {
                followLight = !followLight;
            }

            // Press B to toggle blacken screen
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::B) {
                blackenScreen = !blackenScreen;
            }

            // Press 1 to toggle anti-aliasing
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::Num1) {
                hasAntiAliasing = !hasAntiAliasing;
            }

            // Press 2 to toggle shadow
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::Num2) {
                hasShadow = !hasShadow;
            }
        }

        // Setting input activities
        bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
        bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);

        bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
        bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);

        bool k_w = sf::Keyboard::isKeyPressed(sf::Keyboard::W);
        bool k_a = sf::Keyboard::isKeyPressed(sf::Keyboard::A);
        bool k_s = sf::Keyboard::isKeyPressed(sf::Keyboard::S);
        bool k_d = sf::Keyboard::isKeyPressed(sf::Keyboard::D);
        bool k_space = sf::Keyboard::isKeyPressed(sf::Keyboard::Space);

        bool k_q = sf::Keyboard::isKeyPressed(sf::Keyboard::Q);
        bool k_e = sf::Keyboard::isKeyPressed(sf::Keyboard::E);
        bool k_t = sf::Keyboard::isKeyPressed(sf::Keyboard::T);

        // Fun settings
        if (followLight) {
            lightSrc = CAMERA.pos;
        }

        // Camera movement
        if (CAMERA.focus) {
        // Camera look around
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(
                width / 2, height / 2
            ), window);

            // Move from center
            int dMx = mousepos.x - width / 2;
            int dMy = mousepos.y - height / 2;

            // Camera look around
            CAMERA.rot.x -= dMy * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.y += dMx * CAMERA.mSens * FPS.dTimeSec;

        // Csgo perspective mode movement
            float vel = CAMERA.velSpec;
            // Hold ctrl to go slow, hold shift to go fast
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;
            // Press W/S to move forward/backward
            if (k_w && !k_s) CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
            if (k_s && !k_w) CAMERA.pos -= CAMERA.forward * vel * FPS.dTimeSec;
            // Press A/D to move left/right
            if (k_a && !k_d) CAMERA.pos -= CAMERA.right * vel * FPS.dTimeSec;
            if (k_d && !k_a) CAMERA.pos += CAMERA.right * vel * FPS.dTimeSec;
        }
        // Update camera
        CAMERA.update();

        // Clear framebuffer
        clearFramebuffer<<<blocks, threads>>>(d_framebuffer, width, height);    
        hipDeviceSynchronize();

        // Generate rays
        generateRays<<<blocks, threads>>>(CAMERA, d_rays, width, height);
        hipDeviceSynchronize();

        // Recursive ray tracing

        // Set all to true to kickstart the first iteration
        resetRecursive<<<blocks, threads>>>(d_raycursive, d_recursionidx, width, height);
        hipDeviceSynchronize();

        bool *hasrecursive = new bool(true);
        int recursionCount = -1;
        while (*hasrecursive) {
            *hasrecursive = false;
            hipMemcpy(d_hasrecursive, hasrecursive, sizeof(bool), hipMemcpyHostToDevice); 

            // Cast rays
            castRays<<<blocks, threads>>>(
                d_framebuffer, d_vertexbuffer, d_normalbuffer,
                d_rays, d_raycursive, d_recursionidx, d_hasrecursive,
                lightSrc,
                d_triangles, width, height, tc);
            hipDeviceSynchronize();

            // Copy hasrecursive to host
            hipMemcpy(hasrecursive, d_hasrecursive, sizeof(bool), hipMemcpyDeviceToHost);

            recursionCount++;

            if (recursionCount == 10) break; // Break if it's too much
        }

        // Apply shadow
        if (hasShadow) {
            applyShadow<<<blocks, threads>>>(
                d_framebuffer, d_vertexbuffer, d_normalbuffer,
                lightSrc,
                d_triangles, width, height, tc);
            hipDeviceSynchronize();
        }

        // FXAA
        if (hasAntiAliasing) {
            calcLuminance<<<blocks, threads>>>(d_lumabuffer, d_framebuffer, width, height);
            hipDeviceSynchronize();

            maskEdge<<<blocks, threads>>>(d_edgebuffer, d_lumabuffer, width, height);
            hipDeviceSynchronize();

            FXAA<<<blocks, threads>>>(d_framebuffer, d_edgebuffer, width, height);
            hipDeviceSynchronize();
        }

        // Update "texture"
        SFTex.updateTexture(d_framebuffer, width, height, 1);

        // Log
        LOG.addLog("Welcome to AsczEngineRT v0", sf::Color::Green, 1);
        LOG.addLog("FPS: " + std::to_string(FPS.fps), sf::Color::Green);
        LOG.addLog("Recursion count: " + std::to_string(recursionCount), sf::Color::Red);
        LOG.addLog("Triangles count: " + std::to_string(tc), sf::Color::Red);
        LOG.addLog(CAMERA.data(), sf::Color(160, 255, 160));
        // Print the pixel at the dead center
        int idx = SFTex.pixelCount / 2 + width * 2;
        sf::Uint8 px1 = SFTex.sfPixel[idx + 0];
        sf::Uint8 px2 = SFTex.sfPixel[idx + 1];
        sf::Uint8 px3 = SFTex.sfPixel[idx + 2];
        Vec3f color = Vec3f(px1, px2, px3);
        LOG.addLog("Color: "
            + std::to_string(color.x) + ", "
            + std::to_string(color.y) + ", "
            + std::to_string(color.z),
        sf::Color(255 - px1, 255 - px2, 255 - px3)); // Contrast color for better visibility
        // Settings
        LOG.addLog("Settings:", sf::Color(255, 160, 160), 1);
        LOG.addLog("[L] Follow light: " + std::string(followLight ? "true" : "false"), sf::Color(255, 160, 160));
        LOG.addLog("[B] Blacken screen: " + std::string(blackenScreen ? "true" : "false"), sf::Color(255, 160, 160));
        LOG.addLog("[1] Anti-aliasing: " + std::string(hasAntiAliasing ? "true" : "false"), sf::Color(255, 160, 160));
        LOG.addLog("[2] Shadow: " + std::string(hasShadow ? "true" : "false"), sf::Color(255, 160, 160));


        // Draw to window
        window.clear(sf::Color::Black);
        window.draw(SFTex.sprite);
        // To see the log better
        if (blackenScreen) window.draw(blackScreen);
        LOG.drawLog(window);
        window.draw(crosshair1);
        window.draw(crosshair2);
        window.display();

        // Frame end
        FPS.endFrame();
    }

    return 0;
}