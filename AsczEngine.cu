#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>

#include <Utility.cuh>
#include <random>

#include <TxtrManager.cuh>
#include <MatManager.cuh>
#include <MeshManager.cuh>
#include <BvhManager.cuh>

#include <FXAA.cuh>

#include <RayTrace.cuh>
#include <SFMLTexture.cuh>

int main() {
    // =================== Initialize FPS and Log ==============
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();
    LOG.fontSize = 24;

    // =================== Initialize window ===================

    // Create window (set to fullscreen)
    int winW = sf::VideoMode::getDesktopMode().width;
    int winH = sf::VideoMode::getDesktopMode().height;

    sf::RenderWindow window(sf::VideoMode(winW, winH), "AsczEngine");
    sf::Mouse::setPosition(sf::Vector2i(winW / 2, winH / 2), window);
    window.setMouseCursorVisible(false);

    // Crosshair
    int crosshairSize = 10;
    int crosshairThick = 2;
    sf::Color crosshairColor = sf::Color::Green;
    sf::RectangleShape crosshair1(
        sf::Vector2f(crosshairSize + crosshairThick, crosshairThick)
    );
    crosshair1.setPosition(winW / 2 - crosshairSize / 2, winH / 2);
    crosshair1.setFillColor(crosshairColor);
    sf::RectangleShape crosshair2(
        sf::Vector2f(crosshairThick, crosshairSize + crosshairThick)
    );
    crosshair2.setPosition(winW / 2, winH / 2 - crosshairSize / 2);
    crosshair2.setFillColor(crosshairColor);

    // =============== Initialize Important Managers ================

    // All managers
    BvhManager BvhMgr;
    MeshManager MeshMgr;
    TxtrManager TxtrMgr;
    MatManager MatMgr;
    MatMgr.appendMaterial(Material()); // Default material

    // Create Camera
    // By logic, then this is CameraManager?
    // Idk, just a funny thought
    Camera CAMERA;

    // Set frame buffer properties
    float frmScl = 1;
    int frmW = winW / frmScl;
    int frmH = winH / frmScl;

    // Some debugging values

    Vec3f lightSrc = Vec3f(0, 10, 0);

    bool hasFXAA = true;

    // ====================== Some very scuffed init ==========================
    
    std::ifstream cfgFile(".cfg");
    std::string cfgLine;
    while (std::getline(cfgFile, cfgLine)) {
        if (cfgLine.size() == 0 || cfgLine[0] == '#') continue;

        std::stringstream ss(cfgLine);
        std::string type; ss >> type;

        if (type == "CameraPos") {
            ss >> CAMERA.pos.x >> CAMERA.pos.y >> CAMERA.pos.z;
        }
        else if (type == "CameraRot") {
            ss >> CAMERA.rot.x >> CAMERA.rot.y >> CAMERA.rot.z;
        }
        else if (type == "CameraFov") {
            ss >> CAMERA.fov;
        }

        if (type == "LightSrc") {
            ss >> lightSrc.x >> lightSrc.y >> lightSrc.z;
        }

        if (type == "FrameScl") {
            ss >> frmScl;
            frmW = winW / frmScl;
            frmH = winH / frmScl;
        }

        if (type == "FXAA") {
            ss >> hasFXAA;
        }
    };

    // Allocate frame buffer
    int threads = 256;
    int blocks = (frmW * frmH + threads - 1) / threads;
    Vec3f *d_frmbuffer1, *d_frmbuffer2;
    hipMalloc(&d_frmbuffer1, frmW * frmH * sizeof(Vec3f));
    hipMalloc(&d_frmbuffer2, frmW * frmH * sizeof(Vec3f));

    // Allocate luminance buffer
    float *d_luminance; bool *d_edge;
    hipMalloc(&d_luminance, frmW * frmH * sizeof(float));
    hipMalloc(&d_edge, frmW * frmH * sizeof(bool));

    // Create SFML texture
    SFMLTexture SFTex(frmW, frmH);
    SFTex.sprite.setScale(frmScl, frmScl);

    // ======================================================================== 
    // ======================= Some test geometries ===========================
    // ========================================================================

    // Test object loading
    // Load object file from .model
    std::ifstream objsFile(".model");
    std::string objLine;

    while (std::getline(objsFile, objLine)) {
        if (objLine.size() == 0 || objLine[0] == '#') continue;
        if (objLine[0] == '~') break;

        std::stringstream ss(objLine);

        std::string objPath;
        short objPlacement;
        float objScale;

        ss >> objPath >> objPlacement >> objScale;

        Utils::appendObj(
            MeshMgr, MatMgr, TxtrMgr,
            objPath.c_str(), objPlacement, objScale
        );
    }

    // ================ Terrible attemp at building a BVH =====================

    // NodeHst *node = new NodeHst();
    // BvhManager::buildBvh(
    //     node, MeshMgr.ABmin, MeshMgr.ABmax,
    //     MeshMgr.h_v, MeshMgr.h_fv, MeshMgr.h_fi
    // );

    // ======================= Copy to device memory ==========================

    // Copy to device memory
    MeshMgr.hostToDevice();
    TxtrMgr.hostToDevice();
    MatMgr.hostToDevice();

    // ========================================================================
    // ========================================================================

    Vec3f prevPos = CAMERA.pos + 1;
    Vec3f prevRot = CAMERA.rot + 1;

    // Main loop
    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {      
                window.close();
            }

            // Press f1 to toggle camera focus
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::F1) {
                CAMERA.focus = !CAMERA.focus;
                // To avoid sudden camera movement when changing focus
                sf::Mouse::setPosition(sf::Vector2i(winW / 2, winH / 2), window);

                // Hide cursor
                window.setMouseCursorVisible(!CAMERA.focus);
            }

            // Scroll to change the field of view
            if (event.type == sf::Event::MouseWheelScrolled) {
                if (event.mouseWheelScroll.delta > 0) CAMERA.fov += 0.1;
                else if (event.mouseWheelScroll.delta < 0) CAMERA.fov -= 0.1;

                if (CAMERA.fov < 0.1) CAMERA.fov += 0.1;
                if (CAMERA.fov > M_PI - 0.1) CAMERA.fov -= 0.1;
            }
        }

        // Setting input activities
        bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
        bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);

        bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
        bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);

        bool k_w = sf::Keyboard::isKeyPressed(sf::Keyboard::W);
        bool k_a = sf::Keyboard::isKeyPressed(sf::Keyboard::A);
        bool k_s = sf::Keyboard::isKeyPressed(sf::Keyboard::S);
        bool k_d = sf::Keyboard::isKeyPressed(sf::Keyboard::D);
        bool k_space = sf::Keyboard::isKeyPressed(sf::Keyboard::Space);

        bool k_q = sf::Keyboard::isKeyPressed(sf::Keyboard::Q);
        bool k_e = sf::Keyboard::isKeyPressed(sf::Keyboard::E);
        bool k_t = sf::Keyboard::isKeyPressed(sf::Keyboard::T);

        // Camera movement
        if (CAMERA.focus) {
        // Camera look around
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(winW / 2, winH / 2), window);

            // Move from center
            int dMx = mousepos.x - winW / 2;
            int dMy = mousepos.y - winH / 2;

            // Camera look around
            CAMERA.rot.x -= dMy * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.y += dMx * CAMERA.mSens * FPS.dTimeSec;

        // CSGO perspective movement
            float vel = CAMERA.velSpec;
            // Hold ctrl to go slow, hold shift to go fast
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;
            // Press W/S to move forward/backward
            if (k_w && !k_s) CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
            if (k_s && !k_w) CAMERA.pos -= CAMERA.forward * vel * FPS.dTimeSec;
            // Press A/D to move left/right
            if (k_a && !k_d) CAMERA.pos -= CAMERA.right * vel * FPS.dTimeSec;
            if (k_d && !k_a) CAMERA.pos += CAMERA.right * vel * FPS.dTimeSec;
        }
        // Update camera
        CAMERA.update();

        if (prevPos != CAMERA.pos || prevRot != CAMERA.rot)
        {
            // Prepare frmbuffer
            clearFrameBuffer<<<blocks, threads>>>(d_frmbuffer1, frmW, frmH);
            hipDeviceSynchronize();

            // Render frmbuffer
            iterativeRayTracing<<<blocks, threads>>>(
                CAMERA, d_frmbuffer1, frmW, frmH,
                TxtrMgr.d_txtrFlat, TxtrMgr.d_txtrPtr,
                MatMgr.d_mats,
                MeshMgr.d_v, MeshMgr.d_t, MeshMgr.d_n,
                MeshMgr.d_fv, MeshMgr.d_ft, MeshMgr.d_fn, MeshMgr.d_fm,
                MeshMgr.fNum,

                lightSrc
            );
            hipDeviceSynchronize();

            // FXAA
            if (hasFXAA)
            {
                // FXAA
                calcLuminance<<<blocks, threads>>>(d_luminance, d_frmbuffer1, frmW, frmH);
                hipDeviceSynchronize();

                edgeMask<<<blocks, threads>>>(d_edge, d_luminance, frmW, frmH);
                hipDeviceSynchronize();

                applyFXAAtoBuffer<<<blocks, threads>>>(d_luminance, d_edge, d_frmbuffer1, d_frmbuffer2, frmW, frmH);
                hipDeviceSynchronize();

                SFTex.updateTexture(d_frmbuffer2, frmW, frmH);
            } else
                SFTex.updateTexture(d_frmbuffer1, frmW, frmH);
        }

        prevPos = CAMERA.pos;
        prevRot = CAMERA.rot;

        LOG.addLog("Welcome to AsczEngineRT v0", sf::Color::Green, 1);
        LOG.addLog("FPS: " + std::to_string(FPS.fps), sf::Color::Blue);
        LOG.addLog(CAMERA.data(), sf::Color::White, 0);

        // Clear window
        window.clear();

        window.draw(SFTex.sprite);

        // Draw the crosshair
        window.draw(crosshair1);
        window.draw(crosshair2);

        LOG.drawLog(window);

        // For the time being just draw the window
        window.display();

        // Frame end
        FPS.endFrame();
    }

    // Free device memory
    hipFree(d_frmbuffer1);
    hipFree(d_frmbuffer2);
    TxtrMgr.freeDevice();
    MatMgr.freeDevice();
    MeshMgr.freeDevice();

    return 0;
}
