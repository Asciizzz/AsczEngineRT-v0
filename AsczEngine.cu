#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>

#include <hip/hip_runtime.h>

#include <Camera.cuh>
#include <SFMLTexture.cuh>
#include <Utility.cuh>

/* Goal:

Ray intersection with AABB: Done
Ray reflection: Done
Create rays from camera: Done
Ray casting: Done
Ray recursion: Done

*/

__global__ void clearFramebuffer(Vec3f *framebuffer, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    framebuffer[idx] = Vec3f(0, 0, 0);
}

__global__ void resetRecursive(bool *raycursive, int *recursionidx, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    raycursive[idx] = true; // To kickstart the first iteration
    recursionidx[idx] = idx;
}

__global__ void generateRays(Camera camera, Ray *rays, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    int x = idx % width;
    int y = idx / width;

    rays[y * width + x] = camera.castRay(x, y, width, height);
}

__global__ void castRays(
    Vec3f *framebuffer, Vec3f *vertexbuffer, Vec3f *normalbuffer,
    Ray *rays, bool *raycursive, int *recursionidx, bool *hasrecursive,
    Vec3f lightPos,
    Triangle *triangles, int width, int height, int triangleCount
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    if (!raycursive[idx]) return;

    Ray ray = rays[idx];

    int curTri = -1;
    float curZ = 1000000.0f;
    // This will soon be replaced with BVH traversal method
    for (int i = 0; i < triangleCount; i++) {
        if (!triangles[i].display) continue;

        Vec3f A = triangles[i].v0;
        Vec3f B = triangles[i].v1;
        Vec3f C = triangles[i].v2;

        Vec3f e1 = B - A;
        Vec3f e2 = C - A;

        Vec3f h = ray.direction & e2;
        float a = e1 * h;

        // Ray is parallel to the triangle
        if (a > -0.00001 && a < 0.00001) continue;

        float f = 1.0f / a;
        Vec3f s = ray.origin - A;
        float u = f * (s * h);

        if (u < 0.0f || u > 1.0f) continue;

        Vec3f q = s & e1;
        float v = f * (ray.direction * q);

        if (v < 0.0f || u + v > 1.0f) continue;

        float t = f * (e2 * q);

        if (t > 0.00001 && t < curZ) {
            curZ = t;

            // Saving relevant data to avoid recalculating  
            curTri = i;
        }
    }

    if (curTri == recursionidx[idx] && curTri != -1) {
        raycursive[idx] = false;
        return;
    }

    bool recursive = triangles[curTri].reflect;

    Vec3f vertex = ray.origin + ray.direction * curZ;
    Vec3f bary = Vec3f::bary(vertex, triangles[curTri].v0, triangles[curTri].v1, triangles[curTri].v2);
    float u = bary.x;
    float v = bary.y;
    float w = bary.z;

    Vec3f normal = triangles[curTri].n0 * w
                + triangles[curTri].n1 * u
                + triangles[curTri].n2 * v;
    normal.norm();
    vertexbuffer[idx] = vertex;
    normalbuffer[idx] = normal;

    if (recursive) {
        // Set the recursive ray
        Ray recursiveRay;
        recursiveRay.direction = ray.reflect(normal);
        recursiveRay.origin = vertex + normal * 0.001f; // To avoid self-intersection
        rays[idx] = recursiveRay;

        raycursive[idx] = true;
        recursionidx[idx] = curTri;
        *hasrecursive = true;
    } else {
        raycursive[idx] = false;

        // Interpolate color
        Vec3f color = triangles[curTri].c0 * w
                    + triangles[curTri].c1 * u
                    + triangles[curTri].c2 * v;

        // Lighting
        Vec3f lightDir = lightPos - vertex;
        lightDir.norm();

        float diff = fmaxf(0.0f, normal * lightDir);
        float spec = 0.0f;

        Vec3f reflectDir = lightDir - normal * 2 * (lightDir * normal);
        spec = powf(fmaxf(0.0f, reflectDir * ray.direction), 32);

        Vec3f resultColor = color * diff * 0.8 + Vec3f(1, 1, 1) * spec * 0.2;

        framebuffer[idx] = resultColor;
    }
}

__global__ void applyShadow(
    Vec3f *framebuffer, Vec3f *vertexbuffer, Vec3f *normalbuffer,
    Vec3f lightPos,
    Triangle *triangles, int width, int height, int triangleCount
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    Ray ray;
    ray.origin = vertexbuffer[idx];
    ray.direction = lightPos - ray.origin;

    // Check if the line connecting the point and the light source intersects with any triangle
    for (int i = 0; i < triangleCount; i++) {
        if (!triangles[i].display) continue;

        Vec3f A = triangles[i].v0;
        Vec3f B = triangles[i].v1;
        Vec3f C = triangles[i].v2;

        Vec3f e1 = B - A;
        Vec3f e2 = C - A;

        Vec3f h = ray.direction & e2;
        float a = e1 * h;

        // Ray is parallel to the triangle
        if (a > -0.00001 && a < 0.00001) continue;

        float f = 1.0f / a;
        Vec3f s = ray.origin - A;
        float u = f * (s * h);

        if (u < 0.0f || u > 1.0f) continue;

        Vec3f q = s & e1;
        float v = f * (ray.direction * q);

        if (v < 0.0f || u + v > 1.0f) continue;

        float t = f * (e2 * q);

        if (t > 0.00001 && t < 1.0f) {
            framebuffer[idx] = framebuffer[idx] * 0.2;
            break;
        }
    }
}

int main() {
    /*
    Isn't it funny how these things have been here
    For 3 entire versions of the engine?
    They been through 3 "code wars"
    */
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();

    // Create SFMLTexture
    int width = 1600;
    int height = 900;
    SFMLTexture SFTex(width, height);
    
    int threads = 256;
    int blocks = (width * height + threads - 1) / threads;

    // Test camera
    Camera CAMERA;
    CAMERA.pos = Vec3f(0, 0, 0);
    CAMERA.rot = Vec3f(0, 0, 0);
    CAMERA.updateView();

    // Set up buffers
    Vec3f *d_framebuffer; // or colorbuffer
    Vec3f *d_vertexbuffer;
    Vec3f *d_normalbuffer;
    hipMalloc(&d_framebuffer, width * height * sizeof(Vec3f));
    hipMalloc(&d_vertexbuffer, width * height * sizeof(Vec3f));
    hipMalloc(&d_normalbuffer, width * height * sizeof(Vec3f));

    // Set up rays
    Ray *d_rays;
    bool *d_raycursive; // Pun intended
    int *d_recursionidx; // The origin of the recursive ray
    bool *d_hasrecursive;
    hipMalloc(&d_rays, width * height * sizeof(Ray));
    hipMalloc(&d_raycursive, width * height * sizeof(bool));
    hipMalloc(&d_recursionidx, width * height * sizeof(int));
    hipMalloc(&d_hasrecursive, sizeof(bool));

    // Set the hasrecursive true
    hipMemcpy(d_hasrecursive, new bool(true), sizeof(bool), hipMemcpyHostToDevice);

    // Creating some test triangles
    std::vector<Triangle> shape = Utils::readObjFile("test", "assets/Models/Shapes/Test/test.obj");
    #pragma omp parallel
    for (int i = 0; i < shape.size(); i++) {
        shape[i].reflect = true;
        // shape[i].display = false;

        int scaleFac = 10;
        shape[i].v0.scale(Vec3f(), scaleFac);
        shape[i].v1.scale(Vec3f(), scaleFac);
        shape[i].v2.scale(Vec3f(), scaleFac);

        shape[i].v0 = Vec3f::rotate(shape[i].v0, Vec3f(), Vec3f(1, 0, 0), M_PI_2);
        shape[i].v1 = Vec3f::rotate(shape[i].v1, Vec3f(), Vec3f(1, 0, 0), M_PI_2);
        shape[i].v2 = Vec3f::rotate(shape[i].v2, Vec3f(), Vec3f(1, 0, 0), M_PI_2);

        shape[i].n0 = Vec3f::rotate(shape[i].n0, Vec3f(), Vec3f(1, 0, 0), M_PI_2);
        shape[i].n1 = Vec3f::rotate(shape[i].n1, Vec3f(), Vec3f(1, 0, 0), M_PI_2);
        shape[i].n2 = Vec3f::rotate(shape[i].n2, Vec3f(), Vec3f(1, 0, 0), M_PI_2);
        shape[i].normAll();
    }

    std::vector<Triangle> room = Utils::readObjFile("test", "assets/Models/Shapes/Cube3.obj");
    #pragma omp parallel
    for (int i = 0; i < room.size(); i++) {
        int scaleFac = 40;
        room[i].v0.scale(Vec3f(), scaleFac);
        room[i].v1.scale(Vec3f(), scaleFac);
        room[i].v2.scale(Vec3f(), scaleFac);
    }

    std::vector<Triangle> triangles = shape;
    triangles.insert(triangles.end(), room.begin(), room.end());

    // Copy to device
    Triangle *d_triangles;
    int tc = triangles.size();
    hipMalloc(&d_triangles, tc * sizeof(Triangle));
    hipMemcpy(d_triangles, triangles.data(), tc * sizeof(Triangle), hipMemcpyHostToDevice);

    // Test light   
    Vec3f lightSrc = Vec3f(10, 3, 5);

    // Create window
    sf::RenderWindow window(sf::VideoMode(width, height), "AsczEngine");
    window.setMouseCursorVisible(!CAMERA.focus);

    // Fun settings
    bool followLight = false;

    // Main loop
    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {      
                window.close();
            }

            // Press f1 to toggle camera focus
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::F1) {
                CAMERA.focus = !CAMERA.focus;
                sf::Mouse::setPosition(sf::Vector2i(width / 2, height / 2), window);

                // Hide cursor
                window.setMouseCursorVisible(!CAMERA.focus);
            }

            // Press L to toggle light follow
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::L) {
                followLight = !followLight;
            }
        }

        // Setting input activities
        bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
        bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);

        bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
        bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);

        bool k_w = sf::Keyboard::isKeyPressed(sf::Keyboard::W);
        bool k_a = sf::Keyboard::isKeyPressed(sf::Keyboard::A);
        bool k_s = sf::Keyboard::isKeyPressed(sf::Keyboard::S);
        bool k_d = sf::Keyboard::isKeyPressed(sf::Keyboard::D);
        bool k_space = sf::Keyboard::isKeyPressed(sf::Keyboard::Space);

        bool k_q = sf::Keyboard::isKeyPressed(sf::Keyboard::Q);
        bool k_e = sf::Keyboard::isKeyPressed(sf::Keyboard::E);
        bool k_t = sf::Keyboard::isKeyPressed(sf::Keyboard::T);

        // Fun settings
        if (followLight) {
            lightSrc = CAMERA.pos;
        }

        // Camera movement
        if (CAMERA.focus) {
        // Camera look around
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(
                width / 2, height / 2
            ), window);

            // Move from center
            int dMx = mousepos.x - width / 2;
            int dMy = mousepos.y - height / 2;

            // Camera look around
            CAMERA.rot.x -= dMy * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.y -= dMx * CAMERA.mSens * FPS.dTimeSec;

        // Csgo perspective mode movement
            float vel = CAMERA.velSpec;
            // Hold ctrl to go slow, hold shift to go fast
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;
            // Press W/S to move forward/backward
            if (k_w && !k_s) CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
            if (k_s && !k_w) CAMERA.pos -= CAMERA.forward * vel * FPS.dTimeSec;
            // Press A/D to move left/right
            if (k_a && !k_d) CAMERA.pos -= CAMERA.right * vel * FPS.dTimeSec;
            if (k_d && !k_a) CAMERA.pos += CAMERA.right * vel * FPS.dTimeSec;
        }
        // Update camera
        CAMERA.update();

        // Clear framebuffer
        clearFramebuffer<<<blocks, threads>>>(d_framebuffer, width, height);    
        hipDeviceSynchronize();

        // Generate rays
        generateRays<<<blocks, threads>>>(CAMERA, d_rays, width, height);
        hipDeviceSynchronize();

        // Recursive ray tracing

        // Set all to true to kickstart the first iteration
        resetRecursive<<<blocks, threads>>>(d_raycursive, d_recursionidx, width, height);
        hipDeviceSynchronize();

        bool *hasrecursive = new bool(true);
        int recursionCount = -1;
        while (*hasrecursive) {
            *hasrecursive = false;
            hipMemcpy(d_hasrecursive, hasrecursive, sizeof(bool), hipMemcpyHostToDevice); 

            // Cast rays
            castRays<<<blocks, threads>>>(
                d_framebuffer, d_vertexbuffer, d_normalbuffer,
                d_rays, d_raycursive, d_recursionidx, d_hasrecursive,
                lightSrc,
                d_triangles, width, height, tc);
            hipDeviceSynchronize();

            // Copy hasrecursive to host
            hipMemcpy(hasrecursive, d_hasrecursive, sizeof(bool), hipMemcpyDeviceToHost);

            recursionCount++;

            if (recursionCount == 10) break; // Break if it's too much
        }

        // Apply shadow
        applyShadow<<<blocks, threads>>>(
            d_framebuffer, d_vertexbuffer, d_normalbuffer,
            lightSrc,
            d_triangles, width, height, tc);
        hipDeviceSynchronize();

        // Log
        LOG.addLog("Welcome to AsczEngineRT v0", sf::Color::Green, 1);
        LOG.addLog("FPS: " + std::to_string(FPS.fps), sf::Color::Green);
        LOG.addLog("Recursion count: " + std::to_string(recursionCount), sf::Color::Red);
        LOG.addLog("Triangles count: " + std::to_string(tc), sf::Color::Red);

        // Draw to window
        SFTex.updateTexture(d_framebuffer, width, height, 1);
        window.clear(sf::Color::Black);
        window.draw(SFTex.sprite);
        LOG.drawLog(window);
        window.display();

        // Frame end
        FPS.endFrame();
    }

    return 0;
}