#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>

#include <Camera.cuh>
#include <SFMLTexture.cuh>
#include <Utility.cuh>

struct RayHit {
    bool hit = false;
    int idx = -1;
    float weight = 1.0f;
    float t = 1e8;
    float u = 0;
    float v = 0;
    float w = 0;
    Vec3f vrtx;
    Vec2f txtr;
    Vec3f nrml;
    Vec3f colr;

    __device__ RayHit(float weight=1.0f) : weight(weight) {}
};

__device__ RayHit iterativeRayTracing(
    const Ray &primaryRay, const Triangle *triangles, int triNum
) {
    Ray rays[20] = { primaryRay };
    RayHit hits[20] = { RayHit() };
    int count = 0;

    for (int i = 0; i < count + 1; i++) {
        if (count > 16) break;

        Ray &ray = rays[i];
        RayHit &hit = hits[i];

        // Will be replaced with BVH
        for (int i = 0; i < triNum; i++) {
            const Triangle &tri = triangles[i];
            Vec3f e1 = tri.v1 - tri.v0;
            Vec3f e2 = tri.v2 - tri.v0;
            Vec3f h = ray.direction & e2;
            float a = e1 * h;

            if (a > -0.00001 && a < 0.00001) continue;

            float f = 1.0f / a;
            Vec3f s = ray.origin - tri.v0;
            float u = f * (s * h);

            if (u < 0.0f || u > 1.0f) continue;

            Vec3f q = s & e1;
            float v = f * (ray.direction * q);

            if (v < 0.0f || u + v > 1.0f) continue;

            float t = f * (e2 * q);

            if (t > 0.00001 && t < hit.t) {
                hit.hit = true;
                hit.idx = i;
                hit.t = t;
                hit.u = u;
                hit.v = v;
                hit.w = 1 - u - v;
            }
        }

        if (!hit.hit) break;

        Triangle tri = triangles[hit.idx];

        hit.vrtx = ray.origin + ray.direction * hit.t;
        hit.colr = tri.c0 * hit.w + tri.c1 * hit.u + tri.c2 * hit.v;
        hit.nrml = tri.n0 * hit.w + tri.n1 * hit.u + tri.n2 * hit.v;
        hit.nrml.norm();

        if (tri.reflect > 0.0f) {
            float weightLeft = hit.weight * tri.reflect;
            hit.weight *= (1 - tri.reflect);

            Vec3f reflDir = ray.reflect(hit.nrml);
            Vec3f reflOrigin = hit.vrtx + hit.nrml * 0.0001;

            rays[++count] = Ray(reflOrigin, reflDir);
            hits[count] = RayHit(weightLeft);
        }
        else if (tri.transmit > 0.0f) {
            float weightLeft = hit.weight * tri.transmit;
            hit.weight *= (1 - tri.transmit);

            Vec3f transOrg = hit.vrtx + ray.direction * 0.0001;

            rays[++count] = Ray(transOrg, ray.direction);
            hits[count] = RayHit(weightLeft);
        }
        else if (tri.Fresnel > 0.0f) {
            float weightLeft = hit.weight * tri.Fresnel;
            hit.weight *= (1 - tri.Fresnel);

            // Schlick's approximation
            float cosI = (-ray.direction) * hit.nrml;
            if (cosI < 0) cosI = -cosI;

            // Find the fresnel coefficient
            float R = pow(1 - cosI, 5);
            float Rrefl = R * weightLeft;
            float Rrefr = (1 - R) * weightLeft;

            // Refraction (for the time being just tranparent)
            Vec3f refrDir = ray.direction;
            Vec3f refrOrigin = hit.vrtx + refrDir * 0.0001;

            rays[++count] = Ray(refrOrigin, refrDir);
            hits[count] = RayHit(Rrefr);

            // Reflection
            Vec3f reflDir = ray.reflect(hit.nrml);
            Vec3f reflOrigin = hit.vrtx + hit.nrml * 0.0001;

            rays[++count] = Ray(reflOrigin, reflDir);
            hits[count] = RayHit(Rrefl);
        }
    }

    RayHit finalHit;
    finalHit.hit = true;
    for (int i = 0; i <= count; i++) {
        RayHit &hit = hits[i];
        finalHit.colr += hit.colr * hit.weight;
    }

    return finalHit;
}


__global__ void clearFrameBuffer(Vec3f *framebuffer, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width * height) framebuffer[i] = Vec3f(0, 0, 0);
}

__global__ void renderFrameBuffer(
    Vec3f *framebuffer, Camera camera, Triangle *triangles, int triNum, int width, int height
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= width * height) return;

    int x = i % width;
    int y = i / width;

    Ray ray = camera.castRay(x, y, width, height);
    RayHit hit = iterativeRayTracing(ray, triangles, triNum);

    if (hit.hit) framebuffer[i] = hit.colr;
}

int main() {
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();

    hipFree(0);  // Force context initialization
    hipDeviceSetLimit(hipLimitStackSize, 256 * 1024);

    // Create SFMLTexture
    int width = 1600;
    int height = 900;
    SFMLTexture SFTex(width, height);

    // Test camera
    Camera CAMERA;
    CAMERA.pos = Vec3f(0, 0, -10);
    CAMERA.rot = Vec3f(0, 0, 0);
    CAMERA.updateView();

    // Create window
    sf::RenderWindow window(sf::VideoMode(width, height), "AsczEngine");
    sf::Mouse::setPosition(sf::Vector2i(width / 2, height / 2), window);
    window.setMouseCursorVisible(!CAMERA.focus);

    // Crosshair
    int crosshairSize = 10;
    int crosshairThick = 2;
    sf::Color crosshairColor = sf::Color::Green;
    sf::RectangleShape crosshair1(
        sf::Vector2f(crosshairSize + crosshairThick, crosshairThick)
    );
    crosshair1.setPosition(width / 2 - crosshairSize / 2, height / 2);
    crosshair1.setFillColor(crosshairColor);
    sf::RectangleShape crosshair2(
        sf::Vector2f(crosshairThick, crosshairSize + crosshairThick)
    );
    crosshair2.setPosition(width / 2, height / 2 - crosshairSize / 2);
    crosshair2.setFillColor(crosshairColor);

    int threads = 256;
    int blocks = (width * height + threads - 1) / threads;
    Ray *d_rays;
    hipMalloc(&d_rays, width * height * sizeof(Ray));
    Vec3f *d_framebuffer;
    hipMalloc(&d_framebuffer, width * height * sizeof(Vec3f));

    // Some test triangles
    Triangle tri1;
    tri1.v0 = Vec3f(-20, -20, -15);
    tri1.v1 = Vec3f(20, -20, -15);
    tri1.v2 = Vec3f(0, 20, -15);
    // tri1.c0 = Vec3f(1, 0.6, 0.6);
    // tri1.c1 = Vec3f(0.6, 1, 0.6);
    // tri1.c2 = Vec3f(0.6, 0.6, 1);
    tri1.uniformColor(Vec3f(1, 1, 1));
    tri1.uniformNormal(Vec3f(0, 0, 1));
    tri1.normAll();
    tri1.Fresnel = 1.0f;

    Triangle tri2;
    tri2.v0 = Vec3f(-180, -180, 10);
    tri2.v1 = Vec3f(180, -180, 10);
    tri2.v2 = Vec3f(0, 180, 11);
    tri2.c0 = Vec3f(1, 0, 0);
    tri2.c1 = Vec3f(0, 1, 0);
    tri2.c2 = Vec3f(0, 0, 1);
    tri2.uniformNormal(Vec3f(0, 0, -1));
    tri2.normAll();

    Triangle tri3;
    tri3.v0 = Vec3f(-5, -5, -32);
    tri3.v1 = Vec3f(5, -5, -32);
    tri3.v2 = Vec3f(0, 5, -32);
    tri3.c0 = Vec3f(1, 1, 0);
    tri3.c1 = Vec3f(0, 1, 1);
    tri3.c2 = Vec3f(1, 0, 1);
    tri3.uniformNormal(Vec3f(0, 0, 1));
    tri3.normAll();

    int triNum = 3;
    Triangle *d_triangles;
    hipMalloc(&d_triangles, triNum * sizeof(Triangle));
    hipMemcpy(d_triangles, &tri1, sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_triangles + 1, &tri2, sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_triangles + 2, &tri3, sizeof(Triangle), hipMemcpyHostToDevice);

    // Main loop
    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {      
                window.close();
            }

            // Press f1 to toggle camera focus
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::F1) {
                CAMERA.focus = !CAMERA.focus;
                // To avoid sudden camera movement when changing focus
                sf::Mouse::setPosition(sf::Vector2i(width / 2, height / 2), window);

                // Hide cursor
                window.setMouseCursorVisible(!CAMERA.focus);
            }
        }

        // Setting input activities
        bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
        bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);

        bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
        bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);

        bool k_w = sf::Keyboard::isKeyPressed(sf::Keyboard::W);
        bool k_a = sf::Keyboard::isKeyPressed(sf::Keyboard::A);
        bool k_s = sf::Keyboard::isKeyPressed(sf::Keyboard::S);
        bool k_d = sf::Keyboard::isKeyPressed(sf::Keyboard::D);
        bool k_space = sf::Keyboard::isKeyPressed(sf::Keyboard::Space);

        bool k_q = sf::Keyboard::isKeyPressed(sf::Keyboard::Q);
        bool k_e = sf::Keyboard::isKeyPressed(sf::Keyboard::E);
        bool k_t = sf::Keyboard::isKeyPressed(sf::Keyboard::T);

        // Camera movement
        if (CAMERA.focus) {
        // Camera look around
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(
                width / 2, height / 2
            ), window);

            // Move from center
            int dMx = mousepos.x - width / 2;
            int dMy = mousepos.y - height / 2;

            // Camera look around
            CAMERA.rot.x -= dMy * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.y += dMx * CAMERA.mSens * FPS.dTimeSec;

        // CSGO perspective movement
            float vel = CAMERA.velSpec;
            // Hold ctrl to go slow, hold shift to go fast
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;
            // Press W/S to move forward/backward
            if (k_w && !k_s) CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
            if (k_s && !k_w) CAMERA.pos -= CAMERA.forward * vel * FPS.dTimeSec;
            // Press A/D to move left/right
            if (k_a && !k_d) CAMERA.pos -= CAMERA.right * vel * FPS.dTimeSec;
            if (k_d && !k_a) CAMERA.pos += CAMERA.right * vel * FPS.dTimeSec;
        }
        // Update camera
        CAMERA.update();

        // Prepare framebuffer
        clearFrameBuffer<<<blocks, threads>>>(d_framebuffer, width, height);
        hipDeviceSynchronize();

        // Render framebuffer
        renderFrameBuffer<<<blocks, threads>>>(d_framebuffer, CAMERA, d_triangles, triNum, width, height);
        hipDeviceSynchronize();

        SFTex.updateTexture(d_framebuffer, width, height, 1);

        LOG.addLog(CAMERA.data(), sf::Color::White, 0);

        // Clear window
        window.clear();
        window.draw(SFTex.sprite);
        // Draw the crosshair
        window.draw(crosshair1);
        window.draw(crosshair2);

        LOG.drawLog(window);

        // For the time being just draw the window
        window.display();

        // Frame end
        FPS.endFrame();
    }

    return 0;
}
