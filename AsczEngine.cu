#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>

#include <Camera.cuh>
#include <SFMLTexture.cuh>
#include <Utility.cuh>

struct RayHit {
    bool hit = false;
    Vec3f vrtx;
    Vec2f txtr;
    Vec3f nrml;
    Vec3f colr;
    float t;
};

__device__ RayHit recursiveRayTracing(
    const Ray &ray, const Triangle *triangles, int triNum, int maxDepth
) {
    // Ray intersection with triangles
    int clstIdx = -1;
    float clstT = INFINITY;
    float clstU, clstV;

    for (int i = 0; i < triNum; i++) {
        const Triangle &tri = triangles[i];
        Vec3f e1 = tri.v1 - tri.v0;
        Vec3f e2 = tri.v2 - tri.v0;
        Vec3f h = ray.direction & e2;
        float a = e1 * h;

        if (a > -0.00001 && a < 0.00001) continue;

        float f = 1.0f / a;
        Vec3f s = ray.origin - tri.v0;
        float u = f * (s * h);

        if (u < 0.0f || u > 1.0f) continue;

        Vec3f q = s & e1;
        float v = f * (ray.direction * q);

        if (v < 0.0f || u + v > 1.0f) continue;

        float t = f * (e2 * q);

        if (t > 0.00001 && t < clstT) {
            clstIdx = i;
            clstT = t;
            clstU = u;
            clstV = v;
        }
    }

    if (clstIdx == -1) return RayHit();

    Triangle tri = triangles[clstIdx];
    float clstW = 1 - clstU - clstV;

    Vec3f vrtx = ray.origin + ray.direction * clstT;
    Vec3f nrml = triangles[clstIdx].n0 * clstW +
                    triangles[clstIdx].n1 * clstU +
                    triangles[clstIdx].n2 * clstV;
    Vec3f colr = triangles[clstIdx].c0 * clstW +
                    triangles[clstIdx].c1 * clstU +
                    triangles[clstIdx].c2 * clstV;

    if (tri.reflect > 0) {
        Vec3f reflDir = ray.direction - nrml * 2 * (ray.direction * nrml);
        reflDir.norm();
        Vec3f reflOrg = vrtx + nrml * 0.0001f; // To avoid self-intersection
        Ray reflRay(reflOrg, reflDir);

        RayHit reflHit = recursiveRayTracing(reflRay, triangles, triNum, maxDepth - 1);
        if (reflHit.hit) colr = colr * (1 - tri.reflect) + reflHit.colr * tri.reflect;
    }

    RayHit finalHit;
    finalHit.hit = true;
    finalHit.vrtx = vrtx;
    finalHit.nrml = nrml;
    finalHit.colr = colr;
    finalHit.t = clstT;
    return finalHit;
}


__global__ void clearFrameBuffer(Vec3f *framebuffer, int width, int height) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width * height) framebuffer[i] = Vec3f(0, 0, 0);
}

__global__ void renderFrameBuffer(
    Vec3f *framebuffer, Camera camera, Triangle *triangles, int triNum, int width, int height
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= width * height) return;

    int x = i % width;
    int y = i / width;

    Ray ray = camera.castRay(x, y, width, height);
    RayHit hit = recursiveRayTracing(ray, triangles, triNum, 4);

    if (hit.hit) framebuffer[i] = hit.colr;
}

int main() {
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();

    hipFree(0);  // Force context initialization
    hipDeviceSetLimit(hipLimitStackSize, 256 * 1024);

    // Create SFMLTexture
    int width = 1600;
    int height = 900;
    SFMLTexture SFTex(width, height);

    // Test camera
    Camera CAMERA;
    CAMERA.pos = Vec3f(0, 0, -10);
    CAMERA.rot = Vec3f(0, 0, 0);
    CAMERA.updateView();

    // Create window
    sf::RenderWindow window(sf::VideoMode(width, height), "AsczEngine");
    sf::Mouse::setPosition(sf::Vector2i(width / 2, height / 2), window);
    window.setMouseCursorVisible(!CAMERA.focus);

    // Crosshair
    int crosshairSize = 10;
    int crosshairThick = 2;
    sf::Color crosshairColor = sf::Color::Green;
    sf::RectangleShape crosshair1(
        sf::Vector2f(crosshairSize + crosshairThick, crosshairThick)
    );
    crosshair1.setPosition(width / 2 - crosshairSize / 2, height / 2);
    crosshair1.setFillColor(crosshairColor);
    sf::RectangleShape crosshair2(
        sf::Vector2f(crosshairThick, crosshairSize + crosshairThick)
    );
    crosshair2.setPosition(width / 2, height / 2 - crosshairSize / 2);
    crosshair2.setFillColor(crosshairColor);

    int threads = 256;
    int blocks = (width * height + threads - 1) / threads;
    Ray *d_rays;
    hipMalloc(&d_rays, width * height * sizeof(Ray));
    Vec3f *d_framebuffer;
    hipMalloc(&d_framebuffer, width * height * sizeof(Vec3f));

    // Some test triangles
    Triangle tri1;
    tri1.v0 = Vec3f(-10, -10, -5);
    tri1.v1 = Vec3f(10, -10, -5);
    tri1.v2 = Vec3f(0, 10, -5);
    tri1.uniformColor(Vec3f(1, 0, 0));
    tri1.uniformNormal(Vec3f(0, 0, 1));
    tri1.normAll();

    Triangle tri2;
    tri2.v0 = Vec3f(-10, -10, 5);
    tri2.v1 = Vec3f(10, -10, 5);
    tri2.v2 = Vec3f(0, 10, 5);
    tri2.uniformColor(Vec3f(0, 0, 1));
    tri2.uniformNormal(Vec3f(0, 0, -1));
    tri2.normAll();
    tri2.reflect = 0.5;

    int triNum = 2;
    Triangle *d_triangles;
    hipMalloc(&d_triangles, triNum * sizeof(Triangle));
    hipMemcpy(d_triangles, &tri1, sizeof(Triangle), hipMemcpyHostToDevice);
    hipMemcpy(d_triangles + 1, &tri2, sizeof(Triangle), hipMemcpyHostToDevice);

    // Main loop
    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {      
                window.close();
            }

            // Press f1 to toggle camera focus
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::F1) {
                CAMERA.focus = !CAMERA.focus;
                // To avoid sudden camera movement when changing focus
                sf::Mouse::setPosition(sf::Vector2i(width / 2, height / 2), window);

                // Hide cursor
                window.setMouseCursorVisible(!CAMERA.focus);
            }
        }

        // Setting input activities
        bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
        bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);

        bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
        bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);

        bool k_w = sf::Keyboard::isKeyPressed(sf::Keyboard::W);
        bool k_a = sf::Keyboard::isKeyPressed(sf::Keyboard::A);
        bool k_s = sf::Keyboard::isKeyPressed(sf::Keyboard::S);
        bool k_d = sf::Keyboard::isKeyPressed(sf::Keyboard::D);
        bool k_space = sf::Keyboard::isKeyPressed(sf::Keyboard::Space);

        bool k_q = sf::Keyboard::isKeyPressed(sf::Keyboard::Q);
        bool k_e = sf::Keyboard::isKeyPressed(sf::Keyboard::E);
        bool k_t = sf::Keyboard::isKeyPressed(sf::Keyboard::T);

        // Camera movement
        if (CAMERA.focus) {
        // Camera look around
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(
                width / 2, height / 2
            ), window);

            // Move from center
            int dMx = mousepos.x - width / 2;
            int dMy = mousepos.y - height / 2;

            // Camera look around
            CAMERA.rot.x -= dMy * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.y += dMx * CAMERA.mSens * FPS.dTimeSec;

        // CSGO perspective movement
            float vel = CAMERA.velSpec;
            // Hold ctrl to go slow, hold shift to go fast
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;
            // Press W/S to move forward/backward
            if (k_w && !k_s) CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
            if (k_s && !k_w) CAMERA.pos -= CAMERA.forward * vel * FPS.dTimeSec;
            // Press A/D to move left/right
            if (k_a && !k_d) CAMERA.pos -= CAMERA.right * vel * FPS.dTimeSec;
            if (k_d && !k_a) CAMERA.pos += CAMERA.right * vel * FPS.dTimeSec;
        }
        // Update camera
        CAMERA.update();

        // Prepare framebuffer
        clearFrameBuffer<<<blocks, threads>>>(d_framebuffer, width, height);
        hipDeviceSynchronize();

        // Render framebuffer
        renderFrameBuffer<<<blocks, threads>>>(d_framebuffer, CAMERA, d_triangles, triNum, width, height);
        hipDeviceSynchronize();

        SFTex.updateTexture(d_framebuffer, width, height, 1);

        LOG.addLog(CAMERA.data(), sf::Color::White, 0);

        // Clear window
        window.clear();
        window.draw(SFTex.sprite);
        // Draw the crosshair
        window.draw(crosshair1);
        window.draw(crosshair2);

        LOG.drawLog(window);

        // For the time being just draw the window
        window.display();

        // Frame end
        FPS.endFrame();
    }

    return 0;
}
