#include "hip/hip_runtime.h"
#include <FpsHandler.cuh>
#include <CsLogHandler.cuh>

#include <hip/hip_runtime.h>

#include <Camera.cuh>
#include <SFMLTexture.cuh>

/* Goal:

Ray intersection with AABB: Done
Ray reflection: Done
Create rays from camera: Done
Ray casting:

*/

struct Triangle {
    Vec3f v0, v1, v2;
    // Vec2f t0, t1, t2; // Will be ignored for now
    Vec3f c1, c2, c3; // Placeholder, to test interpolation
    Vec3f n0, n1, n2;

    bool reflect = false; // Test reflection
};

__global__ void clearFramebuffer(Vec3f *framebuffer, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    framebuffer[idx] = Vec3f(0, 0, 0);
}

__global__ void generateRays(Camera camera, Ray *rays, int width, int height) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    int x = idx % width;
    int y = idx / width;

    rays[y * width + x] = camera.castRay(x, y, width, height);
}

__global__ void castRays(
    Vec3f *framebuffer, Ray *rays, bool *raycursive,
    Triangle *triangles, int width, int height, int triangleCount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height) return;

    Ray ray = rays[idx];

    Ray recursiveRay;
    bool recursive = false;

    float zdepth = 1000000.0f;
    // This will soon be replaced with BVH traversal method
    for (int i = 0; i < triangleCount; i++) {
        Vec3f A = triangles[i].v0;  
        Vec3f B = triangles[i].v1;
        Vec3f C = triangles[i].v2;

        Vec3f e1 = B - A;
        Vec3f e2 = C - A;

        Vec3f h = ray.direction & e2;
        float a = e1 * h;

        // Ray is parallel to the triangle
        if (a > -0.00001 && a < 0.00001) continue;

        float f = 1.0f / a;
        Vec3f s = ray.origin - A;
        float u = f * (s * h);

        if (u < 0.0f || u > 1.0f) continue;

        Vec3f q = s & e1;
        float v = f * (ray.direction * q);

        if (v < 0.0f || u + v > 1.0f) continue;

        float t = f * (e2 * q);

        if (t > 0.00001 && t < zdepth) {
            zdepth = t;

            // If reflective, the resulting ray will be the reflection of the current ray
            if (triangles[i].reflect) {
                recursiveRay.origin = ray.origin + ray.direction * t;
                recursiveRay.direction = ray.reflect((B - A) & (C - A));
                recursive = true;

                continue;
            }

            recursive = false;

            // Interpolate color
            Vec3f color = triangles[i].c1 * (1 - u - v) + triangles[i].c2 * u + triangles[i].c3 * v;
            framebuffer[idx] = color;
            // Interpolate normal
            Vec3f normal = triangles[i].n0 * (1 - u - v) + triangles[i].n1 * u + triangles[i].n2 * v;
        }
    }

    if (recursive) {
        rays[idx] = recursiveRay;
        atmoicAdd(raycursive, 1);
    }
}   

int main() {
    /*
    Isn't it funny how these things have been here
    For 3 entire versions of the engine?
    They been through 3 "code wars"
    */
    FpsHandler &FPS = FpsHandler::instance();
    CsLogHandler LOG = CsLogHandler();

    // Create SFMLTexture
    int width = 600;
    int height = 600;
    SFMLTexture SFTex(600, 600);

    // Test camera
    Camera CAMERA;
    CAMERA.pos = Vec3f(0, 0, 0);
    CAMERA.rot = Vec3f(0, 0, 0);
    CAMERA.updateView();

    Ray *d_rays;
    Vec3f *d_framebuffer;
    hipMalloc(&d_rays, width * height * sizeof(Ray));
    hipMalloc(&d_framebuffer, width * height * sizeof(Vec3f));

    // For ray recursion
    Ray *d_raycursive; // Pun intended
    bool *d_hasreflected;
    hipMalloc(&d_raycursive, width * height * sizeof(Ray));
    hipMalloc(&d_hasreflected, sizeof(bool));

    int threads = 256;
    int blocks = (width * height + threads - 1) / threads;

    // Creating some test triangles
    Triangle *d_triangles;
    int triCount = 4;
    hipMalloc(&d_triangles, triCount * sizeof(Triangle));

    Triangle triangles[4];
    // Postive Z
    triangles[0].v0 = Vec3f(-10, -10, 50);
    triangles[0].v1 = Vec3f(10, -10, 50);
    triangles[0].v2 = Vec3f(0, 10, 50);
    triangles[0].c1 = Vec3f(1, 0, 0);   
    triangles[0].c2 = Vec3f(0, 1, 0);
    triangles[0].c3 = Vec3f(0, 0, 1);

    // Negative Z
    triangles[1].v0 = Vec3f(-10, -10, -50);
    triangles[1].v1 = Vec3f(10, -10, -50);
    triangles[1].v2 = Vec3f(0, 10, -50);
    triangles[1].c1 = Vec3f(1, 0, 0);
    triangles[1].c2 = Vec3f(1, 1, 0);
    triangles[1].c3 = Vec3f(0, 1, 0);
    triangles[1].reflect = true;

    // Positive X
    triangles[2].v0 = Vec3f(50, -10, -10);
    triangles[2].v1 = Vec3f(50, -10, 10);
    triangles[2].v2 = Vec3f(50, 10, 0);
    triangles[2].c1 = Vec3f(0, 1, 0);
    triangles[2].c2 = Vec3f(0, 0, 1);
    triangles[2].c3 = Vec3f(1, 0, 0);

    // Negative X
    triangles[3].v0 = Vec3f(-50, -10, -10);
    triangles[3].v1 = Vec3f(-50, -10, 10);
    triangles[3].v2 = Vec3f(-50, 10, 0);
    triangles[3].c1 = Vec3f(0, 1, 0);
    triangles[3].c2 = Vec3f(0, 0, 1);
    triangles[3].c3 = Vec3f(1, 0, 0);

    hipMemcpy(d_triangles, triangles, triCount * sizeof(Triangle), hipMemcpyHostToDevice);   

    sf::RenderWindow window(sf::VideoMode(width, height), "AsczEngine");

    while (window.isOpen()) {
        // Frame start
        FPS.startFrame();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed ||
                sf::Keyboard::isKeyPressed(sf::Keyboard::Escape)) {      
                window.close();
            }

            // Press f1 to toggle camera focus
            if (event.type == sf::Event::KeyPressed && event.key.code == sf::Keyboard::F1) {
                CAMERA.focus = !CAMERA.focus;
                sf::Mouse::setPosition(sf::Vector2i(width / 2, height / 2), window);

                // Hide cursor
                window.setMouseCursorVisible(!CAMERA.focus);
            }
        }

        // Setting input activities
        bool m_left = sf::Mouse::isButtonPressed(sf::Mouse::Left);
        bool m_right = sf::Mouse::isButtonPressed(sf::Mouse::Right);

        bool k_ctrl = sf::Keyboard::isKeyPressed(sf::Keyboard::LControl);
        bool k_shift = sf::Keyboard::isKeyPressed(sf::Keyboard::LShift);

        bool k_w = sf::Keyboard::isKeyPressed(sf::Keyboard::W);
        bool k_a = sf::Keyboard::isKeyPressed(sf::Keyboard::A);
        bool k_s = sf::Keyboard::isKeyPressed(sf::Keyboard::S);
        bool k_d = sf::Keyboard::isKeyPressed(sf::Keyboard::D);
        bool k_space = sf::Keyboard::isKeyPressed(sf::Keyboard::Space);

        bool k_q = sf::Keyboard::isKeyPressed(sf::Keyboard::Q);
        bool k_e = sf::Keyboard::isKeyPressed(sf::Keyboard::E);
        bool k_t = sf::Keyboard::isKeyPressed(sf::Keyboard::T);

        // Camera movement
        if (CAMERA.focus) {
        // Camera look around
            sf::Vector2i mousepos = sf::Mouse::getPosition(window);
            sf::Mouse::setPosition(sf::Vector2i(
                width / 2, height / 2
            ), window);

            // Move from center
            int dMx = mousepos.x - width / 2;
            int dMy = mousepos.y - height / 2;

            // Camera look around
            CAMERA.rot.x -= dMy * CAMERA.mSens * FPS.dTimeSec;
            CAMERA.rot.y -= dMx * CAMERA.mSens * FPS.dTimeSec;

        // Csgo perspective mode movement
            float vel = CAMERA.velSpec;
            // Hold ctrl to go slow, hold shift to go fast
            if (k_ctrl && !k_shift)      vel *= CAMERA.slowFactor;
            else if (k_shift && !k_ctrl) vel *= CAMERA.fastFactor;
            // Press W/S to move forward/backward
            if (k_w && !k_s) CAMERA.pos += CAMERA.forward * vel * FPS.dTimeSec;
            if (k_s && !k_w) CAMERA.pos -= CAMERA.forward * vel * FPS.dTimeSec;
            // Press A/D to move left/right
            if (k_a && !k_d) CAMERA.pos -= CAMERA.right * vel * FPS.dTimeSec;
            if (k_d && !k_a) CAMERA.pos += CAMERA.right * vel * FPS.dTimeSec;
        }
        // Update camera
        CAMERA.update();

        // Clear framebuffer
        clearFramebuffer<<<blocks, threads>>>(d_framebuffer, width, height);    
        hipDeviceSynchronize();

        // Generate rays
        generateRays<<<blocks, threads>>>(CAMERA, d_rays, width, height);
        hipDeviceSynchronize();

        // Cast rays
        castRays<<<blocks, threads>>>(d_framebuffer, d_rays, d_triangles, width, height, triCount);
        hipDeviceSynchronize();

        // Log
        LOG.addLog("Welcome to AsczEngine RTx", sf::Color::White, 1);
        LOG.addLog("FPS: " + std::to_string(FPS.fps), sf::Color::White);

        // Draw to window
        SFTex.updateTexture(d_framebuffer, width, height, 1);
        window.clear(sf::Color::Black);
        window.draw(SFTex.sprite);
        LOG.drawLog(window);
        window.display();

        // Frame end
        FPS.endFrame();
    }

    return 0;
}